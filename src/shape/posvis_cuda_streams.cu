#include "hip/hip_runtime.h"
/*****************************************************************************************
 posvis.c

 Fill in the portion of a plane-of-sky image due to a particular model component: Assign
 each relevant POS pixel a z-value in observer coordinates (distance from the origin
 towards Earth) and a value of cos(scattering angle).

 Return 1 if any portion of this component lies outside the specified POS window,
 0 otherwise.

 If the "src" argument is true, the "observer" is the Sun rather than Earth, and
 "plane-of-sky" becomes "projection as viewed from the Sun."

 Modified 2014 February 20 by CM:
 Allow facets that partly project outside the POS frame to contribute to the POS frame
 (thus avoiding see-through "holes" in the model at the edge of a POS image)

 Modified 2010 May 18 by CM:
 Bug fix: When checking if a POS pixel hasn't already been assigned
 values during a previous call to posvis for a different component,
 check for fac[i][j] < 0 rather than cosa[i][j] == 0.0, since for
 bistatic situations the latter condition will also be true for
 pixels centered on Earth-facing facets that don't face the Sun

 Modified 2009 July 2 by CM:
 Eliminate the check that facets are "active": this term is now being
 interpreted to mean "not lying interior to the model," so the
 check is unnecessary and the determination of active vs. inactive
 status is inaccurate for half-exposed facets at the intersections
 between model components

 Modified 2009 April 3 by CM:
 Compute the "posbnd_logfactor" parameter: if the model extends beyond
 the POS frame, posbnd_logfactor is set to the logarithm of the
 ratio of the area that would have been required to "contain" the
 entire model divided by the area of the actual POS frame
 Work with floating-point pixel numbers (imin_dbl, etc.), at least
 initially, in case the sky rendering for a model with illegal
 parameters would involve huge pixel numbers that exceed the
 limits for valid integers

 Modified 2007 August 4 by CM:
 Add "orbit_offset" and "body" parameters and remove "facet" parameter
 Add body, bodyill, comp, and compill matrices for POS frames

 Modified 2006 June 21 by CM:
 For POS renderings, change res to km_per_pixel

 Modified 2005 September 19 by CM:
 Allow for roundoff error when determining which POS pixels project
 onto each model facet

 Modified 2005 June 27 by CM:
 Renamed "round" function to "iround" to avoid conflicts

 Modified 2005 June 22 by CM:
 Slightly modified some comments

 Modified 2005 January 25 by CM:
 Take care of unused and uninitialized variables

 Modified 2004 December 19 by CM:
 Added more comments
 Put update of rectangular POS area into "POSrect" routine and applied it
 even to facets which lie outside the POS frame

 Modified 2004 Feb 11 by CM:
 Added comments

 Modified 2003 May 5 by CM:
 Removed redundant coordinate transformation of the unit normal n
 for the no-pvs_smoothing case
 *****************************************************************************************/
extern "C" {
#include "head.h"
#include <limits.h>
}
__device__ int posvis_streams_outbnd, pvst_nf, pvst_smooth;
__device__ struct vertices_t *pvst_verts;

/* Note that the following two custom atomic functions are declared in each
 * file they are needed .  As static __device__ functions, this is the only
 * way to handle them. */
__device__ static float atomicMinf(float* address, float val) {
	int* address_as_i = (int*) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__float_as_int(::fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}
__device__ static float atomicMaxf(float* address, float val) {
	int* address_as_i = (int*) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
				__float_as_int(::fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}
__device__ void dev_POSrect_streams(
		struct pos_t **pos,
		int src,
		float imin_dbl,
		float imax_dbl,
		float jmin_dbl,
		float jmax_dbl,
		float4 *ijminmax_overall,
		int frm)	{
	int n, imin, imax, jmin, jmax;
	n = pos[frm]->n;

	/* Update the POS region that contains the target without
	 * regard to whether or not it extends beyond the POS frame */
	atomicMinf(&ijminmax_overall[frm].w, imin_dbl);
	atomicMaxf(&ijminmax_overall[frm].x, imax_dbl);
	atomicMinf(&ijminmax_overall[frm].y, jmin_dbl);
	atomicMaxf(&ijminmax_overall[frm].z, jmax_dbl);

	/*  Update the subset of the POS frame that contains the target  */
	imin = (imin_dbl < INT_MIN) ? INT_MIN : (int) imin_dbl;
	imax = (imax_dbl > INT_MAX) ? INT_MAX : (int) imax_dbl;
	jmin = (jmin_dbl < INT_MIN) ? INT_MIN : (int) jmin_dbl;
	jmax = (jmax_dbl > INT_MAX) ? INT_MAX : (int) jmax_dbl;

	/* Make sure it's smaller than n */
	imin = MAX(imin,-n);
	imax = MIN(imax, n);
	jmin = MAX(jmin,-n);
	jmax = MIN(jmax, n);

	if (src) {
		atomicMin(&pos[frm]->xlim2[0], imin);
		atomicMax(&pos[frm]->xlim2[1], imax);
		atomicMin(&pos[frm]->ylim2[0], jmin);
		atomicMax(&pos[frm]->ylim2[1], jmax);
	} else {
		atomicMin(&pos[frm]->xlim[0], imin);
		atomicMax(&pos[frm]->xlim[1], imax);
		atomicMin(&pos[frm]->ylim[0], jmin);
		atomicMax(&pos[frm]->ylim[1], jmax);
	}
}
__global__ void posvis_init_streams_krnl(
		struct pos_t **pos,
		double3 *oa,
		double3 *usrc,
		int frm,
		int src,
		int *outbndarr) {

	/* Single-threaded kernel */
	if (threadIdx.x == 0) {

		dev_mtrnsps2(oa, pos[frm]->ae, frm);

		if (src) {
			/* We're viewing the model from the sun: at the center of each pixel
			 * in the projected view, we want cos(incidence angle), distance from
			 * the COM towards the sun, and the facet number.                */
			dev_mmmul2(oa, pos[frm]->se, oa, frm); /* oa takes ast into sun coords           */
		} else {
			/* We're viewing the model from Earth: at the center of each POS pixel
			 * we want cos(scattering angle), distance from the COM towards Earth,
			 * and the facet number.  For bistatic situations (lightcurves) we also
							 want cos(incidence angle) and the unit vector towards the source.     */
			dev_mmmul2(oa, pos[frm]->oe, oa, frm); /* oa takes ast into obs coords */
			if (pos[frm]->bistatic) {
				usrc[frm].x = usrc[frm].y = 0.0; /* unit vector towards source */
				usrc[frm].z = 1.0;
				dev_cotrans5(&usrc[frm], pos[frm]->se, usrc[frm], -1);
				dev_cotrans5(&usrc[frm], pos[frm]->oe, usrc[frm], 1); /* in observer coordinates */
			}
		}
		outbndarr[frm] = 0;
	}
}
__global__ void posvis_facet_streams_krnl(
		struct pos_t **pos,
		struct vertices_t *verts,
		float4 *ijminmax_overall,
		float3 orbit_offs,
		double3 *oa,
		double3 *usrc,
		int src,
		int body,
		int comp,
		int nfacets,
		int frm,
		int smooth,
		int *outbndarr) {
	/* (nf * nframes)-threaded kernel */

	int f = blockIdx.x * blockDim.x + threadIdx.x;
	int pxa, k, i, i1, i2, j, j1, j2, imin, imax, jmin, jmax;
	double n[3], v0[3], v1[3], v2[3], x[3], s, t, z, den;
	float imin_dbl, imax_dbl, jmin_dbl, jmax_dbl, old;
	int3 fidx;

	if (f < nfacets) {

		fidx.x = verts->f[f].v[0];
		fidx.y = verts->f[f].v[1];
		fidx.z = verts->f[f].v[2];

		/* Get the normal to this facet in body-fixed (asteroid) coordinates
		 * and convert it to observer coordinates     */
		for (i = 0; i <= 2; i++)
			n[i] = verts->f[f].n[i];

		dev_cotrans6(n, oa, n, 1, frm);
		//dev_cotrans3(n, oa, n, 1);

		/* Consider this facet further only if its normal points somewhat
		 * towards the observer rather than away         */
		if (n[2] > 0.0) {
			/* Convert the three sets of vertex coordinates from body to ob-
			 * server coordinates; orbit_offset is the center-of-mass offset
			 * (in observer coordinates) for this model at this frame's epoch
			 * due to orbital motion, in case the model is half of a binary
			 * system.  */
			dev_cotrans6(v0, oa, verts->v[fidx.x].x, 1, frm);
			dev_cotrans6(v1, oa, verts->v[fidx.y].x, 1, frm);
			dev_cotrans6(v2, oa, verts->v[fidx.z].x, 1, frm);
//			dev_cotrans3(v0, oa, verts->v[fidx.x].x, 1);
//			dev_cotrans3(v1, oa, verts->v[fidx.y].x, 1);
//			dev_cotrans3(v2, oa, verts->v[fidx.z].x, 1);
			for (i = 0; i <= 2; i++) {
				v0[i] += orbit_offs.x;
				v1[i] += orbit_offs.y;
				v2[i] += orbit_offs.z;
			}

			/* Find rectangular region (in POS pixels) containing the projected
			 * facet - use floats in case model has illegal parameters and the
			 * pixel numbers exceed the limits for valid integers                         */
			imin_dbl = floor(MIN(v0[0],MIN(v1[0],v2[0])) / pos[frm]->km_per_pixel
							- SMALLVAL + 0.5);
			imax_dbl = floor(MAX(v0[0],MAX(v1[0],v2[0])) / pos[frm]->km_per_pixel
							+ SMALLVAL + 0.5);
			jmin_dbl = floor(MIN(v0[1],MIN(v1[1],v2[1])) / pos[frm]->km_per_pixel
							- SMALLVAL + 0.5);
			jmax_dbl = floor(MAX(v0[1],MAX(v1[1],v2[1])) / pos[frm]->km_per_pixel
							+ SMALLVAL + 0.5);
			imin = (imin_dbl < INT_MIN) ? INT_MIN : (int) imin_dbl;
			imax = (imax_dbl > INT_MAX) ? INT_MAX : (int) imax_dbl;
			jmin = (jmin_dbl < INT_MIN) ? INT_MIN : (int) jmin_dbl;
			jmax = (jmax_dbl > INT_MAX) ? INT_MAX : (int) jmax_dbl;

			/*  Set the outbnd flag if the facet extends beyond the POS window  */
			if ((imin < (-pos[frm]->n)) || (imax > pos[frm]->n) ||
					(jmin < (-pos[frm]->n))	|| (jmax > pos[frm]->n)) {
				posvis_streams_outbnd = 1;
				outbndarr[f] = 1;
			}

			/* Figure out if facet projects at least partly within POS window;
			 * if it does, look at each "contained" POS pixel and get the
			 * z-coordinate and cos(scattering angle)           */
			i1 = MAX(imin, -pos[frm]->n);		j1 = MAX(jmin, -pos[frm]->n);
			i2 = MIN(imax,  pos[frm]->n);		j2 = MIN(jmax,  pos[frm]->n);

			if (i1 > pos[frm]->n || i2 < -pos[frm]->n || j1 > pos[frm]->n || j2 < -pos[frm]->n) {

				/* Facet is entirely outside the POS frame: just keep track of
				 * changed POS region     */
				dev_POSrect_streams(pos, src, imin_dbl, imax_dbl, jmin_dbl, jmax_dbl,
						ijminmax_overall, frm);

			} else {

				dev_POSrect_streams(pos, src, (float)i1, (float)i2, (float)j1,
						(float)j2, ijminmax_overall, frm);

				/* Facet is at least partly within POS frame: find all POS
				 * pixels whose centers project onto this facet  */
				for (i = i1; i <= i2; i++) {
					x[0] = i * pos[frm]->km_per_pixel;
					for (j = j1; j <= j2; j++) {
						x[1] = j * pos[frm]->km_per_pixel;

						/* Calculate the pixel address for 1D arrays */
						pxa = (j+pos[frm]->n) * (2*pos[frm]->n + 1) + (i+pos[frm]->n);

						/* Compute parameters s(x,y) and t(x,y) which define a
						 * facet's surface as
						 *         z = z0 + s*(z1-z0) + t*(z2-z1)
						 * where z0, z1, and z2 are the z-coordinates at the
						 * vertices. The conditions 0 <= s <= 1 and
						 * 0 <= t <= s require the POS pixel center to be
						 * "within" the (projected) perimeter of facet f.    */
						den = 1	/ ((v1[0] - v0[0]) * (v2[1] - v1[1])
								 - (v2[0] - v1[0]) * (v1[1] - v0[1]));
						s = ((x[0] - v0[0]) * (v2[1] - v1[1])
						  - (v2[0] - v1[0]) * (x[1] - v0[1])) * den;

						if ((s >= -SMALLVAL) && (s <= 1.0 + SMALLVAL)) {

							t = ((v1[0] - v0[0]) * (x[1] - v0[1])
							    - (x[0] - v0[0]) * (v1[1]- v0[1])) * den;
							if ((t >= -SMALLVAL) && (t <= s + SMALLVAL)) {

								/* Compute z-coordinate of pixel center: its
								 * distance measured from the origin towards
								 * Earth.    */
								z = v0[2] + s*(v1[2]-v0[2]) + t*(v2[2]-v1[2]);

								/* If fac[i][j] is >= 0, pixel [i][j] was al-
								 * ready assigned values during a previous call
								 * to posvis for a different model component.
								 * If so, override only if the current component
								 * is blocking our view of (i.e., is closer to
								 * us than) the previous one.   */

								/* Following line replaces the previous if check
								 * for z > zz[i][j]
								 * atomicMaxf returns the value that was sitting
								 * at zzf[pxa] at time of call.  So if that value
								 * matches the z we compared to*/

								if (src)
									old = atomicMaxf(&pos[frm]->zill_s[pxa], z);
								else
									old = atomicMaxf(&pos[frm]->z_s[pxa], z);

								if (old < z || pos[frm]->fill[i][j] < 0 ||
										pos[frm]->f[i][j] < 0) {

									/* Next line assigns distance of POS pixel
									 * center from COM towards Earth; that is,
									 * by changing zz,it changes pos->z or
									 * pos->zill                */
									/* following line is a first time z calc
									 * for this pixel  */
									if ( (pos[frm]->fill[i][j] < 0) || (pos[frm]->f[i][j] < 0)){
										if (src)	atomicExch(&pos[frm]->zill_s[pxa], z);
										else 		atomicExch(&pos[frm]->z_s[pxa], z);
									}

									if (smooth) {

										/* Get pvs_smoothed version of facet unit
										 * normal: Take the linear combination
										 * of the three vertex normals; trans-
										 * form from body to observer coordina-
										 * tes; and make sure that it points
										 * somewhat in our direction.         */
										for (k = 0; k <= 2; k++)
											n[k] =	verts->v[fidx.x].n[k]
											 + s * (verts->v[fidx.y].n[k] - verts->v[fidx.x].n[k])
											 + t * (verts->v[fidx.z].n[k] - verts->v[fidx.y].n[k]);
										dev_cotrans6(n, oa, n, 1, frm);
										//dev_cotrans3(n, oa, n, 1);
										dev_normalize(n);
									}

									/* Determine scattering and/or incidence
									 * angles. Next lines change pos->cose/
									 * cosill. If bistatic (lightcurves), where
									 * we are viewing from Earth (src = 0),
									 * pos->cosi is also changed.                 */
									if (n[2] > 0.0) {
										if (src) atomicExch(&pos[frm]->cosill_s[pxa], n[2]);
										else	 atomicExch(&pos[frm]->cose_s[pxa], n[2]);
										if ((!src) && (pos[frm]->bistatic)) {
											float temp = (float)dev_dot2(n,usrc[f]);
											atomicExch(&pos[frm]->cosi_s[pxa], temp);
											if (pos[frm]->cosi_s[pxa] <= 0.0)
												pos[frm]->cose_s[pxa] = 0.0;
										}
									}

									/* Next lines change pos->body/bodyill,
									 * pos->comp/compill, pos->f/fill          */
									if (src) {
										pos[frm]->bodyill[i][j] = body;
										pos[frm]->compill[i][j] = comp;
										pos[frm]->fill[i][j] = f;
									} else {
										pos[frm]->body[i][j] = body;
										pos[frm]->comp[i][j] = comp;
										pos[frm]->f[i][j] = f;
									}

								} /* end if (no other facet yet blocks this facet from view) */
							} /* end if 0 <= t <= s (facet center is "in" this POS pixel) */
						} /* end if 0 <= s <= 1 */
					} /* end j-loop over POS rows */
				} /* end i-loop over POS columns */
			} /* end else of if (i1 > pos->n || i2 < -pos->n || j1 > pos->n || j2 < -pos->n) */
		} /* End if (n[2] > 0.0) */
	} /* end if (f < nf) */
}
__host__ int posvis_cuda_streams(struct par_t *dpar, struct mod_t *dmod,
		struct dat_t *ddat, double orbit_offset[3], int set, int nframes,
		int src, int body, int comp, int *outbndarr, hipStream_t *posvis_stream) {
	int nf, outbnd, smooth;
	double xfactor, yfactor;
	unsigned char type;
	dim3 BLK,THD;

	struct pos_t **pos;
	struct vertices_t *verts;
	float4 *ijminmax_overall;
	double3 *oa, *usrc;
	float3 orbit_offs;
	int posn[nframes];

	/* Allocate temporary arrays/structs */
	cudaCalloc1((void**)&ijminmax_overall, sizeof(float4), nframes);
	cudaCalloc1((void**)&pos, sizeof(struct pos_t*), nframes);
	cudaCalloc1((void**)&verts, sizeof(struct vertices_t), nframes);
	cudaCalloc1((void**)&oa, sizeof(double3), (nframes*3));
	cudaCalloc1((void**)&usrc, sizeof(double3), nframes);

	/* Create the timer events */
	hipEvent_t start, stop;
	float milliseconds;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/* Transfer orbit offset to a better variable type for CUDA */
	orbit_offs.x = orbit_offset[0];
	orbit_offs.y = orbit_offset[1];
	orbit_offs.z = orbit_offset[2];

	verts = &dmod->shape.comp[comp].real;
	type = ddat->set[set].type;
	nf = dmod->shape.comp[comp].real.nf;
	smooth = dpar->pos_smooth;

	/* Init on host-side */
	for (int f=0; f<nframes; f++) {
		ijminmax_overall[f].w = ijminmax_overall[f].y = HUGENUMBER;
		ijminmax_overall[f].x = ijminmax_overall[f].z = -HUGENUMBER;
		if (type==DELAY)	pos[f] = &ddat->set[set].desc.deldop.frame[f].pos;
		if (type==DOPPLER)	pos[f] = &ddat->set[set].desc.doppler.frame[f].pos;
		if (type==POS)		pos[f] = &ddat->set[set].desc.poset.frame[f].pos;
		if (type==LGHTCRV)	pos[f] = &ddat->set[set].desc.lghtcrv.rend[f].pos;
		pos[f]->posbnd_logfactor = 0.0;
		posn[f] = pos[f]->n;
	}

	/* Launch parameters for the facet_streams kernel */
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + nf) / THD.x);
	if (TIMING)	hipEventRecord(start);

	for (int f=0; f<nframes; f++) {

		/* Initialize via single-thread kernel first */
		posvis_init_streams_krnl<<<1,1,0,posvis_stream[f]>>>(pos, oa, usrc,
				f, src, outbndarr);

		/* Now the main facet kernel */
		posvis_facet_streams_krnl<<<BLK,THD, 0, posvis_stream[f]>>>(pos, verts,
				ijminmax_overall, orbit_offs, oa, usrc,	src, body, comp,
				nf, f, smooth, outbndarr);
		//hipDeviceSynchronize();
	}

	//hipDeviceSynchronize();
	if (TIMING) {
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("%i facets in posvis_cuda_2 in %3.3f ms.\n", nf, milliseconds);
	}
	checkErrorAfterKernelLaunch("posvis_facet_streams_krnl");
	gpuErrchk(hipMemcpyFromSymbol(&outbnd, HIP_SYMBOL(posvis_streams_outbnd), sizeof(outbnd), 0,
			hipMemcpyDeviceToHost));
//	for (int f=0; f<nframes; f++)
//		hipStreamSynchronize(posvis_stream[f]);
//dbg_print_pos_z(ddat, set, 0, posn[0]);
	if (outbnd) {
		for (int f=0; f<nframes; f++) {
			/* ijminmax_overall.w = imin_overall
			 * ijminmax_overall.x = imax_overall
			 * ijminmax_overall.y = jmin_overall
			 * ijminmax_overall.z = jmax_overall	 */
			xfactor = (MAX( ijminmax_overall[f].x,  posn[f]) -
					MIN( ijminmax_overall[f].w, -posn[f]) + 1) / (2*posn[f]+1);
			yfactor = (MAX( ijminmax_overall[f].z,  posn[f]) -
					MIN( ijminmax_overall[f].y, -posn[f]) + 1) / (2*posn[f]+1);
			pos[f]->posbnd_logfactor = log(xfactor*yfactor);
		}
	}

	/* Free temp arrays, destroy streams and timers, as applicable */
	hipFree(pos);
	hipFree(ijminmax_overall);
	hipFree(oa);
	//hipFree(verts);

	if (TIMING) {
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
	return outbnd;
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

__global__ void posvis_streams_init_krnl(
		struct par_t *dpar,
		struct pos_t **pos,
		float4 *ijminmax_overall,
		float3 *oa,
		float3 *usrc,
		int *outbndarr,
		int c,
		int f,
		int start,
		int end,
		int src) {

	/* Single-threaded, streamed kernel */
	if (threadIdx.x == 0) {
		if (f == start) {
			posvis_streams_outbnd = 0;
			pvst_smooth = dpar->pos_smooth;
		}
		ijminmax_overall[f].w = ijminmax_overall[f].y = HUGENUMBER;
		ijminmax_overall[f].x = ijminmax_overall[f].z = -HUGENUMBER;
		pos[f]->posbnd_logfactor = 0.0;

		dev_mtrnsps3(oa, pos[f]->ae, f);

		if (src) {
			/* We're viewing the model from the sun: at the center of each pixel
			 * in the projected view, we want cos(incidence angle), distance from
			 * the COM towards the sun, and the facet number.                */
			dev_mmmul3(oa, pos[f]->se, oa, f); /* oa takes ast into sun coords           */
		} else {
			/* We're viewing the model from Earth: at the center of each POS pixel
			 * we want cos(scattering angle), distance from the COM towards Earth,
			 * and the facet number.  For bistatic situations (lightcurves) we also
									 want cos(incidence angle) and the unit vector towards the source.     */
			dev_mmmul3(oa, pos[f]->oe, oa, f); /* oa takes ast into obs coords */
			if (pos[f]->bistatic) {
				usrc[f].x = usrc[f].y = 0.0; /* unit vector towards source */
				usrc[f].z = 1.0;
				dev_cotrans9(&usrc[f], pos[f]->se, usrc[f], -1);
				dev_cotrans9(&usrc[f], pos[f]->oe, usrc[f], 1); /* in observer coordinates */
			}
		}
		outbndarr[f] = 0;
	}
}
__global__ void posvis_facet_streams2_krnl(
		struct pos_t **pos,
		struct vertices_t **verts,
		float4 *ijminmax_overall,
		float3 orbit_offs,
		double3 *oa,
		double3 *usrc,
		int src,
		int body,
		int comp,
		int nfacets,
		int frm,
		int smooth,
		int *outbndarr) {
	/* (nf * nframes)-threaded kernel */

	int f = blockIdx.x * blockDim.x + threadIdx.x;
	int pxa, k, i, i1, i2, j, j1, j2, imin, imax, jmin, jmax;
	double n[3], v0[3], v1[3], v2[3], x[3], s, t, z, den;
	float imin_dbl, imax_dbl, jmin_dbl, jmax_dbl, old, kmpxl;
	int3 fidx;

	if (f < nfacets) {
		kmpxl = (float)pos[frm]->km_per_pixel;
		fidx.x = verts[0]->f[f].v[0];
		fidx.y = verts[0]->f[f].v[1];
		fidx.z = verts[0]->f[f].v[2];

		/* Get the normal to this facet in body-fixed (asteroid) coordinates
		 * and convert it to observer coordinates     */
		for (i = 0; i <= 2; i++)
			n[i] = verts[0]->f[f].n[i];

		dev_cotrans6(n, oa, n, 1, frm);
		//dev_cotrans3(n, oa, n, 1);

		/* Consider this facet further only if its normal points somewhat
		 * towards the observer rather than away         */
		if (n[2] > 0.0) {
			/* Convert the three sets of vertex coordinates from body to ob-
			 * server coordinates; orbit_offset is the center-of-mass offset
			 * (in observer coordinates) for this model at this frame's epoch
			 * due to orbital motion, in case the model is half of a binary
			 * system.  */
			dev_cotrans6(v0, oa, verts[0]->v[fidx.x].x, 1, frm);
			dev_cotrans6(v1, oa, verts[0]->v[fidx.y].x, 1, frm);
			dev_cotrans6(v2, oa, verts[0]->v[fidx.z].x, 1, frm);

			for (i = 0; i <= 2; i++) {
				v0[i] += orbit_offs.x;
				v1[i] += orbit_offs.y;
				v2[i] += orbit_offs.z;
			}

			/* Find rectangular region (in POS pixels) containing the projected
			 * facet - use floats in case model has illegal parameters and the
			 * pixel numbers exceed the limits for valid integers                         */
			imin_dbl = floor(MIN(v0[0],MIN(v1[0],v2[0])) / kmpxl
							- SMALLVAL + 0.5);
			imax_dbl = floor(MAX(v0[0],MAX(v1[0],v2[0])) / kmpxl
							+ SMALLVAL + 0.5);
			jmin_dbl = floor(MIN(v0[1],MIN(v1[1],v2[1])) / kmpxl
							- SMALLVAL + 0.5);
			jmax_dbl = floor(MAX(v0[1],MAX(v1[1],v2[1])) / kmpxl
							+ SMALLVAL + 0.5);
			imin = (imin_dbl < INT_MIN) ? INT_MIN : (int) imin_dbl;
			imax = (imax_dbl > INT_MAX) ? INT_MAX : (int) imax_dbl;
			jmin = (jmin_dbl < INT_MIN) ? INT_MIN : (int) jmin_dbl;
			jmax = (jmax_dbl > INT_MAX) ? INT_MAX : (int) jmax_dbl;

			/*  Set the outbnd flag if the facet extends beyond the POS window  */
			if ((imin < (-pos[frm]->n)) || (imax > pos[frm]->n) ||
					(jmin < (-pos[frm]->n))	|| (jmax > pos[frm]->n)) {
				posvis_streams_outbnd = 1;
				outbndarr[f] = 1;
			}

			/* Figure out if facet projects at least partly within POS window;
			 * if it does, look at each "contained" POS pixel and get the
			 * z-coordinate and cos(scattering angle)           */
			i1 = MAX(imin, -pos[frm]->n);		j1 = MAX(jmin, -pos[frm]->n);
			i2 = MIN(imax,  pos[frm]->n);		j2 = MIN(jmax,  pos[frm]->n);

			if (i1 > pos[frm]->n || i2 < -pos[frm]->n || j1 > pos[frm]->n || j2 < -pos[frm]->n) {

				/* Facet is entirely outside the POS frame: just keep track of
				 * changed POS region     */
				dev_POSrect_streams(pos, src, imin_dbl, imax_dbl, jmin_dbl, jmax_dbl,
						ijminmax_overall, frm);

			} else {

				dev_POSrect_streams(pos, src, (float)i1, (float)i2, (float)j1,
						(float)j2, ijminmax_overall, frm);

				/* Facet is at least partly within POS frame: find all POS
				 * pixels whose centers project onto this facet  */
				for (i = i1; i <= i2; i++) {
					x[0] = i * kmpxl;
					for (j = j1; j <= j2; j++) {
						x[1] = j * kmpxl;

						/* Calculate the pixel address for 1D arrays */
						pxa = (j+pos[frm]->n) * (2*pos[frm]->n + 1) + (i+pos[frm]->n);

						/* Compute parameters s(x,y) and t(x,y) which define a
						 * facet's surface as
						 *         z = z0 + s*(z1-z0) + t*(z2-z1)
						 * where z0, z1, and z2 are the z-coordinates at the
						 * vertices. The conditions 0 <= s <= 1 and
						 * 0 <= t <= s require the POS pixel center to be
						 * "within" the (projected) perimeter of facet f.    */
						den = 1	/ ((v1[0] - v0[0]) * (v2[1] - v1[1])
								 - (v2[0] - v1[0]) * (v1[1] - v0[1]));
						s = ((x[0] - v0[0]) * (v2[1] - v1[1])
						  - (v2[0] - v1[0]) * (x[1] - v0[1])) * den;

						if ((s >= -SMALLVAL) && (s <= 1.0 + SMALLVAL)) {

							t = ((v1[0] - v0[0]) * (x[1] - v0[1])
							    - (x[0] - v0[0]) * (v1[1]- v0[1])) * den;
							if ((t >= -SMALLVAL) && (t <= s + SMALLVAL)) {

								/* Compute z-coordinate of pixel center: its
								 * distance measured from the origin towards
								 * Earth.    */
								z = v0[2] + s*(v1[2]-v0[2]) + t*(v2[2]-v1[2]);

								/* If fac[i][j] is >= 0, pixel [i][j] was al-
								 * ready assigned values during a previous call
								 * to posvis for a different model component.
								 * If so, override only if the current component
								 * is blocking our view of (i.e., is closer to
								 * us than) the previous one.   */

								/* Following line replaces the previous if check
								 * for z > zz[i][j]
								 * atomicMaxf returns the value that was sitting
								 * at zzf[pxa] at time of call.  So if that value
								 * matches the z we compared to*/

								if (src)
									old = atomicMaxf(&pos[frm]->zill_s[pxa], z);
								else
									old = atomicMaxf(&pos[frm]->z_s[pxa], z);

								if (old < z || pos[frm]->fill[i][j] < 0 ||
										pos[frm]->f[i][j] < 0) {

									/* Next line assigns distance of POS pixel
									 * center from COM towards Earth; that is,
									 * by changing zz,it changes pos->z or
									 * pos->zill                */
									/* following line is a first time z calc
									 * for this pixel  */
									if ( (pos[frm]->fill[i][j] < 0) || (pos[frm]->f[i][j] < 0)){
										if (src)	atomicExch(&pos[frm]->zill_s[pxa], z);
										else 		atomicExch(&pos[frm]->z_s[pxa], z);
									}

									if (smooth) {

										/* Get pvs_smoothed version of facet unit
										 * normal: Take the linear combination
										 * of the three vertex normals; trans-
										 * form from body to observer coordina-
										 * tes; and make sure that it points
										 * somewhat in our direction.         */
										for (k = 0; k <= 2; k++)
											n[k] =	verts[0]->v[fidx.x].n[k]
											 + s * (verts[0]->v[fidx.y].n[k] - verts[0]->v[fidx.x].n[k])
											 + t * (verts[0]->v[fidx.z].n[k] - verts[0]->v[fidx.y].n[k]);
										dev_cotrans6(n, oa, n, 1, frm);
										//dev_cotrans3(n, oa, n, 1);
										dev_normalize(n);
									}

									/* Determine scattering and/or incidence
									 * angles. Next lines change pos->cose/
									 * cosill. If bistatic (lightcurves), where
									 * we are viewing from Earth (src = 0),
									 * pos->cosi is also changed.                 */
									if (n[2] > 0.0) {
										if (src) atomicExch(&pos[frm]->cosill_s[pxa], n[2]);
										else	 atomicExch(&pos[frm]->cose_s[pxa], n[2]);
										if ((!src) && (pos[frm]->bistatic)) {
											float temp = (float)dev_dot2(n,usrc[frm]);
											atomicExch(&pos[frm]->cosi_s[pxa], temp);
											if (pos[frm]->cosi_s[pxa] <= 0.0)
												pos[frm]->cose_s[pxa] = 0.0;
										}
									}

									/* Next lines change pos->body/bodyill,
									 * pos->comp/compill, pos->f/fill          */
									if (src) {
										pos[frm]->bodyill[i][j] = body;
										pos[frm]->compill[i][j] = comp;
										pos[frm]->fill[i][j] = f;
									} else {
										pos[frm]->body[i][j] = body;
										pos[frm]->comp[i][j] = comp;
										pos[frm]->f[i][j] = f;
									}

								} /* end if (no other facet yet blocks this facet from view) */
							} /* end if 0 <= t <= s (facet center is "in" this POS pixel) */
						} /* end if 0 <= s <= 1 */
					} /* end j-loop over POS rows */
				} /* end i-loop over POS columns */
			} /* end else of if (i1 > pos->n || i2 < -pos->n || j1 > pos->n || j2 < -pos->n) */
		} /* End if (n[2] > 0.0) */
	} /* end if (f < nf) */
}
__global__ void posvis_facet_streams3_krnl(
		struct pos_t **pos,
		struct vertices_t **verts,
		float4 *ijminmax_overall,
		float3 orbit_offs,
		float3 *oa,
		float3 *usrc,
		int src,
		int body,
		int comp,
		int nfacets,
		int frm,
		int smooth,
		int *outbndarr) {
	/* (nf * nframes)-threaded kernel.  This version eliminates as much double
	 * math as possible */

	int f = blockIdx.x * blockDim.x + threadIdx.x;
	int pxa, i, i1, i2, j, j1, j2, imin, imax, jmin, jmax;
	float imin_dbl, imax_dbl, jmin_dbl, jmax_dbl, old, kmpxl;
	int3 fidx;

	float3 n, v0, v1, v2, tv0, tv1, tv2, x;
	float s, t, z, den;

	if (f < nfacets) {
		/* The following section transfers vertex coordinates from double[3]
		 * storage to float3		 */
		kmpxl = (float)pos[frm]->km_per_pixel;
		fidx.x = verts[0]->f[f].v[0];
		fidx.y = verts[0]->f[f].v[1];
		fidx.z = verts[0]->f[f].v[2];
		tv0.x = (float) verts[0]->v[fidx.x].x[0];
		tv0.y = (float) verts[0]->v[fidx.x].x[1];
		tv0.z = (float) verts[0]->v[fidx.x].x[2];
		tv1.x = (float) verts[0]->v[fidx.y].x[0];
		tv1.y = (float) verts[0]->v[fidx.y].x[1];
		tv1.z = (float) verts[0]->v[fidx.y].x[2];
		tv2.x = (float) verts[0]->v[fidx.z].x[0];
		tv2.y = (float) verts[0]->v[fidx.z].x[1];
		tv2.z = (float) verts[0]->v[fidx.z].x[2];
		v0.x = v0.y = v0.z = v1.x = v1.y = v1.z = v2.x = v2.y = v2.z = 0.0;

		/* Get the normal to this facet in body-fixed (asteroid) coordinates
		 * and convert it to observer coordinates     */
		n.x = verts[0]->f[f].n[0];
		n.y = verts[0]->f[f].n[1];
		n.z = verts[0]->f[f].n[2];

		dev_cotrans8(&n, oa, n, 1, frm);

		/* Consider this facet further only if its normal points somewhat
		 * towards the observer rather than away         */
		if (n.z > 0.0) {
			/* Convert the three sets of vertex coordinates from body to ob-
			 * server coordinates; orbit_offset is the center-of-mass offset
			 * (in observer coordinates) for this model at this frame's epoch
			 * due to orbital motion, in case the model is half of a binary
			 * system.  */
			dev_cotrans8(&v0, oa, tv0, 1, frm);
			dev_cotrans8(&v1, oa, tv1, 1, frm);
			dev_cotrans8(&v2, oa, tv2, 1, frm);

			v0.x += orbit_offs.x;
			v0.y += orbit_offs.x;
			v0.z += orbit_offs.x;
			v1.x += orbit_offs.y;
			v1.y += orbit_offs.y;
			v1.z += orbit_offs.y;
			v2.x += orbit_offs.z;
			v2.y += orbit_offs.z;
			v2.z += orbit_offs.z;

			/* Find rectangular region (in POS pixels) containing the projected
			 * facet - use floats in case model has illegal parameters and the
			 * pixel numbers exceed the limits for valid integers                         */
			imin_dbl = floor(MIN(v0.x,MIN(v1.x,v2.x)) / kmpxl
							- SMALLVAL + 0.5);
			imax_dbl = floor(MAX(v0.x,MAX(v1.x,v2.x)) / kmpxl
							+ SMALLVAL + 0.5);
			jmin_dbl = floor(MIN(v0.y,MIN(v1.y,v2.y)) / kmpxl
							- SMALLVAL + 0.5);
			jmax_dbl = floor(MAX(v0.y,MAX(v1.y,v2.y)) / kmpxl
							+ SMALLVAL + 0.5);
			imin = (imin_dbl < INT_MIN) ? INT_MIN : (int) imin_dbl;
			imax = (imax_dbl > INT_MAX) ? INT_MAX : (int) imax_dbl;
			jmin = (jmin_dbl < INT_MIN) ? INT_MIN : (int) jmin_dbl;
			jmax = (jmax_dbl > INT_MAX) ? INT_MAX : (int) jmax_dbl;

			/*  Set the outbnd flag if the facet extends beyond the POS window  */
			if ((imin < (-pos[frm]->n)) || (imax > pos[frm]->n) ||
					(jmin < (-pos[frm]->n))	|| (jmax > pos[frm]->n)) {
				posvis_streams_outbnd = 1;
				outbndarr[f] = 1;
			}

			/* Figure out if facet projects at least partly within POS window;
			 * if it does, look at each "contained" POS pixel and get the
			 * z-coordinate and cos(scattering angle)           */
			i1 = MAX(imin, -pos[frm]->n);		j1 = MAX(jmin, -pos[frm]->n);
			i2 = MIN(imax,  pos[frm]->n);		j2 = MIN(jmax,  pos[frm]->n);

			if (i1 > pos[frm]->n || i2 < -pos[frm]->n || j1 > pos[frm]->n || j2 < -pos[frm]->n) {

				/* Facet is entirely outside the POS frame: just keep track of
				 * changed POS region     */
				dev_POSrect_streams(pos, src, imin_dbl, imax_dbl, jmin_dbl, jmax_dbl,
						ijminmax_overall, frm);

			} else {

				dev_POSrect_streams(pos, src, (float)i1, (float)i2, (float)j1,
						(float)j2, ijminmax_overall, frm);

				/* Facet is at least partly within POS frame: find all POS
				 * pixels whose centers project onto this facet  */
				for (i = i1; i <= i2; i++) {
					x.x = i * kmpxl;
					for (j = j1; j <= j2; j++) {
						x.y = j * kmpxl;

						/* Calculate the pixel address for 1D arrays */
						pxa = (j+pos[frm]->n) * (2*pos[frm]->n + 1) + (i+pos[frm]->n);

						/* Compute parameters s(x,y) and t(x,y) which define a
						 * facet's surface as
						 *         z = z0 + s*(z1-z0) + t*(z2-z1)
						 * where z0, z1, and z2 are the z-coordinates at the
						 * vertices. The conditions 0 <= s <= 1 and
						 * 0 <= t <= s require the POS pixel center to be
						 * "within" the (projected) perimeter of facet f.    */
						den = 1	/ ((v1.x - v0.x) * (v2.y - v1.y)
								 - (v2.x - v1.x) * (v1.y - v0.y));
						s = ((x.x - v0.x) * (v2.y - v1.y)
						  - (v2.x - v1.x) * (x.y - v0.y)) * den;

						if ((s >= -SMALLVAL) && (s <= 1.0 + SMALLVAL)) {

							t = ((v1.x - v0.x) * (x.y - v0.y)
							    - (x.x- v0.x) * (v1.y- v0.y)) * den;
							if ((t >= -SMALLVAL) && (t <= s + SMALLVAL)) {

								/* Compute z-coordinate of pixel center: its
								 * distance measured from the origin towards
								 * Earth.    */
								z = v0.z + s*(v1.z-v0.z) + t*(v2.z-v1.z);

								/* If fac[i][j] is >= 0, pixel [i][j] was al-
								 * ready assigned values during a previous call
								 * to posvis for a different model component.
								 * If so, override only if the current component
								 * is blocking our view of (i.e., is closer to
								 * us than) the previous one.   */

								/* Following line replaces the previous if check
								 * for z > zz[i][j]
								 * atomicMaxf returns the value that was sitting
								 * at zzf[pxa] at time of call.  So if that value
								 * matches the z we compared to*/

								if (src)
									old = atomicMaxf(&pos[frm]->zill_s[pxa], z);
								else
									old = atomicMaxf(&pos[frm]->z_s[pxa], z);

								if (old < z || pos[frm]->fill[i][j] < 0 ||
										pos[frm]->f[i][j] < 0) {

									/* Next line assigns distance of POS pixel
									 * center from COM towards Earth; that is,
									 * by changing zz,it changes pos->z or
									 * pos->zill                */
									/* following line is a first time z calc
									 * for this pixel  */
									if ( (pos[frm]->fill[i][j] < 0) || (pos[frm]->f[i][j] < 0)){
										if (src)	atomicExch(&pos[frm]->zill_s[pxa], z);
										else 		atomicExch(&pos[frm]->z_s[pxa], z);
									}

									if (smooth) {
										/* Assign temp. normal components as float3 */
										tv0.x = (float)verts[0]->v[fidx.x].n[0];
										tv0.y = (float)verts[0]->v[fidx.x].n[1];
										tv0.z = (float)verts[0]->v[fidx.x].n[2];
										tv1.x = (float)verts[0]->v[fidx.y].n[0];
										tv1.y = (float)verts[0]->v[fidx.y].n[1];
										tv1.z = (float)verts[0]->v[fidx.y].n[2];
										tv2.x = (float)verts[0]->v[fidx.z].n[0];
										tv2.y = (float)verts[0]->v[fidx.z].n[1];
										tv2.z = (float)verts[0]->v[fidx.z].n[2];

										/* Get pvs_smoothed version of facet unit
										 * normal: Take the linear combination
										 * of the three vertex normals; trans-
										 * form from body to observer coordina-
										 * tes; and make sure that it points
										 * somewhat in our direction.         */

										n.x = tv0.x + s * (tv1.x - tv0.x) + t * (tv2.x - tv1.x);
										n.y = tv0.y + s * (tv1.y - tv0.y) + t * (tv2.y - tv1.y);
										n.z = tv0.z + s * (tv1.z - tv0.z) + t * (tv2.z - tv1.z);

										dev_cotrans8(&n, oa, n, 1, frm);
										dev_normalize2(n);
									}

									/* Determine scattering and/or incidence
									 * angles. Next lines change pos->cose/
									 * cosill. If bistatic (lightcurves), where
									 * we are viewing from Earth (src = 0),
									 * pos->cosi is also changed.                 */
									if (n.z > 0.0) {
										if (src) atomicExch(&pos[frm]->cosill_s[pxa], n.z);
										else	 atomicExch(&pos[frm]->cose_s[pxa], n.z);
										if ((!src) && (pos[frm]->bistatic)) {
											float temp = (float)dev_dot4(n,usrc[frm]);
											atomicExch(&pos[frm]->cosi_s[pxa], temp);
											if (pos[frm]->cosi_s[pxa] <= 0.0)
												pos[frm]->cose_s[pxa] = 0.0;
										}
									}

									/* Next lines change pos->body/bodyill,
									 * pos->comp/compill, pos->f/fill          */
									if (src) {
										pos[frm]->bodyill[i][j] = body;
										pos[frm]->compill[i][j] = comp;
										pos[frm]->fill[i][j] = f;
									} else {
										pos[frm]->body[i][j] = body;
										pos[frm]->comp[i][j] = comp;
										pos[frm]->f[i][j] = f;
									}

								} /* end if (no other facet yet blocks this facet from view) */
							} /* end if 0 <= t <= s (facet center is "in" this POS pixel) */
						} /* end if 0 <= s <= 1 */
					} /* end j-loop over POS rows */
				} /* end i-loop over POS columns */
			} /* end else of if (i1 > pos->n || i2 < -pos->n || j1 > pos->n || j2 < -pos->n) */
		} /* End if (n[2] > 0.0) */
	} /* end if (f < nf) */
}
__global__ void posvis_streams_outbnd_krnl(struct pos_t **pos, int posn,
		int *outbndarr, float4 *ijminmax_overall, int f) {
	/* Single-threaded, streamed kernel */
	double xfactor, yfactor;
	if (threadIdx.x == 0) {
		if (outbndarr[f]) {
			/* ijminmax_overall.w = imin_overall
			 * ijminmax_overall.x = imax_overall
			 * ijminmax_overall.y = jmin_overall
			 * ijminmax_overall.z = jmax_overall	 */
			xfactor = (MAX( ijminmax_overall[f].x,  posn) -
					MIN( ijminmax_overall[f].w, -posn) + 1) / (2*posn+1);
			yfactor = (MAX( ijminmax_overall[f].z,  posn) -
					MIN( ijminmax_overall[f].y, -posn) + 1) / (2*posn+1);
			pos[f]->posbnd_logfactor = log(xfactor*yfactor);
		}
	}
}

__host__ int posvis_cuda_streams2(
		struct par_t *dpar,
		struct mod_t *dmod,
		struct dat_t *ddat,
		struct pos_t **pos,
		struct vertices_t **verts,
		float3 orbit_offset,
		int *posn,
		int *outbndarr,
		int set,
		int nframes,
		int src,
		int nf,
		int body, int comp, unsigned char type, hipStream_t *posvis_stream) {

	int outbnd, smooth, start, end, frames_alloc, i=0, f;
	dim3 BLK,THD;
	hipEvent_t start1, stop1;
	float milliseconds;
	float4 *ijminmax_overall;
	float3 *oa, *usrc;

	/* Launch parameters for the facet_streams kernel */
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + nf) / THD.x);

	/* Set up the offset addressing for lightcurves if this is a lightcurve */
	if (type == LGHTCRV) {
		start = 1;	/* fixes the lightcurve offsets */
		end = nframes + 1;
		frames_alloc = nframes + 1;
	} else {
		start = 0;
		end = nframes;
		frames_alloc = nframes;
	}
	float4 hijmm[frames_alloc];
	int oasize = frames_alloc*3;
	/* Allocate temporary arrays/structs */
	gpuErrchk(hipMalloc((void**)&ijminmax_overall, sizeof(float4) * frames_alloc));
	gpuErrchk(hipMalloc((void**)&oa, sizeof(float3) * oasize));
	gpuErrchk(hipMalloc((void**)&usrc, sizeof(float3) * frames_alloc));

	if (TIMING) {
		/* Create the timer events */
		hipEventCreate(&start1);
		hipEventCreate(&stop1);
		hipEventRecord(start1);
	}

	/* Debug stuff */
	int debug = 0;
	int size;
	/* End debug stuff */

	for (int f=start; f<end; f++) {

		/* Initialize via single-thread kernel first */
		posvis_streams_init_krnl<<<1,1,0,posvis_stream[f-start]>>>(dpar,
				pos, ijminmax_overall, oa, usrc, outbndarr, comp, f, start,
				end, src);

//		hipDeviceSynchronize();

		/* Now the main facet kernel */
		posvis_facet_streams3_krnl<<<BLK,THD, 0, posvis_stream[f-start]>>>(pos, verts,
				ijminmax_overall, orbit_offset, oa, usrc,	src, body, comp,
				nf, f, smooth, outbndarr);

//		hipDeviceSynchronize();
//		size = (2*posn[1]+1)*(2*posn[1]+1);
//		if (debug)
//			dbg_print_lghtcrv_pos_arrays(ddat, set, 1, size, posn[1]);

		/* Take care of any posbnd flags */
		posvis_streams_outbnd_krnl<<<1,1,0,posvis_stream[f-start]>>>(pos, posn[f],
				outbndarr, ijminmax_overall, f);

//		hipDeviceSynchronize();

	}
//	hipMemcpy(&hijmm, ijminmax_overall, sizeof(float4)*frames_alloc, hipMemcpyDeviceToHost);

	if (TIMING) {
		hipEventRecord(stop1);
		hipEventSynchronize(stop1);
		milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start1, stop1);
		printf("%i facets in posvis_cuda_2 in %3.3f ms with %i frames.\n", nf, milliseconds, nframes);
	}
	checkErrorAfterKernelLaunch("The three posvis_cuda_streams2 kernels");
	gpuErrchk(hipMemcpyFromSymbol(&outbnd, HIP_SYMBOL(posvis_streams_outbnd), sizeof(outbnd), 0,
			hipMemcpyDeviceToHost));

	//	for (int f=0; f<nframes; f++)
//		hipStreamSynchronize(posvis_stream[f]);
//dbg_print_pos_z(ddat, 0, 1, posn[1], "streams_pos_z_s0f0.csv");
//	dbg_print_lghtcrv_pos_arrays(ddat, s, 1, nThreadspx[1], hposn[1]);
	/* Free temp arrays, destroy streams and timers, as applicable */


	hipFree(ijminmax_overall);
	hipFree(oa);
	hipFree(usrc);


	if (TIMING) {
		hipEventDestroy(start1);
		hipEventDestroy(stop1);
	}
	return outbnd;
}

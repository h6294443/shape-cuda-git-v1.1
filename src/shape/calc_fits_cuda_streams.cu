#include "hip/hip_runtime.h"
/*****************************************************************************************
                                                                              calc_fits.c

As the name implies, this routine calculates the fits to each data frame for the current
set of model parameters.  For example, for each delay-Doppler frame it calls routine
posvis to create the model plane-of-sky image and then routine pos2deldop to create the
model delay-Doppler image from this POS image.

calc_fits also performs some of the screen and file output required by the "write" action;
in particular, it carries out tasks that require information associated with plane-of-sky
renderings, since such information is quickly overwritten if the "pos_scope" parameter is
set to "global" (i.e., if all frames and lightcurve points share the same memory for their
"pos" structures).

Modified 2015 June 10 by CM:
    Implement smearing for the "fit" and "write" actions

Modified 2014 February 14 by CM:
    Add "ilaw" argument to the apply_photo routine

Modified 2013 July 28 by CM:
    For the "write" action, output ppm POS images when the "write_highlight" parameter is
        turned on

Modified 2013 July 7 by CM:
    For the "write" action for lightcurve points and plane-of-sky frames, display the
        body-fixed longitude and latitude of the phase-angle bisector

Modified 2013 June 25 by CM:
    Allow POS images written for optical data to be annotated with principal-axis shafts
        and the angular momentum vector
    For POS images (sky renderings), display the name of the image file and the maximum
        pixel value in the plot_surface routine (called by the write_pos routine) rather
        than here

Modified 2013 April 24 by CM:
    Implement the "listpos_deldop" "listpos_opt" and "listpos_path" parameters
    Adjust names of output images so they are in alphanumeric order if > 100 per dataset

Modified 2012 April 2 by CM:
    Correct instantaneous maximum breadth calculation for Doppler scaling factor

Modified 2011 August 14 by CM:
    Display sidereal spin vector at each epoch, even for a PA rotator, if
        any spin impulses are used

Modified 2010 September 1 by CM:
    Initialize variables to avoid compilation warnings

Modified 2010 July 29 by CM:
    Fix bug introduced in calc_lghtcrv: rotation phases weren't being
        displayed for the "write" action
    For the "write" action for lightcurve datasets, include shadowed
        regions in projected area (and geometric albedo calculation)
        and display percentage of projected area that's shadowed

Modified 2010 June 15 by CM:
    Revise arguments to pos2deldop and pos2doppler routines

Modified 2010 May 28 by CM:
    Fix bug introduced with preceding change: in calc_lghtcrv, only
        deallocate memory for the "write" action (since it wasn't
        allocated in the first place for other actions)

Modified 2010 May 24 by CM:
    For the "write" action for lightcurves, output the projected area and
        (for absolute photometry) geometric albedo

Modified 2010 April 12 by CM:
    For the "write" action, include overflow region when computing
        cross sections

Modified 2009 July 29 by CM:
    For the "write" action, fix bug: output ppm images rather than pgm
        images if the "plot_angmom" parameter is turned on
    For the "write" action, pass an argument to the "write_pos" routine
        explicitly telling it whether or not to produce a colored image

Modified 2009 April 3 by CM:
    Initialize the "posbnd_logfactor" parameter and later set it for
        models that extend beyond the POS frame
    Add "badposet" and "badposet_logfactor" parameters: initialize them
        here and then use the new "checkposet" routine to adjust them for
        plane-of-sky fit images that are too small to "contain" the
        target
    Add "badradar" and "badradar_logfactor" parameters: initialize them
        here and then use the "pos2deldop" and "pos2doppler" routines
        (which are now int rather than void) to adjust them for models that
        are too wide in delay-Doppler space for the routines to handle
    Add "warn_badradar" argument to pos2deldop and pos2doppler routines
    For the "write" action, display each plane-of-sky fit frame's linear
        dimensions, the linear dimensions of the rectangular subset that
        contains the target, and the linear COM offsets

Modified 2008 December 12 by CM:
    For the "write" action for NPA rotators, list Euler angles (giving
        the body-fixed axes' orientations in ecliptic coordinates) and
        spin vector components (in body-fixed coordinates) for each
        observation epoch
    For the "write" action for NPA rotators, ensure that maximum breadth
        is nonnegative

Modified 2007 August 10 by CM:
    Eliminated unused variables and cleaned up a printf format
    For POS model frames (sky renderings) associated with lightcurve points
        and with plane-of-sky data frames, don't display the maximum pixel
        value unless the "optposmax" parameter is nonzero

Modified 2007 August 4 by CM:
    Add comp matrix for POS frames
    Add orbit_offset and body arguments to posvis routine and remove
        facet argument
    Add orbit_xoff, orbit_yoff, orbit_dopoff and body parameters to
        pos2deldop and pos2doppler routines
    Add body argument to apply_photo routine

Modified 2007 January 17 by CM:
    For the "write" action, display instantaneous folded zero-crossing
        bandwidth for Doppler and delay-Doppler frames

Modified 2007 January 11 by CM:
    In calc_lghtcrv for the "write" action, count lightcurve points
        from 0 rather than 1, as is already done for lightcurve POS images
        (and for Doppler, delay-Doppler, and plane-of-sky frames)

Modified 2007 January 6 by CM:
    In calc_lghtcrv for the "write" action, save rotation phase for each
        calculated lightcurve point so they can be output by routine chi2,
        and use cubic spline interpolation to obtain rotation phase at
        each observation epoch.  Also display range of rotation phases
        if only one calculated point per lightcurve is displayed in full

Modified 2006 October 1 by CM:
    In calc_lghtcrv, model lightcurve points are now intensities
        (relative to the solar intensity) rather than magnitudes
    In calc_lghtcrv and calc_poset, apply_photo routine has been revised
        to account for the POS pixel area and the 1 AU Sun-target distance

Modified 2006 September 1 by CM and MCN:
    When "exclude_seen" parameter is used, add check that facet number
        pos->f[i][j] is nonnegative
    For the "write" action, don't display cross sections and albedos
        for uncalibrated (delay-)Doppler frames

Modified 2006 June 21 by CM:
    In calc_deldop, changed delres to del_per_pixel and dopres to
        dop_per_pixel
    In calc_doppler, changed dopres to dop_per_bin
    For POS renderings and plane-of-sky fit frames, changed res to
        km_per_pixel

Modified 2006 June 18 by CM:
    Allow each delay-Doppler frame within a dataset to have different
        dimensions after vignetting
    Allow each Doppler frame within a dataset to have different
        dimensions after vignetting
    Allow plane-of-sky frames to be rectangular rather than square,
        and no longer require an odd number of pixels per side
    Eliminate range datasets

Modified 2006 March 10 by CM:
    Add "speckle" argument to pos2deldop and pos2doppler routines

Modified 2005 October 6 by CM:
    For lightcurve datasets, replace SUNMAG constant by "sun_appmag"
        parameter, so that absolute photometry with filters other than
        V band can be used

Modified 2005 July 25 by CM:
    For "write" action, display the model radar cross section and albedo
        for each delay-Doppler and Doppler frame

Modified 2005 July 22 by CM:
    Created five separate routines for writing POS frames as images
        so that they can be called separately if the "mark_unseen"
        parameter is turned on for the "write" action (since in this
        case we must first process all datasets to see which model
        facets were "seen" and only then can write the POS images)

Modified 2005 July 14 by CM:
    Fix bug in computing LE-to-COM delay and distance, LE-to-TE
        delay and distance, and instantantaneous bandwidth and breadth

Modified 2005 July 13 by CM:
    For "write" action for lightcurve points and plane-of-sky frames,
        display the body-fixed longitude and latitude of the
        Sun-to-asteroid line

Modified 2005 July 5 by CM:
    Remove the "dir" argument from pos2deldop and pos2doppler and add
        the "set" argument

Modified 2005 July 3 by CM:
    For "write" action for lightcurve datasets, implement the
        "lcrv_writeall" parameter, which produces screen display for
        every model lightcurve point rather than just the one point
        which falls closest to the midpoint of the observations.

Modified 2005 June 25 by CM:
    For "write" action for delay-Doppler frames, display the delay and
        distance between the leading edge and the center of mass and
        between the leading edge and the trailing edge;
        for delay-Doppler and Doppler frames, display the instantaneous
        zero-crossing bandwidth and maximum breadth.  All of the above
        are obtained from the model's delay-Doppler limits as
        determined PRIOR to convolution with the delay and Doppler
        response functions.

Modified 2005 June 22 by CM:
    Keep track of which model facets have been "seen" (i.e., are visible
        from Earth, are unshadowed, and have sufficiently low scattering
        and incidence angles) in at least one data frame or lightcurve
        point

Modified 2005 April 23 by CM:
    For the "write" action, list whether or not epochs have been corrected
        for one-way light travel time

Modified 2005 March 1 by CM:
    Adjust arguments to the revised "resampim" routine to permit rotation
        of resampled plane-of-sky frames
    Initialize the "posbnd" parameter (flag indicating that the model
        extends beyond the model POS frame) to 0 here rather than in
        bestfit.c so that it can used for actions other than "fit"
    Fix bug in calc_poset which was incorrectly flagging the model as
        being too small for the model POS frame

Modified 2005 February 21 by CM:
    Use the new "poset_resample" parameter to allow interpolation methods
        other than bilinear for constructing plane-of-sky fit images for
        plane-of-sky data frames
    Add the new "image_rebin" argument to function resampim to handle
        plane-of-sky fit frames which have much coarser resolution
        than the model POS frames from which they are constructed
        (i.e., which are greatly undersampled)
    For "write" action, display maximum pixel value for model POS images
        for plane-of-sky frames and calculated lightcurve images
        (in case someone wants to use the "optposmax" parameter to
        truncate the image brightness)

Modified 2005 February 6 by CM:
    For "write" action, display rotation phase
    For "write" action, fix bug in computing the angular body-fixed
        coordinates of the line of sight for lightcurve datasets

Modified 2005 January 25 by CM:
    Take care of unused and uninitialized variables

Modified 2005 January 24 by CM:
    Add "calc_poset" routine to handle POS datasets
    For "write" action, display the angular body-fixed coordinates of
        the line of sight
    For "write" action, display calendar dates in addition to Julian dates
    For "write" action, display the date for range datasets

Modified 2004 December 19 by CM:
    For "write" action, display the projected area for each Doppler and
        delay-Doppler frame

Modified 2004 May 3 by CM:
    For "write" action, display the (delay-)Doppler corrections for each
        frame

Modified 2004 April 9 by CM:
    For "write" action, display the solar azimuth angles (N->E in the POS)

Modified 2004 March 27 by CM:
    Eliminate output of range (rng) plane-of-sky images for
        delay-Doppler frames
    For "write" action, display the epoch, solar phase angle and
        apparent spin vector direction at the midpoint of lightcurve
        datasets
    For "write" action, if "plot_spinvec" parameter is turned on, 
        POS pgm images include an arrow indicating the target's
        intrinsic spin vector.
    For "write" action, if "plot_subradar" parameter is turned on, 
        POS pgm images for (delay-)Doppler datasets include an X
        indicating the target's subradar point.
    For "write" action, if "plot_com" parameter is turned on, 
        POS pgm images for (delay-)Doppler datasets include a cross
        indicating the target's projected COM.
    For "write" action, if "plot_pa" parameter vector has any
        component(s) turned on, POS ppm images for (delay-)Doppler
        datasets include colored cylindrical shaft(s) indicating the
        positive end of the corresponding principal axis/axes.

Modified 2004 Feb 29 by CM:
    Add comments for lightcurves
    Remove "sdev" argument to routine gamma_trans
    Compute lightcurve magnitudes rather than negative magnitudes
    Eliminate the "curve_mm" lightcurve output file, since it nearly
        duplicates the "fit.mm" file (except that the cal factor
        isn't included)
    Move the lightcurve calculations to the new "calc_lghtcrv" routine
    Eliminate the unused dat argument to calc_deldop, calc_doppler,
        and calc_range
    Eliminate "type" argument to the "apply_photo" routine, and
        add the "phase" (solar phase angle) argument
    Label lightcurve POS images as 0 through (ncalc-1) rather than
        1 through ncalc, similar to (delay-)Doppler pgm images

Modified 2003 July 30 by CM:
    Add three parameters for rotating/flipping output pgm files
        for delay-Doppler images (fit, data, residuals)

Modified 2003 May 16 by CM:
    Add listres parameter for producing output files containing
        residual matrices

Modified 2003 May 13 by CM:
    Don't resample and recenter residual pgm images if dd_scaling = none
    Correct a bug in normalizing file output for Doppler fits

Modified 2003 May 10 by CM:
    Add scalefitobs parameter so that user can choose whether to scale
        the data and fit pgm images separately (default), to the maximum
        value of the two taken together, to the maximum fit value, or to
        the maximum data value

Modified 2003 May 7 by CM:
    Add sinc2width argument to pos2deldop and pos2doppler

Modified 2003 April 29 by CM:
    Don't truncate residuals to integer values before making pgm images
    Add nsinc2 argument to pos2deldop and pos2doppler

Modified 2003 April 28 by CM:
    Display two angles for the spin vector, not just one

Modified 2003 April 24 by CM:
    Move "delcom" from delay-Doppler datasets to individual frames

Modified 2003 April 23 by CM:
    Removed "deldopoffs" call from calc_deldop and "dopoffs" call from
        calc_deldop, since these calls are now included in realize_delcor
 *****************************************************************************************/
extern "C" {
#include "head.h"
}

__host__ void calc_deldop_cuda_streams(struct par_t *dpar, struct mod_t *dmod,
		struct dat_t *ddat, int s, int nframes, int nviews, unsigned char type,
		hipStream_t *cf_stream);
//__host__ void calc_doppler_cuda(struct par_t *dpar, struct mod_t *dmod,
//		struct dat_t *ddat, int s);
////__host__ void calc_poset_cuda( struct par_t *par, struct mod_t *mod, int s);
//__host__ void calc_lghtcrv_cuda(struct par_t *dpar, struct mod_t *dmod,
//		struct dat_t *ddat, int s);

__device__ int cfs_nf, cfs_nsets, cfs_v0_index, cfs_exclude_seen;
__device__ __managed__ double cfs_lghtcrv_posbnd_logfactor;
__device__ struct deldop_t *cfs_deldop;
__device__ struct doppler_t *cfs_doppler;
__device__ struct lghtcrv_t *cfs_lghtcrv;

__device__ void dev_spline(double *x,double *y,int n,double yp1,double ypn,double *y2, double *u)
{
	int i,k;
	double p,qn,sig,un;

	if (yp1 > 0.99e30)
		y2[1]=u[1]=0.0;
	else {
		y2[1] = -0.5;
		u[1] = (3.0 / (x[2]-x[1])) * ((y[2]-y[1]) / (x[2]-x[1])-yp1);
	}

	for (i=2;i<=n-1;i++) {
		sig = (x[i]-x[i-1]) / (x[i+1]-x[i-1]);
		p = sig * y2[i-1] + 2.0;
		y2[i] = (sig-1.0) / p;
		u[i] = (y[i+1]-y[i]) / (x[i+1]-x[i]) - (y[i]-y[i-1]) / (x[i]-x[i-1]);
		u[i] = (6.0 * u[i]/(x[i+1]-x[i-1]) - sig * u[i-1]) / p;
	}

	if (ypn > 0.99e30)
		qn=un=0.0;
	else {
		qn=0.5;
		un=(3.0/(x[n]-x[n-1]))*(ypn-(y[n]-y[n-1])/(x[n]-x[n-1]));
	}
	y2[n]=(un-qn*u[n-1])/(qn*y2[n-1]+1.0);

	for (k=n-1;k>=1;k--)
		y2[k]=y2[k]*y2[k+1]+u[k];

}
__device__ void dev_splint(double *xa,double *ya,double *y2a,int n,double x,double *y)
{
	int klo,khi,k;
	double h,b,a;

	klo = 1;
	khi = n;
	while (khi-klo > 1) {
		k = (khi+klo) >> 1;
		if (xa[k] > x) 	khi=k;
		else klo = k;
	}
	h = xa[khi] - xa[klo];
	if (h == 0.0) 	printf("Bad XA input to routine SPLINT");
	a = (xa[khi] - x) / h;
	b = (x - xa[klo]) / h;
	*y = a * ya[klo] + b * ya[khi] + ((a*a*a-a) * y2a[klo] + (b*b*b-b) *
			y2a[khi]) * (h*h)/6.0;
}
__global__ void cfs_init_devpar_krnl(struct par_t *dpar, struct mod_t
		*dmod, struct dat_t *ddat) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		dpar->posbnd = 0;
		dpar->badposet = 0;
		dpar->badradar = 0;
		dpar->posbnd_logfactor = 0.0;
		dpar->badposet_logfactor = 0.0;
		dpar->badradar_logfactor = 0.0;
		cfs_nf = dmod->shape.comp[0].real.nf;
		cfs_nsets = ddat->nsets;
	}
}
__global__ void cfs_get_set_type_krnl(struct dat_t *ddat, int nsets,
		unsigned char *type, int *nframes, int *nviews, int *lc_n) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		for (int s=0; s<nsets; s++) {
			type[s] = ddat->set[s].type;
			switch (type[s]) {
			case DELAY:
				nframes[s] = ddat->set[s].desc.deldop.nframes;
				nviews[s] = ddat->set[s].desc.deldop.nviews;
				lc_n[s] = 0;
				break;
			case DOPPLER:
				nframes[s] = ddat->set[s].desc.doppler.nframes;
				nviews[s] = ddat->set[s].desc.doppler.nviews;
				lc_n[s] = 0;
				break;
			case POS:
				nframes[s] = ddat->set[s].desc.poset.nframes;
				nviews[s] = ddat->set[s].desc.poset.nviews;
				lc_n[s] = 0;
				break;
			case LGHTCRV:
				nframes[s] = ddat->set[s].desc.lghtcrv.ncalc;
				nviews[s] = ddat->set[s].desc.lghtcrv.nviews;
				lc_n[s] = ddat->set[s].desc.lghtcrv.n;
				break;
			}
		}
	}
}

__host__ void calc_fits_cuda(struct par_t *dpar, struct mod_t *dmod,
		struct dat_t *ddat)
{
	int s, nf, nsets, f, *nframes, *nviews, *lc_n;
	unsigned char *type;
	dim3 BLK,THD;
	THD.x = maxThreadsPerBlock;

	/* Initialize flags that indicate the model extends beyond POS frame, that
	 * plane-of-sky fit images are too small to "contain" the target, and that
	 * model is too wide in (delay-)Doppler space to create (delay-)Doppler fit
	 * frames.  Note that this also gets mod->shape.nf and nsets            */

	cfs_init_devpar_krnl<<<1,1>>>(dpar, dmod, ddat);
	checkErrorAfterKernelLaunch("cfs_init_devpar_krnl");
	gpuErrchk(hipMemcpyFromSymbol(&nf, HIP_SYMBOL(cfs_nf), sizeof(int),
				0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&nsets, HIP_SYMBOL(cfs_nsets), sizeof(int),
					0, hipMemcpyDeviceToHost));

	/* Allocate temporary constructs (host and device) */
	unsigned char htype[nsets];
	int hnframes[nsets], hnviews[nsets], n[nsets];
	gpuErrchk(hipMalloc((void**)&type, sizeof(unsigned char)*(nsets+1)));
	gpuErrchk(hipMalloc((void**)&nframes, sizeof(int)*(nsets+1)));
	gpuErrchk(hipMalloc((void**)&lc_n, sizeof(int)*(nsets+1)));
	gpuErrchk(hipMalloc((void**)&nviews, sizeof(int)*(nsets+1))); // +1 is a safety margin

	/* Initialize the flags that indicate whether or not each facet of each
	 * model component is ever visible and unshadowed from Earth
	 * Note:  Single component only for now.  */
	//for (c=0; c<mod->shape.ncomp; c++)
	BLK.x = floor((THD.x - 1 + nf)/THD.x);
	cf_init_seen_flags_krnl<<<BLK,THD>>>(dmod,nf);
	checkErrorAfterKernelLaunch("cf_init_seen_flags_krnl (calc_fits_cuda_streams)");

	/* Get type, nframes, nviews */
	cfs_get_set_type_krnl<<<1,1>>>(ddat, nsets, type, nframes, nviews, lc_n);
	checkErrorAfterKernelLaunch("cf_init_seen_flags_krnl (calc_fits_cuda)");
	gpuErrchk(hipMemcpy(&htype, type, sizeof(unsigned char)*nsets,
			hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&hnframes, nframes, sizeof(int)*nsets,
			hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&hnviews, nviews, sizeof(int)*nsets,
			hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&n, lc_n, sizeof(int)*nsets,
			hipMemcpyDeviceToHost));

	/* Calculate the fits for each dataset in turn  */
	for (s=0; s<nsets; s++) {

		/* Now create the streams we need */
		hipStream_t cf_stream[hnframes[s]];
		for (f=0; f<hnframes[s]; f++)
			hipStreamCreate(&cf_stream[f]);

		switch (type[s]) {
		case DELAY:
			calc_deldop_cuda_streams(dpar, dmod, ddat, s, hnframes[s],
					hnviews[s], type[f], cf_stream );
			break;
		case DOPPLER:
//			calc_doppler_cuda_streams(dpar, dmod, ddat, s, hnframes[s],
//					hnviews[s], type[f], cf_stream );
			break;
		case POS:
			printf("Write calc_poset_cuda!");
//			calc_poset_cuda(dpar, dmod, s);
			break;
		case LGHTCRV:
//			calc_lghtcrv_cuda_streams(dpar, dmod, ddat, s, hnframes[s],
//					hnviews[s], n, lc_n, type[f], cf_stream );
			break;
		default:
			printf("calc_fits_cuda.c: can't handle this type yet\n");
		}
		/* Lastly, destroy streams */
		for (f=0; f<nframes[s]; f++)
			hipStreamDestroy(cf_stream[f]);
	}

	/* Complete calculations of values that will be used during a fit to
	 * increase the objective function for models with bad properties   */
	cf_set_final_pars_krnl<<<1,1>>>(dpar, ddat);
	checkErrorAfterKernelLaunch("cf_set_final_pars_krnl (calc_fits_cuda)");

	/* Free temporary memory  */
	free(htype);
	free(hnframes);
	free(hnviews);
	hipFree(type);
	hipFree(nframes);
	hipFree(nviews);
}
__global__ void cfs_set_deldop_shortcuts_krnl(struct dat_t *ddat,
		struct deldopfrm_t **frame, struct pos_t **pos,
		struct deldopview_t **view0, int *ndop, int *ndel, float *overflow,
		int *posn, int s, int f) {
	/* Single-threaded kernel */
	/* 	overflow[0] - overflow_o2_store
	 * 	overflow[1] - overflow_m2_store
	 * 	overflow[2] - overflow_xsec_store
	 * 	overflow[3] - overflow_dopmean_store
	 * 	overflow[4] - overflow_delmean_store
	 */
	if (threadIdx.x == 0) {
		if (f == 0)
			cfs_deldop 	 = &ddat->set[s].desc.deldop;

		frame[f] 	  = &cfs_deldop->frame[f];
		ndop[f]  	  = frame[f]->ndop;
		ndel[f]	 	  = frame[f]->ndel;
		view0[f] 	  = &frame[f]->view[cfs_deldop->v0];
		cfs_v0_index  = cfs_deldop->v0;
		pos[f]		  = &frame[f]->pos;
		posn[f]		  = pos[f]->n;
		overflow[0] = 0.0;
		overflow[1] = 0.0;
		overflow[2] = 0.0;
		overflow[3] = 0.0;
		overflow[4] = 0.0;

	}
}
__global__ void cfs_set_doppler_shortcuts_krnl(struct dat_t *ddat,
		struct dopfrm_t **frame, struct pos_t **pos, struct dopview_t **view0,
		float *overflow, int *ndop, int *posn, int s, int f) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		if (f == 0)
			cfs_doppler = &ddat->set[s].desc.doppler;

		frame[f] = &cfs_doppler->frame[f];
		view0[f] = &frame[f]->view[cfs_doppler->v0];
		cfs_v0_index  = cfs_doppler->v0;
		ndop[f]	 = frame[f]->ndop;
		pos[f]		 = &frame[f]->pos;
		posn[f]	= pos[f]->n;
		overflow[0] = 0.0;
		overflow[1] = 0.0;
		overflow[2] = 0.0;
		overflow[3] = 0.0;
		overflow[4] = 0.0;
	}
}
__global__ void cfs_set_lghtcrv_shortcuts_krnl(struct dat_t *ddat,
		struct crvrend_t **rend, struct pos_t **pos, double *kmppxl,
		float *overflow, int *posn, int s, int f) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		if (f == 0)
			cfs_lghtcrv = &ddat->set[s].desc.lghtcrv;
		rend[f] = &cfs_lghtcrv->rend[f];

		pos[f] = &rend[f]->pos;
		kmppxl[f] = pos[f]->km_per_pixel;
		posn[f] = pos[f]->n;

		overflow[0] = 0.0;
		overflow[1] = 0.0;
		overflow[2] = 0.0;
		overflow[3] = 0.0;
		overflow[4] = 0.0;
	}
}
__global__ void cf_set_lghtcrv_shortcuts_krnl(struct dat_t *ddat,
		struct pos_t *pos, int s, int f) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {

		cf_lghtcrv = &ddat->set[s].desc.lghtcrv;
		cf_rend = &cf_lghtcrv->rend[f];
		cf_pos = &cf_rend->pos;	/* Backup use, delete later if warranted */
		pos = &cf_rend->pos;
		cf_n = cf_lghtcrv->n;
		cf_km_p_pxl = pos->km_per_pixel;

		cf_overflow_o2_store = 0.0;
		cf_overflow_m2_store = 0.0;
		cf_overflow_xsec_store = 0.0;
		cf_overflow_delmean_store = 0.0;
		cf_overflow_dopmean_store = 0.0;
	}
}
__global__ void cf_spline_lghtcrv_krnl(double yp1, double ypn, double *u) {
	/* ncalc-threaded kernel */
	int k, i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	double p, qn, sig, un;
	int n = cf_ncalc;

	/* Single-threaded task first */
	if (i == 1) {
		if (yp1 > 0.99e30)
			cf_lghtcrv->y2[1] = u[1] = 0.0;
		else {
			cf_lghtcrv->y2[1] = -0.5;
			u[1] = (3.0 / (cf_lghtcrv->x[2] - cf_lghtcrv->x[1])) *
				   ((cf_lghtcrv->y[2] - cf_lghtcrv->y[1]) /
				    (cf_lghtcrv->x[2] - cf_lghtcrv->x[1]) - yp1);
		}
	}
	__syncthreads();

	if ((i > 1) && (i <= (n-1))) {
		sig = (cf_lghtcrv->x[i]   - cf_lghtcrv->x[i-1]) /
			  (cf_lghtcrv->x[i+1] - cf_lghtcrv->x[i-1]);

		p = sig * cf_lghtcrv->y2[i-1] + 2.0;

		cf_lghtcrv->y2[i] = (sig - 1.0) / p;

		u[i] = (cf_lghtcrv->y[i+1] - cf_lghtcrv->y[i]) / (cf_lghtcrv->x[i+1] -
				cf_lghtcrv->x[i]) - (cf_lghtcrv->y[i]  -  cf_lghtcrv->y[i-1]) /
		   	   (cf_lghtcrv->x[i]  -  cf_lghtcrv->x[i-1]);

		u[i] = (6.0 *u[i] / (cf_lghtcrv->x[i+1] - cf_lghtcrv->x[i-1]) -
				sig * u[i-1]) / p;
	}
	__syncthreads();

	/* Another single-threaded task */
	if (i == 1) {
		if (ypn > 0.99e30)
			qn = un = 0.0;
		else {
			qn = 0.5;
			un = (3.0 / (cf_lghtcrv->x[n] - cf_lghtcrv->x[n-1])) * (ypn -
						(cf_lghtcrv->y[n] - cf_lghtcrv->y[n-1]) /
						(cf_lghtcrv->x[n] - cf_lghtcrv->x[n-1]));
		}
		cf_lghtcrv->y2[n]=(un - qn * u[n-1]) /
				(qn * cf_lghtcrv->y2[n-1] + 1.0);

		for (k=n-1; k>=1; k--)
			cf_lghtcrv->y2[k] = cf_lghtcrv->y2[k] * cf_lghtcrv->y2[k+1] + u[k];
	}
	__syncthreads();
}
__global__ void cf_spline_lghtcrv_serial_krnl(double *u) {
	/* single-threaded kernel */
	if (threadIdx.x == 0)
		dev_spline( cf_lghtcrv->x, cf_lghtcrv->y, cf_ncalc, 2.0e30, 2.0e30, cf_lghtcrv->y2, u);

}
__global__ void cf_splint_lghtcrv_krnl(struct par_t *dpar) {
	/* ncalc-threaded kernel */
	int v, i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	double interp;

	if ((i >= 1) && (i <= cf_lghtcrv->n)) {

		cf_lghtcrv->fit[i] = 0.0;

		for (v=0; v<cf_lghtcrv->nviews; v++) {
			dev_splint(cf_lghtcrv->x, cf_lghtcrv->y, cf_lghtcrv->y2, cf_ncalc,
					cf_lghtcrv->t[i][v], &interp);
			cf_lghtcrv->fit[i] += interp;
		}
		cf_lghtcrv->fit[i] /= cf_lghtcrv->nviews;
	}
	__syncthreads();

	/* Single-threaded task: */
	if (i == 1) {
		/* Deal with flags for model that extends beyond the POS frame  */
		dpar->posbnd_logfactor += cf_lghtcrv->dof *
				(lghtcrv_posbnd_logfactor/cf_ncalc);
	}
}
__global__ void cfs_set_pos_ae_streams_krnl(struct pos_t **pos, int f,
		int *bistatic, unsigned char type, int v) {
	/* 9-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % 3;
	int j = offset / 3;

	if (offset < 9) {
		switch (type) {
		case DELAY:
			pos[f]->ae[i][j] = cfs_deldop->frame[f].view[v].ae[i][j];
			pos[f]->oe[i][j] = cfs_deldop->frame[f].view[v].oe[i][j];
			break;
		case DOPPLER:
			pos[f]->ae[i][j] = cfs_doppler->frame[f].view[v].ae[i][j];
			pos[f]->oe[i][j] = cfs_doppler->frame[f].view[v].oe[i][j];
			break;
		case LGHTCRV:
			pos[f]->ae[i][j] = cfs_lghtcrv->rend[f].ae[i][j];
			pos[f]->oe[i][j] = cfs_lghtcrv->rend[f].oe[i][j];
			pos[f]->se[i][j] = cfs_lghtcrv->rend[f].se[i][j];
		}
		/* Single-thread task */
		if (offset == 0) {
			if ((type == LGHTCRV) || (type == POS))
				pos[f]->bistatic = 1;
			else if ((type == DELAY) || (type == DOPPLER))
				pos[f]->bistatic = 0;

			bistatic[f] = pos[f]->bistatic;
		}
	}
}
__global__ void cfs_set_posbnd_streams_krnl(struct par_t *dpar, struct pos_t **pos,
		int f, unsigned char type) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		dpar->posbnd = 1;
		switch (type) {
		case DELAY:
			dpar->posbnd_logfactor += cfs_deldop->frame[f].dof * pos[f]->posbnd_logfactor;
			break;
		case DOPPLER:
			dpar->posbnd_logfactor += cf_doppler->frame[f].dof * pos[f]->posbnd_logfactor;
			break;
		case LGHTCRV:
			if (pos[f]->bistatic)
				cfs_lghtcrv_posbnd_logfactor += 0.5 * pos[f]->posbnd_logfactor;
			else
				cfs_lghtcrv_posbnd_logfactor += pos[f]->posbnd_logfactor;
			break;
		}
	}
}
__global__ void cf_get_exclude_seen_krnl(struct par_t *dpar, struct pos_t **pos,
		int4 *xylim, int f) {
	/* single-threaded kernel */

	if (threadIdx.x == 0) {
		cfs_exclude_seen = dpar->exclude_seen;
		xylim[f].w = pos[f]->xlim[0];
		xylim[f].x = pos[f]->xlim[1];
		xylim[f].y = pos[f]->ylim[0];
		xylim[f].z = pos[f]->ylim[1];
	}
}
__global__ void cf_mark_pixels_seen_streams_krnl(struct par_t *dpar,
		struct mod_t *dmod, struct pos_t **pos, int4 *xylim, int npixels,
		int xspan, int f) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int k = (offset % xspan) + xylim[f].w;
	int l = (offset / xspan) + xylim[f].y;
	int facetnum;

	if (offset < npixels) {
		if ((pos[f]->cose_s[offset] > dpar->mincosine_seen)
				&& (pos[f]->f[k][l] >= 0)) {
			facetnum = pos[f]->f[k][l];
			//c = cf_pos->comp[k][l];
			dmod->shape.comp[0].real.f[facetnum].seen = 1;
		}
	}
}
__global__ void cf_compute_and_set_lghtcrv_brightness_krnl(double brightness_temp,
		int i) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		cf_lghtcrv->y[i] = brightness_temp;
	}
}
__global__ void cf_set_badradar_streams_krnl(struct par_t *dpar, int f,
		unsigned char type) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		switch (type) {
		case DELAY:
			dpar->badradar = 1;
			dpar->badradar_logfactor += cfs_deldop->frame[f].dof *
			cfs_deldop->frame[f].badradar_logfactor / cfs_deldop->nviews;
			break;
		case DOPPLER:
			dpar->badradar = 1;
			dpar->badradar_logfactor += cfs_doppler->frame[f].dof *
				cfs_doppler->frame[f].badradar_logfactor / cfs_doppler->nviews;
			break;
		}
	}
}
__global__ void cf_add_fit_store_streams_krnl1(struct dat_t *ddat, float **fit_store,
		int nThreads, int s, int f) {
	/* ndel*ndop-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;

	if (offset < (nThreads)) {
		switch (cf_type) {
		case DELAY:
			fit_store[f][offset] += ddat->set[s].desc.deldop.frame[f].fit_s[offset];
			break;
		case DOPPLER:
			fit_store[f][offset] += ddat->set[s].desc.doppler.frame[f].fit_s[offset];
			break;
		}
	}
}
__global__ void cf_add_fit_store_streams_krnl2(float *overflow, int f,
		unsigned char type) {
	/* ndel*ndop-threaded kernel */
	if (threadIdx.x == 0) {
		switch (type) {
		case DELAY:
			atomicAdd(&overflow[0], (float)cfs_deldop->frame[f].overflow_o2);
			atomicAdd(&overflow[1], (float)cfs_deldop->frame[f].overflow_m2);
			atomicAdd(&overflow[2], (float)cfs_deldop->frame[f].overflow_xsec);
			atomicAdd(&overflow[3], (float)cfs_deldop->frame[f].overflow_delmean);
			atomicAdd(&overflow[4], (float)cfs_deldop->frame[f].overflow_dopmean);
			break;
		case DOPPLER:
			atomicAdd(&overflow[0], (float)cfs_doppler->frame[f].overflow_o2);
			atomicAdd(&overflow[1], (float)cfs_doppler->frame[f].overflow_m2);
			atomicAdd(&overflow[2], (float)cfs_doppler->frame[f].overflow_xsec);
			atomicAdd(&overflow[3], (float)cfs_doppler->frame[f].overflow_dopmean);
		}
	}
}
__global__ void cf_finish_fit_store_streams_krnl(float **fit_store,
		int s, int f, int nThreads, unsigned char type) {
	/* multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;

	if (offset < nThreads) {
		switch (type) {
		case DELAY:
			cfs_deldop->frame[f].fit_s[offset] = fit_store[f][offset];
			break;
		case DOPPLER:
			cfs_doppler->frame[f].fit_s[offset] = fit_store[f][offset];
			break;
		}
	}
}
__global__ void cf_finish_fit_streams_krnl2(float *overflow, int f, unsigned char type) {
	/* Single-threaded Kernel */
	int nviews;
	if (threadIdx.x == 0) {
		switch (type) {
		case DELAY:
			nviews = cfs_deldop->nviews;
			cfs_deldop->frame[f].overflow_o2 = overflow[0] / nviews;
			cfs_deldop->frame[f].overflow_m2 = overflow[1] / nviews;
			cfs_deldop->frame[f].overflow_xsec = overflow[2] / nviews;
			cfs_deldop->frame[f].overflow_delmean = overflow[3] / nviews;
			cfs_deldop->frame[f].overflow_dopmean = overflow[4] / nviews;
			break;
		case DOPPLER:
			nviews = cf_doppler->nviews;
			cfs_doppler->frame[f].overflow_o2 = overflow[0] / nviews;
			cfs_doppler->frame[f].overflow_m2 = overflow[1] / nviews;
			cfs_doppler->frame[f].overflow_xsec = overflow[2] / nviews;
			cfs_doppler->frame[f].overflow_dopmean = overflow[3] / nviews;
		}

	}
}
__global__ void cf_gamma_trans_streams_krnl(struct par_t *dpar, struct dat_t *ddat,
		int s, int f, int nThreads, unsigned char type) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (offset < nThreads) {
		/*  Carry out a gamma transformation on the fit image if requested  */
		if (dpar->dd_gamma != 1.0) {
			switch (type) {
			case DELAY:
				dev_gamma_trans(&ddat->set[s].desc.deldop.frame[f].fit_s[offset],
						dpar->dd_gamma);
				break;
			case DOPPLER:
				//cf_dop_frame->fit[offset] = fit[offset];
				break;
			}
		}
	}
}
__global__ void cf_posmask_krnl(struct par_t *dpar, int nThreads, int xspan)
{
	/* multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int n = cf_pos_n;
	int i = offset % xspan - n;
	int j = offset / xspan - n;
	double tol = dpar->mask_tol;
	int im, jm, i1, j1, i2, j2, i_sign, j_sign;
	double xk[3], so[3][3], pixels_per_km, i0_dbl, j0_dbl, zill, t, u, bignum;

	if (offset == 0){
		bignum = 0.99*HUGENUMBER;  /* z = -HUGENUMBER for blank-sky pixels */

		dev_mtrnsps( so, cf_pos->oe);
		dev_mmmul( so, cf_pos->se, so);    /* so takes obs into src coords */
		pixels_per_km = 1/cf_pos->km_per_pixel;
	}
	__syncthreads();

	/*  Loop through all POS pixels  */
	if (offset < nThreads) {
		//	n = cf_pos->n;
		//	for (i=(-n); i<=n; i++) {               /* for each pixel in the */
		//		for (j=(-n); j<=n; j++) {             /* observer's view */
		if (cf_pos->cose_s[offset] != 0.0) {     /* if there's something there */
			xk[0] = i*cf_pos->km_per_pixel;     /* calculate 3D position */
			xk[1] = j*cf_pos->km_per_pixel;
			xk[2] = cf_pos->z_s[offset];

			/* Given the observer coordinates x of of POS pixel (i,j), find
			 * which pixel (im,jm) this corresponds to in the projected view as
			 * seen from the source (sun)             */

			dev_cotrans2( xk, so, xk, 1);           /* go into source coordinates */
			i0_dbl = xk[0]*pixels_per_km;     /* unrounded (double precision) */
			j0_dbl = xk[1]*pixels_per_km;
			im = dev_vp_iround( i0_dbl);            /* center of nearest pixel in mask */
			jm = dev_vp_iround( j0_dbl);

			/* If center of projected pixel "seen" from source (as determined
			 * by routine posvis) lies within the boundaries of the mask,
			 * projects onto model rather than onto blank space, and represents
			 * a body, component, and facet different from those seen in the
			 * POS, calculate distance from mask pixel to source and compare to
			 * distance from POS pixel to source.                             */

			if (fabs(i0_dbl) < n && fabs(j0_dbl) < n
					&& cf_pos->zill[im][jm] > -bignum
					&& (cf_pos->f[i][j]    != cf_pos->fill[im][jm]    ||
							cf_pos->comp[i][j] != cf_pos->compill[im][jm] ||
							cf_pos->body[i][j] != cf_pos->bodyill[im][jm]    )) {

				/* Rather than using distance towards source of CENTER of mask
				 * pixel, use bilinear interpolation to get distance towards
				 * source where the line between source and POS pixel's center
				 * intersects the mask pixel.                                */
				i1 = (int) floor( i0_dbl);
				j1 = (int) floor( j0_dbl);

				if (cf_pos->zill[i1][j1]     > -bignum &&
						cf_pos->zill[i1+1][j1]   > -bignum &&
						cf_pos->zill[i1][j1+1]   > -bignum &&
						cf_pos->zill[i1+1][j1+1] > -bignum    ) {

					/* Do standard bilinear interpolation: None of the four
					 * surrounding "grid square" pixels in the mask is
					 * blank sky                           */
					t = i0_dbl - i1;
					u = j0_dbl - j1;
					zill = (1 - t)*(1 - u)*cf_pos->zill[i1][j1]
                               + t*(1 - u)*cf_pos->zill[i1+1][j1]
                                     + t*u*cf_pos->zill[i1+1][j1+1]
	                           + (1 - t)*u*cf_pos->zill[i1][j1+1];
				} else {

					/* The following code block is a kludge: One or more of the
					 * four surrounding "grid square" pixels in mask is blank
					 * sky, so standard bilinear interpolation won't work.  */
					zill = cf_pos->zill[im][jm];

					i_sign = (i0_dbl >= im) ? 1 : -1;
					i2 = im + i_sign;
					if (abs(i2) <= n && cf_pos->zill[i2][jm] > -bignum) {
						zill += fabs(i0_dbl - im)
           				  * (cf_pos->zill[i2][jm] - cf_pos->zill[im][jm]);
					} else {
						i2 = im - i_sign;
						if (abs(i2) <= n && cf_pos->zill[i2][jm] > -bignum)
							zill -= fabs(i0_dbl - im)
							* (cf_pos->zill[i2][jm] - cf_pos->zill[im][jm]);
					}

					j_sign = (j0_dbl >= jm) ? 1 : -1;
					j2 = jm + j_sign;
					if (abs(j2) <= n && cf_pos->zill[im][j2] > -bignum) {
						zill += fabs(j0_dbl - jm)
                          * (cf_pos->zill[im][j2] - cf_pos->zill[im][jm]);
					} else {
						j2 = jm - j_sign;
						if (abs(j2) <= n && cf_pos->zill[im][j2] > -bignum)
							zill -= fabs(j0_dbl - jm)
							* (cf_pos->zill[im][j2] - cf_pos->zill[im][jm]);
					}
				}

				/* If interpolated point within mask pixel is at least tol km
				 * closer to source than is the center of POS pixel, the facet
				 * represented by the mask pixel is shadowing the POS pixel:
				 * represent this by setting
				 * 		cos(scattering angle) = 0.0 for the POS pixel.      */
				if (zill - xk[2] > tol)
					cf_pos->cose_s[offset] = 0.0;
			}
		}
	}
}

__host__ void calc_deldop_cuda_streams(struct par_t *dpar, struct mod_t *dmod,
		struct dat_t *ddat, int s, int nframes, int nviews, unsigned char type,
		hipStream_t *cf_stream)
{
	double orbit_offset[3] = {0.0, 0.0, 0.0};
	int *ndel, *ndop, *posn, *bistatic, v0_index, pos_n, nx, exclude_seen,
		f, v2, c=0;
	float **fit_store, *overflow;
	dim3 BLKdd[nframes], BLKpx[nframes], THD, THD9;
	THD.x = maxThreadsPerBlock; THD9.x = 9;
	int4 *xylim, hxylim[nframes];
	struct pos_t **pos;
	struct deldopfrm_t **frame;
	struct deldopview_t **view0;
	int hndop[nframes], hndel[nframes], hposn[nframes], hbistatic[nframes],
		outbndarr[nframes], xspan[nframes], nThreadspx[nframes], nThreadsdd[nframes],
		nThreadspx1[nframes];

	cudaCalloc((void**)&pos, sizeof(pos_t*), nframes);
	cudaCalloc((void**)&frame, sizeof(deldopfrm_t*), nframes);
	cudaCalloc((void**)&view0, sizeof(deldopview_t*), nframes);
	cudaCalloc((void**)&fit_store, sizeof(float*), nframes);
	cudaCalloc((void**)&ndel, sizeof(int), nframes);
	cudaCalloc((void**)&ndop, sizeof(int), nframes);
	cudaCalloc((void**)&posn, sizeof(int), nframes);
	cudaCalloc((void**)&bistatic, sizeof(int), nframes);
	cudaCalloc((void**)&xylim, sizeof(int4), nframes);
	cudaCalloc((void**)&overflow, sizeof(float), 6);

	for (f=0; f<nframes; f++)
		/* Set deldop, frame, view0, and pos in nframes streamed kernels */
		cfs_set_deldop_shortcuts_krnl<<<1,1,0,cf_stream[f]>>>(ddat, frame, pos,
				view0, ndop, ndel, overflow, posn, s, f);
	checkErrorAfterKernelLaunch("cfs_set_deldop_shortcuts_krnl");
	gpuErrchk(hipMemcpy(&hndel, ndel, sizeof(int)*nframes, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&hndop, ndop, sizeof(int)*nframes, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&hposn, posn, sizeof(int)*nframes, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&v0_index, HIP_SYMBOL(cfs_v0_index), sizeof(int),
				0, hipMemcpyDeviceToHost));

	for (f=0; f<nframes; f++) {
		nThreadsdd[f] = hndel[f]*hndop[f];
		BLKdd[f].x = floor((THD.x - 1 + nThreadsdd[f]) /	THD.x);
		nThreadspx[f] = (2 * hposn[f] + 1) * (2 * hposn[f] + 1);
		BLKpx[f].x = floor((THD.x -1 + nThreadspx[f]) / THD.x);

		/* If smearing is being modeled, initialize variables that
		 * will be used to sum results calculated for individual views.  */
		if (nviews > 1) {
			/* Allocate fit_store as a single pointer, originally a double
			 * pointer. This also initializes the entire array to zero. */
			cudaCalloc((void**)&fit_store, sizeof(float), hndel[f]*hndop[f]);
		}
	}
	/*  Loop over all views for this (smeared) frame, going in an order that
        ends with the view corresponding to the epoch listed for this frame
        in the obs file; this way we can use the calculated information for
        that view in the "write" action screen and disk output that follows   */
	int no_views = (v0_index+nviews) - (v0_index+1) + 1;
	int v[no_views];
	int index = 0;
	for (v2=v0_index+1; v2<=v0_index+nviews; v2++) {
		v[index] = v2 % nviews;
		index++;
	}

	for (f=0; f<nframes; f++) {
		for (v2=0; v2<no_views; v2++) {
			/* Launch 9-threaded kernel to set pos->ae,pos->oe,pos->bistatic.*/
			cfs_set_pos_ae_streams_krnl<<<1,THD,0,cf_stream[f]>>>(pos, f,
					bistatic, type, v[v2]);

			/* Launch posclr_krnl to initialize POS view */
			posclr_streams_krnl<<<BLKpx[f],THD,0,cf_stream[f]>>>(pos, posn, f);

		}
	} checkErrorAfterKernelLaunch("posclr_streams_krnl (calc_fits_cuda_streams)");
	gpuErrchk(hipMemcpy(&hbistatic, bistatic, sizeof(int)*nframes,
			hipMemcpyDeviceToHost));

	/* Call posvis_cuda_2 to get facet number, scattering angle, distance
	 * toward Earth at center of each POS pixel; set flag posbnd if any model
	 * portion extends beyond POS frame limits.*/
	/* NOTE: Limited to single component for now */

	for (v2=0; v2<no_views; v2++)
		posvis_cuda_streams(dpar, dmod, ddat, orbit_offset, s, nframes,
						0, 0, c, outbndarr, cf_stream);

	for (f=0; f<nframes; f++) {
		for (v2=0; v2<no_views; v2++) {
			if ((outbndarr[f]) && (v[v2] == v0_index)) {
			/* Call single-threaded kernel to set dpar->posbnd and dpar->posbnd_logfactor */
			cfs_set_posbnd_krnl<<<1,1,0,cf_stream[f]>>>(dpar, pos, f, type);
			outbndarr[f]=0;
			}
		}
		/* Get xlim and ylim and exclude_seen flag */
		cfs_get_exclude_seen_streams_krnl<<<1,1,0,cf_stream[f]>>>(dpar,pos,xylim,f);
	} checkErrorAfterKernelLaunch("cfs_set_posbnd_streams_krnl and"
			"cfs_get_exclude_seen_streams_krnl");

	/* Now copy the flag and all frame pos's xlim and ylim values back from GPU */
	gpuErrchk(hipMemcpyFromSymbol(&exclude_seen, HIP_SYMBOL(cf_exclude_seen), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&hxylim, xylim, sizeof(int4)*nframes, hipMemcpyDeviceToHost));

	/* Calculate launch parameters for all frames */
	for (f=0; f<nframes; f++) {
		xspan[f] = hxylim[f].x - hxylim[f].w + 1;
		yspan = hxylim[f].y - hxylim[f].z + 1;
		nThreadspx1[f] = xspan[f] * yspan;
		BLKpx[f].x = (THD.x -1 + nThreadspx1[f]) / THD.x;
	}

	for (f=0; f<nframes; f++) {
		for (v2=0; v2<no_views; v2++) {
			/* Go through all POS pixels which are visible with low enough
			 * scattering angle and mark the facets which project onto their
			 * centers as having been "seen" at least once                   */
			if (s != exclude_seen && v[v2] == v0_index)
				cf_mark_pixels_seen_streams_krnl<<<BLKpx[f],THD,0,cf_stream[f]>>>(
						dpar, dmod, pos, xylim, nThreadspx1[f], xspan[f], f);

			/* Zero out the fit delay-Doppler image, then call pos2deldop to
			 * create the fit image by mapping power from the plane of the sky
			 * to delay-Doppler space.                             */
			clrvect_krnl<<<BLKdd[f],THD,0,cf_stream[f]>>>(ddat, s, f, nThreadspx[f]);
		} checkErrorAfterKernelLaunch("clrvect_krnl and cf_mark_pixels_seen_streams_krnl");
	}

	for (v2=0; v2<no_views; v2++)
		pos2deldop_cuda_streams(dpar,dmod,ddat, pos, ndel, ndop, 0.0,0.0,0.0,0,
				s, nframes, v[f2], outbndarr, cf_stream);

	for (f=0; f<nframes; f++) {
		for (v2=0; v2<no_views; v2++) {
			if (outbndarr[f]) {
				/* Call single-threaded kernel to set badradar flag and
				 * associated badradar_logfactor			 */
				cf_set_badradar_streams_krnl<<<1,1,0,cf_stream[f]>>>(dpar, f, type);
				checkErrorAfterKernelLaunch("cf_deldop_set_badradar_krnl (calc_fits_cuda)");
			}
		}
	}

	for (f=0; f<nframes; f++) {
		/* If smearing is being modeled, include delay-Doppler calculations
		 * from this view in the summed results for this frame  */
		if (nviews > 1) {
			/* Launch ndel*ndop-threaded kernel to add fit[i][j] to
			 * fit_store[i][j]*/

			cf_add_fit_store_streams_krnl1<<<BLKdd[f],THD,0,cf_stream[f]>>>(ddat,fit_store,nThreadsdd[f],s,f);

			cf_add_fit_store_streams_krnl2<<<1,1>>>(overflow, f, type);
		}
	} checkErrorAfterKernelLaunch("cf_add_fit_store_streams_krnl1 and 2");


	/* If smearing is being modeled, compute mean values over all views for
	 * this frame and store them in the standard frame structure     */
	/* This kernel also carries out the gamma transformation on the fit
	 * image if the par->dd_gamma flag is not set  */
	if (nviews > 1) {
		for (f=0; f<nframes; f++) {

			cf_finish_fit_store_streams_krnl<<<BLKdd[f],THD,0,cf_stream[f]>>>(
					fit_store, s, f, nThreadsdd[f], type);

			cf_finish_fit_streams_krnl2<<<1,1,0,cf_stream[f]>>>(overflow, f, type);

			cf_gamma_trans_streams_krnl<<<BLKdd[f],THD,0,cf_stream[f]>>>(dpar, ddat, s, f, nThreadsdd[f], type);
		} checkErrorAfterKernelLaunch("cf_finish_fit_store_streams kernels and "
				"cf_gamma_trans_krnl");
		hipFree(fit_store);
	}
}

//__host__ void calc_doppler_cuda(struct par_t *dpar, struct mod_t *dmod,
//		struct dat_t *ddat, int s)
//{
//	double orbit_offset[3] = {0.0, 0.0, 0.0};
//	float *fit_store;
//	int ndop, v0_index, pos_n, xlim0, xlim1, ylim0, ylim1, exclude_seen,
//		nviews, nframes, nx, f, v, v2;
//	dim3 BLK,THD;
//
//	/* Get # of frames for this doppler frame */
//	cf_get_frames_krnl<<<1,1>>>(ddat, s);
//	checkErrorAfterKernelLaunch("cf_get_nframes_krnl (calc_deldop_cuda)");
//	gpuErrchk(hipMemcpyFromSymbol(&nframes, HIP_SYMBOL(cf_nframes), sizeof(int),
//			0, hipMemcpyDeviceToHost));
//
//	for (f=0; f<nframes; f++) {
//		/* Set deldop, frame, view0, and pos */
//		cf_set_shortcuts_krnl<<<1,1>>>(ddat, s, f);
//		checkErrorAfterKernelLaunch("cf_deldop_1st_krnl (calc_deldop_cuda)");
//		gpuErrchk(hipMemcpyFromSymbol(&nviews, HIP_SYMBOL(cf_nviews), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//		gpuErrchk(hipMemcpyFromSymbol(&ndop, HIP_SYMBOL(cf_ndop), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//		gpuErrchk(hipMemcpyFromSymbol(&v0_index, HIP_SYMBOL(cf_v0_index), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//
//		/* If smearing is being modeled, initialize variables that
//		 * will be used to sum results calculated for individual views.  */
//		if (nviews > 1)
//			/* Allocate fit_store as a single pointer, originally a double
//			 * pointer. This also initializes the entire array to zero. */
//			cudaCalloc((void**)&fit_store, sizeof(float), ndop);
//
//		/* Loop over all views for this (smeared) frame, going in an order that
//		 * ends with the view corresponding to the epoch listed for this frame
//		 * in the obs file; this way we can use the calculated information for
//		 * that view in the "write" action screen and disk output that follows*/
//
//		for (v2=v0_index+1; v2<=v0_index+nviews; v2++) {
//			v = v2 % nviews;
//
//			/* Launch 9-threaded kernel to set pos->ae,pos->oe,pos->bistatic.*/
//			THD.x = 9;
//			cf_set_pos_ae_krnl<<<BLK,THD>>>(v);
//			checkErrorAfterKernelLaunch("cf_deldop_set_pos_ae_krnl "
//					"(calc_doppler_cuda)");
//			gpuErrchk(hipMemcpyFromSymbol(&pos_n, HIP_SYMBOL(cf_pos_n), sizeof(int),
//					0, hipMemcpyDeviceToHost));
//
//			/* Configure & launch posclr_krnl to initialize POS view */
//			BLK.x = floor((maxThreadsPerBlock - 1 + (2*pos_n+1)*(2*pos_n+1)) /
//					maxThreadsPerBlock);
//			THD.x = maxThreadsPerBlock; // Thread block dimensions
//			nx = 2*pos_n + 1;
//			cf_posclr_krnl<<<BLK,THD>>>(pos_n, nx);
//			checkErrorAfterKernelLaunch("cf_posclr_krnl (calc_fits_doppler)");
//
//			/* Call posvis_cuda_2 to get facet number, scattering angle,
//			 * distance toward Earth at center of each POS pixel; set flag
//			 * posbnd if any model portion extends beyond POS frame limits.*/
//			/* NOTE: Limited to single component for now */
//
//			if (posvis_cuda_2(dpar, dmod, ddat, orbit_offset, s, f, 0, 0, 0) &&
//					v == v0_index) {
//				/* Call single-threaded kernel to set dpar->posbnd and
//				 * dpar->posbnd_logfactor */
//				cf_set_posbnd_krnl<<<1,1>>>(dpar);
//				checkErrorAfterKernelLaunch("cf_deldop_set_posbnd_krnl (calc_fits_cuda)");
//			}
//
//			/* Launch single-threaded kernel to get dpar->exclude_seen */
//			cf_get_exclude_seen_krnl<<<1,1>>>(dpar);
//			checkErrorAfterKernelLaunch("cf_get_exclude_seen_krnl (calc_fits_cuda)");
//			gpuErrchk(hipMemcpyFromSymbol(&exclude_seen, HIP_SYMBOL(cf_exclude_seen), sizeof(int),
//					0, hipMemcpyDeviceToHost));
//			gpuErrchk(hipMemcpyFromSymbol(&xlim0, HIP_SYMBOL(cf_xlim0), sizeof(int),
//					0, hipMemcpyDeviceToHost));
//			gpuErrchk(hipMemcpyFromSymbol(&xlim1, HIP_SYMBOL(cf_xlim1), sizeof(int),
//					0, hipMemcpyDeviceToHost));
//			gpuErrchk(hipMemcpyFromSymbol(&ylim0, HIP_SYMBOL(cf_ylim0), sizeof(int),
//					0, hipMemcpyDeviceToHost));
//			gpuErrchk(hipMemcpyFromSymbol(&ylim1, HIP_SYMBOL(cf_ylim1), sizeof(int),
//					0, hipMemcpyDeviceToHost));
//
//			/* Go through all POS pixels visible with low enough scattering
//			 * angle and mark the facets which project onto their centers as
//			 * having been "seen" at least once                        */
//			/* I'll launch a multi-threaded kernel here:
//			 * (xlim1 - xlim0 + 1)^2 threads			 */
//			if (s != exclude_seen && v == v0_index) {
//
//				int xspan = xlim1 - xlim0 + 1;
//				int yspan = ylim1 - ylim0 + 1;
//				int nThreads = xspan * yspan;
//
//				/* Configure & launch posclr_krnl to initialize POS view */
//				BLK.x = floor((maxThreadsPerBlock - 1 + nThreads) /
//						maxThreadsPerBlock);
//				THD.x = maxThreadsPerBlock; // Thread block dimensions
//				cf_mark_pixels_seen_krnl<<<BLK,THD>>>(dpar, dmod,
//						nThreads, xspan);
//				checkErrorAfterKernelLaunch("cf_mark_pixels_krnl (calc_fits_cuda)");
//			}
//
//			/* Zero out fit Doppler spectrum, then call pos2doppler to create
//			 * the fit image by mapping power from the plane of the sky to
//			 * Doppler space.                             */
//			/* Zero out the fit delay-Doppler image, then call pos2deldop to
//			 * create the fit image by mapping power from the plane of the sky
//			 * to delay-Doppler space.                             */
//			BLK.x = floor((maxThreadsPerBlock - 1 + ndop) /
//					maxThreadsPerBlock);
//			THD.x = maxThreadsPerBlock; // Thread block dimensions
//			clrvect_krnl<<<BLK,THD>>>(ddat, s, f, ndop);
//			checkErrorAfterKernelLaunch("clrvect_krnl, calc_fits_cuda:1060");
//
//			if (pos2doppler_cuda_2(dpar,dmod,ddat,0.0,0.0,0.0,0, s,f,v)) {
//				/* Call single-threaded kernel to set badradar flag and
//				 * associated badradar_logfactor			 */
//				cf_set_badradar_krnl<<<1,1>>>(dpar);
//				checkErrorAfterKernelLaunch("cf_set_badradar_krnl (calc_fits_cuda)");
//			}
//
//			/* If smearing is being modeled, include Doppler calculations from
//			 * this view in the summed results for this frame  */
//			if (nviews > 1) {
//				/* Launch ndel*ndop-threaded kernel to add fit[i][j] to
//				 * fit_store[i][j]*/
//				BLK.x = floor((maxThreadsPerBlock - 1 + ndop) /
//						maxThreadsPerBlock);
//				THD.x = maxThreadsPerBlock; // Thread block dimensions
//				cf_add_fit_store_krnl1<<<BLK,THD>>>(ddat,fit_store,ndop,s,f);
//				checkErrorAfterKernelLaunch("cf_add_fit_store_krnl1 (calc_fits_cuda)");
//				cf_add_fit_store_krnl2<<<1,1>>>();
//				checkErrorAfterKernelLaunch("cf_add_fit_store_krnl2 (calc_fits_cuda)");
//			}
//		}
//
//		/* If smearing is being modeled, compute mean values over all views for
//		 * this frame and store them in the standard frame structure     */
//		/* This kernel also carries out the gamma transformation on the fit
//		 * image if the par->dd_gamma flag is not set  */
//		if (nviews > 1) {
//			/* Launch ndop-threaded kernel to add fit[i] to
//			 * fit_store[i]*/
//			BLK.x = floor((maxThreadsPerBlock - 1 + ndop) /
//					maxThreadsPerBlock);
//			THD.x = maxThreadsPerBlock; // Thread block dimensions
//			cf_finish_fit_store_krnl<<<BLK,THD>>>(ddat, fit_store,s,f,ndop);
//			checkErrorAfterKernelLaunch("cf_deldop_finish_fit_store (calc_fits_cuda)");
//			cf_finish_fit_krnl2<<<1,1>>>();
//			checkErrorAfterKernelLaunch("cf_finish_fit_krnl2");
//			cf_gamma_trans_krnl<<<BLK,THD>>>(dpar, ddat, s, f, ndop, type);
//			checkErrorAfterKernelLaunch("cf_gamma_trans_krnl");
//			hipFree(fit_store);
//		}
//	}  /* end loop over frames */
//}
//
////void calc_poset( struct par_t *par, struct mod_t *mod, struct poset_t *poset,
////		int s)
////{
////	const char *monthName[12] = {"Jan", "Feb", "Mar", "Apr", "May", "Jun",
////			"Jul", "Aug", "Sep", "Oct", "Nov", "Dec"};
////	double orbit_offset[3] = {0.0, 0.0, 0.0};
////
////	FILE *fpopt;
////	char tempstring[MAXLEN], name[MAXLEN];
////	int year, mon, day, hour, min, sec, f, c, i, j, k, l, nrow_fit, ncol_fit, n_pos,
////	facetnum, x, y, v, v2;
////	double w[3], spin_colat, spin_azim, xoff, yoff, resamp_fact, resamp_x0, resamp_y0,
////	xcom_fit, ycom_fit, resamp_xwidth, resamp_ywidth, resamp_angle, oa[3][3],
////	to_earth[3], to_earth_lat, to_earth_long, rotphase, sa[3][3], to_sun[3],
////	to_sun_lat, to_sun_long, pab[3], pab_lat, pab_long, intensityfactor,
////	phi, theta, psi, intspin_body[3], badposet_logfactor_view;
////	double **fit_store;
////	struct posetfrm_t *frame;
////	struct posetview_t *view0;
////	struct pos_t *pos;
////
////	for (f=0; f<poset->nframes; f++) {
////
////		frame = &poset->frame[f];
////		view0 = &frame->view[poset->v0];
////		pos = &frame->pos;
////
////		ncol_fit = frame->ncol;
////		nrow_fit = frame->nrow;
////
////		/*  If smearing is being modeled, initialize variables that
////        will be used to sum results calculated for individual views  */
////
////		if (poset->nviews > 1) {
////			fit_store = matrix( 1, ncol_fit, 1, nrow_fit);
////			for (i=1; i<=ncol_fit; i++)
////				for (j=1; j<=nrow_fit; j++)
////					fit_store[i][j] = 0.0;
////		}
////
////		/*  Loop over all views for this (smeared) frame, going in an order that
////        ends with the view corresponding to the epoch listed for this frame
////        in the obs file; this way we can use the calculated information for
////        that view in the "write" action screen and disk output that follows   */
////
////		for (v2=poset->v0+1; v2<=poset->v0+poset->nviews; v2++) {
////			v = v2 % poset->nviews;
////
////			for (i=0; i<=2; i++)
////				for (j=0; j<=2; j++) {
////					pos->ae[i][j] = frame->view[v].ae[i][j];
////					pos->oe[i][j] = frame->view[v].oe[i][j];
////					pos->se[i][j] = frame->view[v].se[i][j];
////				}
////			pos->bistatic = 1;
////
////			/*  Initialize the plane-of-sky view  */
////
////			posclr( pos);
////
////			/*  Call routine posvis to get the facet number, scattering angle,
////          incidence angle, and distance toward Earth at the center of
////          each POS pixel; set the posbnd parameter to 1 if any portion
////          of the model extends beyond the POS frame limits.              */
////
////			for (c=0; c<mod->shape.ncomp; c++)
////				if (posvis( &mod->shape.comp[c].real, orbit_offset, pos,
////						(int) par->pos_smooth, 0, 0, c) && v == poset->v0) {
////					par->posbnd = 1;
////					if (pos->bistatic)
////						par->posbnd_logfactor += 0.5 * frame->dof * pos->posbnd_logfactor;
////					else
////						par->posbnd_logfactor += frame->dof * pos->posbnd_logfactor;
////				}
////
////			/*  Now view the model from the source (sun) and get the facet number
////          and distance toward the source of each pixel in this projected view;
////          use this information to determine which POS pixels are shadowed       */
////
////			if (pos->bistatic) {
////				for (c=0; c<mod->shape.ncomp; c++)
////					if (posvis( &mod->shape.comp[c].real, orbit_offset, pos,
////							0, 1, 0, c)) {
////						par->posbnd = 1;
////						par->posbnd_logfactor += 0.5 * frame->dof * pos->posbnd_logfactor;
////					}
////
////				/*  Identify and mask out shadowed POS pixels  */
////
////				posmask( pos, par->mask_tol);
////			}
////
////			/*  Go through all POS pixels which are visible and unshadowed with
////          sufficiently low scattering and incidence angles, and mark the facets
////          which project onto their centers as having been "seen" at least once   */
////
////			if (s != par->exclude_seen && v == poset->v0) {
////				for (k=pos->xlim[0]; k<=pos->xlim[1]; k++)
////					for (l=pos->ylim[0]; l<=pos->ylim[1]; l++) {
////						if ((pos->cose[k][l] > par->mincosine_seen)
////								&& (pos->cosi[k][l] > par->mincosine_seen)
////								&& (pos->f[k][l] >= 0)) {
////							facetnum = pos->f[k][l];
////							c = pos->comp[k][l];
////							mod->shape.comp[c].real.f[facetnum].seen = 1;
////						}
////					}
////			}
////
////			/*  Compute the sky rendering  */
////
////			intensityfactor = pow( pos->km_per_pixel/AU, 2.0);
////			apply_photo( mod, poset->ioptlaw, frame->view[v].solar_phase,
////					intensityfactor, pos, 0);
////
////			/*  Resample the sky rendering to get the model plane-of-sky image    */
////			/*  (if using bicubic interpolation or cubic convolution, force       */
////			/*  all model pixel values to be nonnegative)                         */
////			/*                                                                    */
////			/*  Implement the x and y COM offsets, xoff and yoff, by first        */
////			/*  using them to compute xcom_fit and ycom_fit -- the COM position   */
////			/*  in the fit image, relative to the center of the fit image -- and  */
////			/*  then shifting the resampled region in the *opposite* direction    */
////			/*  by the appropriate proportional amount.  Then implement the       */
////			/*  "northangle" setting (clockwise heading of north) by rotating     */
////			/*  the resampling grid *counterclockwise* by northangle.             */
////
////			n_pos = pos->n;
////			xoff = frame->off[0].val;
////			yoff = frame->off[1].val;
////			xcom_fit = (frame->colcom_vig - (ncol_fit + 1)/2.0) + xoff;
////			ycom_fit = (frame->rowcom_vig - (nrow_fit + 1)/2.0) + yoff;
////			resamp_fact = frame->fit.km_per_pixel / pos->km_per_pixel;
////			resamp_x0 = -xcom_fit*resamp_fact;
////			resamp_y0 = -ycom_fit*resamp_fact;
////			resamp_xwidth = resamp_fact*(ncol_fit - 1);
////			resamp_ywidth = resamp_fact*(nrow_fit - 1);
////			resamp_angle = -frame->northangle;
////			resampim( frame->pos.b, -n_pos, n_pos, -n_pos, n_pos,
////					frame->fit.b, 1, ncol_fit, 1, nrow_fit,
////					resamp_x0, resamp_xwidth, resamp_y0, resamp_ywidth, resamp_angle,
////					(int) par->poset_resample, (int) par->image_rebin);
////			if (par->poset_resample == BICUBIC || par->poset_resample == CUBICCONV) {
////				for (k=1; k<=ncol_fit; k++)
////					for (l=1; l<=nrow_fit; l++)
////						frame->fit.b[k][l] = MAX( 0.0, frame->fit.b[k][l]);
////			}
////
////			/*  Set the badposet flag and increase badposet_logfactor if the model   */
////			/*  plane-of-sky image is too small to "contain" all of the sky          */
////			/*  rendering's nonzero pixels.                                          */
////
////			if (checkposet( pos->b, -n_pos, n_pos, -n_pos, n_pos,
////					resamp_x0, resamp_xwidth, resamp_y0, resamp_ywidth, resamp_angle,
////					&badposet_logfactor_view)) {
////				par->badposet = 1;
////				par->badposet_logfactor += frame->dof * badposet_logfactor_view
////						/ poset->nviews;
////			}
////
////			/*  If smearing is being modeled, include the plane-of-sky
////          calculations from this view in the summed results for this frame  */
////
////			if (poset->nviews > 1)
////				for (i=1; i<=ncol_fit; i++)
////					for (j=1; j<=nrow_fit; j++)
////						fit_store[i][j] += frame->fit.b[i][j];
////
////		}
////
////		/*  If smearing is being modeled, compute mean values over all views
////        for this frame and store them in the standard frame structure     */
////
////		if (poset->nviews > 1) {
////			for (i=1; i<=ncol_fit; i++)
////				for (j=1; j<=nrow_fit; j++)
////					frame->fit.b[i][j] = fit_store[i][j] / poset->nviews;
////			free_matrix( fit_store, 1, ncol_fit, 1, nrow_fit);
////		}
////
////
////	}  /* end loop over frames */
////}
////
////
//__host__ void calc_lghtcrv_cuda( struct par_t *dpar, struct mod_t *dmod, struct dat_t *ddat, int s)
//{
//	int ncalc, c=0, i, pos_n, n, xspan, yspan, nThreads,
//			bistatic, exclude_seen, xlim[2], ylim[2];
//	double km_p_pxl, brightness_temp, orbit_offset[3] = {0.0, 0.0, 0.0};
//
//	dim3 BLK,THD;
////	struct pos_t *pos;
////	cudaCalloc((void**)&pos, sizeof(struct pos_t), 1);
//
//	/* Get n (# of observed points for this lightcurve), and ncalc (# of epochs
//	 * at which model lightcurve brightnesses are to be computed            */
//	cf_get_frames_krnl<<<1,1>>>(ddat, s);
//	checkErrorAfterKernelLaunch("cf_get_frames_krnl (calc_lghtcrv_cuda)");
//	gpuErrchk(hipMemcpyFromSymbol(&ncalc, HIP_SYMBOL(cf_ncalc), sizeof(int),
//			0, hipMemcpyDeviceToHost));
//
//	/* Calculate model lightcurve values at each user-specified epochs x[i],
//	 * with i=1,2,...,ncalc; these may/may not be the same as epochs t[i]
//	 * (i=1,2,...,n) at which actual lightcurve observations were made.  */
//
//	/* Problem description:  cf_lghtcrv->rend[i=8].oe for set 2 changes between i=1 and i=2
//	 * it should stay at small numbers (<1.0), but the change prompts an identity matrix like
//	 * pattern of 1,1; 2,2; 3,3 being equal to lghtcrv->x[i] = 2447670.5833
//	 */
//	for (i=1; i<=ncalc; i++) {
//		/* Set lghtcrv, rend, and pos */
//		cf_set_shortcuts_krnl<<<1,1>>>(ddat, s, i);
//		checkErrorAfterKernelLaunch("cf_set_lghtcrv_shortcuts_krnl");
//		gpuErrchk(hipMemcpyFromSymbol(&km_p_pxl, HIP_SYMBOL(cf_km_p_pxl), sizeof(double),
//				0, hipMemcpyDeviceToHost));
//		gpuErrchk(hipMemcpyFromSymbol(&n, HIP_SYMBOL(cf_n), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//
//		/* Launch 9-threaded kernel to set pos->ae,pos->oe,pos->bistatic.*/
//		THD.x = 9;
//		cf_set_pos_ae_krnl<<<BLK,THD>>>(0);
//		checkErrorAfterKernelLaunch("cf_set_pos_ae_krnl "
//				"(calc_lghtcrv_cuda)");
//		gpuErrchk(hipMemcpyFromSymbol(&pos_n, HIP_SYMBOL(cf_pos_n), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//
//		/* Configure & launch posclr_krnl to initialize POS view */
//		xspan = 2*pos_n + 1;
//		nThreads = xspan * xspan;
//		BLK.x = floor((maxThreadsPerBlock-1+nThreads)/maxThreadsPerBlock);
//		THD.x = maxThreadsPerBlock;
//		cf_posclr_krnl<<<BLK,THD>>>(pos_n, xspan);
//		checkErrorAfterKernelLaunch("cf_posclr_krnl (calc_lghtcrv_cuda)");
//
//		/* Call routine posvis to get  facet number, scattering & incidence
//		 * angle, distance toward Earth at center of each POS pixel; set posbnd
//		 * parameter = 1 if any model portion extends beyond POS frame limits*/
////		for (c=0; c<mod->shape.ncomp; c++)
//		if (posvis_cuda_2(dpar, dmod, ddat, orbit_offset, s, i, 0, 0, c)) {
//			/* Call single-threaded kernel to set dpar->posbnd and
//			 * dpar->posbnd_logfactor */
//			cf_set_posbnd_krnl<<<1,1>>>(dpar);
//			checkErrorAfterKernelLaunch("cf_set_posbnd_krnl (calc_lghtcrv_cuda)");
//		}
//		gpuErrchk(hipMemcpyFromSymbol(&bistatic, HIP_SYMBOL(cf_bistatic), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//
////		if ((s==2)) {
////			nThreads = (2*pos_n+1)*(2*pos_n+1);
////			dbg_print_lghtcrv_pos_arrays(ddat, s, i, nThreads, pos_n);
////			printf("\n");
////		}
//
//
//		/* Now view model from source (sun) and get facet number and distance
//		 * toward source of each pixel in this projected view; use this
//		 * information to determine which POS pixels are shadowed       */
//		if (bistatic) {
//			//for (c=0; c<mod->shape.ncomp; c++)
//			if (posvis_cuda_2(dpar, dmod, ddat, orbit_offset, s, i, 1, 0, c)) {
//				cf_set_posbnd_krnl<<<1,1>>>(dpar);
//				checkErrorAfterKernelLaunch("cf_set_posbnd_krnl");
//			}
//
//			/* Identify and mask out shadowed POS pixels  */
//			cf_posmask_krnl<<<BLK,THD>>>(dpar, nThreads, xspan);
//			checkErrorAfterKernelLaunch("cf_posmask_krnl");
//		}
//		/* Go through all visible and unshadowed POS pixels with low enough
//		 * scattering and incidence angles, and mark facets which project onto
//		 * their centers as having been "seen" at least once   */
//		/* First call kernel to get the exclude_seen flag and xlim/ylim */
//		cf_get_exclude_seen_krnl<<<1,1>>>(dpar);
//		checkErrorAfterKernelLaunch("cf_get_exclude_seen_krnl (calc_lghtcrv_cuda)");
//		gpuErrchk(hipMemcpyFromSymbol(&exclude_seen, HIP_SYMBOL(cf_exclude_seen), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//		gpuErrchk(hipMemcpyFromSymbol(&xlim[0], HIP_SYMBOL(cf_xlim0), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//		gpuErrchk(hipMemcpyFromSymbol(&xlim[1], HIP_SYMBOL(cf_xlim1), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//		gpuErrchk(hipMemcpyFromSymbol(&ylim[0], HIP_SYMBOL(cf_ylim0), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//		gpuErrchk(hipMemcpyFromSymbol(&ylim[1], HIP_SYMBOL(cf_ylim1), sizeof(int),
//				0, hipMemcpyDeviceToHost));
//
//		/* Now calculate launch parameters, check exclude_seen, and launch  */
//		xspan = xlim[1] - xlim[0] + 1;
//		yspan = ylim[1] - ylim[0] + 1;
//		nThreads = xspan * yspan;
//
//		if (s != exclude_seen) {
//			BLK.x = floor((maxThreadsPerBlock-1+nThreads)/maxThreadsPerBlock);
//			THD.x = maxThreadsPerBlock;
//			cf_mark_pixels_seen_krnl<<<BLK,THD>>>(dpar, dmod,
//					nThreads, xspan);
//			checkErrorAfterKernelLaunch("cf_mark_pixels_krnl (calc_lghtcrv_cuda)");
//		}
//
//		/* Compute the model brightness for this model lightcurve point  */
//		brightness_temp = apply_photo_cuda(dmod, ddat, 0, s, i);
//		cf_compute_and_set_lghtcrv_brightness_krnl<<<1,1>>>(brightness_temp, i);
//		checkErrorAfterKernelLaunch("cf_compute_and_set_lghtcrv_brightness_krnl");
//	}
//
//
////	xspan = 2*pos_n + 1;
////	nThreads = xspan * xspan;
////	BLK.x = floor((maxThreadsPerBlock-1+nThreads)/maxThreadsPerBlock);
////	THD.x = maxThreadsPerBlock;
////	dbg_print_lghtcrv_pos_arrays(ddat, s, 22, nThreads, pos_n);
//
//
//	/* Now that we have calculated the model lightcurve brightnesses y at each
//	 * of the epochs x, we use cubic spline interpolation (Numerical Recipes
//	 * routines spline and splint) to get model lightcurve brightness fit[i] at
//	 * each OBSERVATION epoch t[i], with i=1,2,...,n. This will allow us (in
//	 * routine chi2) to compare model to data (fit[i] to obs[i]) to get chi-
//	 * square. Note that vector y2 contains the second derivatives of the
//	 * interpolating function at the calculation epochs x. Smearing is handled
//	 * by interpolating the brightness at the time t of each individual view
//	 * and then taking the mean of all views that correspond to a given
//	 * observed lightcurve point.                         */
//	/* Configure and launch an ncalc-threaded kernel that performs what NR
//	 * function spline does.  Original call:
//	 *
//	 * spline( lghtcrv->x, lghtcrv->y, ncalc, 2.0e30, 2.0e30, lghtcrv->y2);
//	 */
//
//	nThreads = ncalc;
//	double *u;
//	cudaCalloc((void**)&u, sizeof(double), ncalc);
//	int threads = 128;
//	BLK.x = floor((threads-1+nThreads)/threads);
//	THD.x = threads;
//	//cf_spline_lghtcrv_krnl<<<BLK,THD>>>(2.0e30, 2.0e30, u);
//	cf_spline_lghtcrv_serial_krnl<<<1,1>>>(u);
//	checkErrorAfterKernelLaunch("cf_spline_lghtcrv_krnl");
//
//	/* Start debug */
//	/* Pull out lghtcrv->x, lghtcrv->y, lghtcrv->y2 (all of length ncalc) */
//	//dbg_print_lghtcrv_xyy2(ddat, s, ncalc, "xyy2_arrays_CUDA.csv");
//
//
//	/* Launch n-threaded kernel to do the following:
//	 * 	- set each fit[i] = 0
//	 * 	- loop through all views and splint
//	 * 	- add interp to fit[i]
//	 * 	- divide final fit[i] over nviews
//	 */
//	BLK.x = floor((threads-1+n)/threads);
//	THD.x = threads;
//	cf_splint_lghtcrv_krnl<<<BLK,THD>>>(dpar);
//	checkErrorAfterKernelLaunch("cf_splint_lghtcrv_krnl");
//
//	hipFree(u);
//}

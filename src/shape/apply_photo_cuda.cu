#include "hip/hip_runtime.h"
/*****************************************************************************************
                                                                            apply_photo.c

For each plane-of-sky pixel, compute the model's scattered optical power per unit
projected (POS) area per unit solid angle per unit incident flux, and then sum these
values over the entire POS.  (The POS pixel area is multiplied in elsewhere.)

The expressions given here differ from the bidirectional reflectance functions defined by,
say, Hapke 1993: bidirectional reflectance includes an extra factor of
cos(scattering angle), since it is defined per unit surface area rather than per unit
projected area.

Modified 2014 February 12 by CM:
    Implement multiple optical scatering laws

Modified 2011 September 2 by CM:
    Add the "harmlambert" and "inholambert" optical scattering laws

Modified 2007 August 4 by CM:
    Add body parameter for use with the "orbit" action: it denotes which
        orbiting body's optical power contributions are being computed
        on this call to the routine
    Don't zero out blank-sky and shadowed POS pixels in the sky rendering
        (the pos->b matrix): do it instead in the calling routine by
        having it call the posclr routine.  This way apply_photo can be
        called twice for the "orbit" action, once for each orbiting body.
    Add comp matrix for POS frames

Modified 2006 October 1 by CM:
    Add "intensityfactor" parameter: account for POS pixel area,
        1 AU Sun-target distance, and solar apparent magnitude here
        rather than after calling the routine

Modified 2006 September 1 by CM and MCN:
    For inhomogeneous laws, add check that facet number pos->f[i][j]
        is nonnegative

Modified 2005 September 7 by CM:
    Implement the "harmlommel" "harmhapke" and "harmkaas" optical
        scattering laws

Modified 2005 August 8 by CM:
    Implement the "inhokaas" optical scattering law
    Add some (cosi > 0) checks
    Move "sum == 0" check to the end

Modified 2005 July 4 by CM:
    Changed structure name for the INHOLOMMEL optical scattering law

Modified 2005 March 1 by CM:
    Add NOLAW case

Modified 2005 January 25 by CM:
    Eliminate unused variables

Modified 2004 April 29 by CM:
    Modify Kaasalainen scattering law to use "wt" as the relative
        weighting factor (0 = pure Lommel-Seeliger, 1 = pure Lambert)
        rather than "c" (which ranged from 0 to infinity)

Modified 2004 March 25 by CM:
    hapke routine now takes phase rather than cos(phase) as argument

Modified 2004 February 29 by CM:
    Added comments
    Added Kaasalainen "Lommel-Seeliger + Lambert" scattering law
    Eliminated "type" argument, since this routine was only being
       used to handle optical scattering.  (Radar scattering is
       instead handled by the "radlaw" routine.)
    Added "phase" argument (solar phase angle) so that we can compute
       the phase just once per calculated lightcurve point (in read_dat)
       rather than computing it every time we call apply_photo
*****************************************************************************************/
extern "C" {
#include "head.h"
}

#define TINY 1.0e-40

__device__ struct pos_t *ap_pos;
__device__ unsigned char *ap_type;
__device__ int ap_ilaw, ap_posn;
__device__ int4 ap_xylim;
__device__ double phasefunc, scale_lommsee, scale_lambert, intensityfactor, phase;
__device__ float sum, phasefuncf, scale_lommseef, scale_lambertf;
__device__ double dblsum;

__global__ void ap_get_pos_krnl(struct dat_t *ddat, struct mod_t *dmod,
		int set, int frm, unsigned char *type) {
	/* Single-thread kernel */
	if (threadIdx.x == 0) {
		ap_pos = &ddat->set[set].desc.lghtcrv.rend[frm].pos;
		ap_ilaw = ddat->set[set].desc.lghtcrv.ioptlaw;
		//ap_type = &dmod->photo.opttype[ap_ilaw];
		type[0] = dmod->photo.opttype[ap_ilaw];
		ap_posn = ap_pos->n;
		sum = 0.0;
		intensityfactor = (ap_pos->km_per_pixel/AU) * (ap_pos->km_per_pixel/AU);
		phase = ddat->set[set].desc.lghtcrv.solar_phase[frm];
		ap_xylim.w = ap_pos->xlim[0];
		ap_xylim.x = ap_pos->xlim[1];
		ap_xylim.y = ap_pos->ylim[0];
		ap_xylim.z = ap_pos->ylim[1];
	}
}
__global__ void ap_init_streams_krnl(
		struct dat_t *ddat,
		struct mod_t *dmod,
		struct pos_t **pos,
		int set, int nframes, unsigned char *type, float *dsum,
		double *intensity_factor,
		double *phase_d) {
	/* nframes-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (f <= nframes) {
		ap_ilaw = ddat->set[set].desc.lghtcrv.ioptlaw;
		//ap_type = &dmod->photo.opttype[ap_ilaw];
		type[0] = dmod->photo.opttype[ap_ilaw];
		dsum[f] = 0.0;
		intensity_factor[f] = (pos[f]->km_per_pixel/AU) * (pos[f]->km_per_pixel/AU);
		phase_d[f] = ddat->set[set].desc.lghtcrv.solar_phase[f];
	}
}
__global__ void ap_init_streams_f_krnl(
		struct dat_t *ddat,
		struct mod_t *dmod,
		struct pos_t **pos,
		int set, int nframes, unsigned char *type, float *dsum,
		float *intensity_factor, float *phase_f) {
	/* nframes-threaded kernel */
	int f = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (f <= nframes) {
		ap_ilaw = ddat->set[set].desc.lghtcrv.ioptlaw;
		type[0] = dmod->photo.opttype[ap_ilaw];
		dsum[f] = 0.0;
		intensity_factor[f] = (__double2float_rn(pos[f]->km_per_pixel)/AU) *
				(__double2float_rn(pos[f]->km_per_pixel)/AU);
		phase_f[f] = __double2float_rn(ddat->set[set].desc.lghtcrv.solar_phase[f]);
	}
}
__global__ void ap_lambertlaw_krnl(struct mod_t *dmod, int nThreads,
		int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	double scale;
	float b;

	if (offset < nThreads) {
		scale = dmod->photo.optical[ap_ilaw].R.R.val/PIE;

		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
				&& ap_pos->body[i][j] == body) {
			ap_pos->b[i][j] = intensityfactor * scale * ap_pos->cosi[i][j];
			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_lambertlaw_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int4 *xylim,
		int nThreads,
		int body,
		int2 span,
		float *dsum,
		int f) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[f].w;
	int j = offset / span.x + xylim[f].y;
	double scale;

	if (offset < nThreads) {
		scale = dmod->photo.optical[ap_ilaw].R.R.val/PIE;

		if (pos[f]->cose_s[offset] > 0.0 && pos[f]->cosi_s[offset] > 0.0
				&& pos[f]->body[i][j] == body) {
			pos[f]->b_s[offset] = intensityfactor * scale * pos[f]->cosi_s[offset];
			atomicAdd(&dsum[f], pos[f]->b_s[offset]);
		}
	}
}
__global__ void ap_lambertlaw_streams_f_krnl(struct mod_t *dmod, struct pos_t **pos,
		int4 *xylim, int nThreads, int body, int2 span, float *dsum, int f) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[f].w;
	int j = offset / span.x + xylim[f].y;
	float scale;

	if (offset < nThreads) {
		scale = __double2float_rn(dmod->photo.optical[ap_ilaw].R.R.val)/PIE;

		if (pos[f]->cose_s[offset] > 0.0 && pos[f]->cosi_s[offset] > 0.0
				&& pos[f]->body[i][j] == body) {
			pos[f]->b_s[offset] = intensityfactor * scale * pos[f]->cosi_s[offset];
			atomicAdd(&dsum[f], pos[f]->b_s[offset]);
		}
	}
}
__global__ void ap_harmlambert_krnl(struct mod_t *dmod, int nThreads, int body,
		int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	int c, f;
	double scale;
	float b;

	if (offset < nThreads) {

		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body && ap_pos->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val/PIE;
			ap_pos->b[i][j] = intensityfactor * scale * ap_pos->cosi[i][j];
			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_harmlambert_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	double scale;

	if (offset < nThreads) {

		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val/PIE;
			pos[frm]->b_s[offset] = intensity_factor[frm] * scale *
					pos[frm]->cosi_s[offset];
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_harmlambert_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	float scale;

	if (offset < nThreads) {

		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = __double2float_rn(dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val)/PIE;
			pos[frm]->b_s[offset] = intensity_factor[frm] * scale *
					pos[frm]->cosi_s[offset];
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_inholambert_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	int c, f;
	double scale;
	float b;

	if (offset < nThreads) {

		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body && ap_pos->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val/PIE;
			ap_pos->b[i][j] = intensityfactor * scale * ap_pos->cosi[i][j];
			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_inholambert_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	double scale;

	if (offset < nThreads) {

		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val/PIE;
			pos[frm]->b_s[offset] = intensity_factor[frm] * scale * pos[frm]->cosi_s[offset];
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_inholambert_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	float scale;

	if (offset < nThreads) {

		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = __double2float_rn(dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val)/PIE;
			pos[frm]->b_s[offset] = intensity_factor[frm] * scale * pos[frm]->cosi_s[offset];
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_lommel_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	double scale;
	float b;

	if (offset < nThreads) {
		scale = dmod->photo.optical[ap_ilaw].R.R.val/(4*PIE);
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
				&& ap_pos->body[i][j] == body) {
			ap_pos->b[i][j] = intensityfactor * scale * ap_pos->cosi[i][j]
			  / (ap_pos->cosi[i][j] + ap_pos->cose[i][j]);
			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_lommel_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	double scale;

	if (offset < nThreads) {
		scale = dmod->photo.optical[ap_ilaw].R.R.val/(4*PIE);
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
				&& pos[frm]->body[i][j] == body) {
			pos[frm]->b_s[offset] = intensityfactor * scale * pos[frm]->cosi_s[offset]
			  / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_lommel_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	float scale;

	if (offset < nThreads) {
		scale = __double2float_rn(dmod->photo.optical[ap_ilaw].R.R.val)/(4*PIE);
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
				&& pos[frm]->body[i][j] == body) {
			pos[frm]->b_s[offset] = intensityfactor * scale * pos[frm]->cosi_s[offset]
			  / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_harmlommel_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	int c, f;
	double scale;
	float b;

	if (offset < nThreads) {
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body && ap_pos->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val/(4*PIE);
			ap_pos->b[i][j] = intensityfactor * scale * ap_pos->cosi[i][j]
			   / (ap_pos->cosi[i][j] + ap_pos->cose[i][j]);
			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_harmlommel_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		int frm) {

	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	double scale;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val/(4*PIE);
			pos[frm]->b_s[offset] = intensityfactor * scale * pos[frm]->cosi_s[offset]
			   / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_harmlommel_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		int frm) {

	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	float scale;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = __double2float_rn(dmod->photo.optical[ap_ilaw].harmR.local[c][f].R.val)/(4*PIE);
			pos[frm]->b_s[offset] = intensityfactor * scale * pos[frm]->cosi_s[offset]
			   / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_inholommel_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	int c, f;
	double scale;
	float b;

	if (offset < nThreads) {
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body && ap_pos->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val/(4*PIE);
			ap_pos->b[i][j] = intensityfactor * scale * ap_pos->cosi[i][j]
			   / (ap_pos->cosi[i][j] + ap_pos->cose[i][j]);
			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_inholommel_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	double scale;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val/(4*PIE);
			pos[frm]->b_s[offset] = intensityfactor * scale * pos[frm]->cosi_s[offset]
			   / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_inholommel_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	float scale;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			scale = __double2float_rn(dmod->photo.optical[ap_ilaw].inhoR.local[c][f].R.val)/(4*PIE);
			pos[frm]->b_s[offset] = intensityfactor * scale * pos[frm]->cosi_s[offset]
			   / (pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]);
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_geometrical_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	float b;

	if (offset < nThreads) {
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body) {
			ap_pos->b[i][j] = intensityfactor * dmod->photo.optical[ap_ilaw].R.R.val;
			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_geometrical_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b_s[offset] = intensityfactor * dmod->photo.optical[ap_ilaw].R.R.val;
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_geometrical_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b_s[offset] = intensityfactor * dmod->photo.optical[ap_ilaw].R.R.val;
			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_hapke_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	float b;

	if (offset < nThreads) {
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body) {
			ap_pos->b[i][j] = intensityfactor
					* dev_hapke(ap_pos->cosi[i][j], ap_pos->cose[i][j],
							phase,
							dmod->photo.optical[ap_ilaw].hapke.w.val,
							dmod->photo.optical[ap_ilaw].hapke.h.val,
							dmod->photo.optical[ap_ilaw].hapke.B0.val,
							dmod->photo.optical[ap_ilaw].hapke.g.val,
							dmod->photo.optical[ap_ilaw].hapke.theta.val);

			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_hapke_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		double *phase_d,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b_s[offset] = intensityfactor
					* dev_hapke(pos[frm]->cosi_s[offset], pos[frm]->cose_s[offset],
							phase_d[frm],
							dmod->photo.optical[ap_ilaw].hapke.w.val,
							dmod->photo.optical[ap_ilaw].hapke.h.val,
							dmod->photo.optical[ap_ilaw].hapke.B0.val,
							dmod->photo.optical[ap_ilaw].hapke.g.val,
							dmod->photo.optical[ap_ilaw].hapke.theta.val);

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_hapke_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		float *phase_f,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b_s[offset] = intensity_factor[frm]
					* dev_hapke_f(__double2float_rn(pos[frm]->cosi_s[offset]),
							__double2float_rn(pos[frm]->cose_s[offset]),
							phase_f[frm],
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.w.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.h.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.B0.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.g.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].hapke.theta.val));

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_harmhapke_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	int c, f;
	float b;

	if (offset < nThreads) {
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
	     && ap_pos->body[i][j] == body && ap_pos->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			ap_pos->b[i][j] = intensityfactor
					* dev_hapke(ap_pos->cosi[i][j], ap_pos->cose[i][j],
							phase,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].w.val,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].h.val,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].B0.val,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].g.val,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].theta.val);

			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_harmhapke_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		double *phase_d,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
	     && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			pos[frm]->b_s[offset] = intensity_factor[frm]
					* dev_hapke(pos[frm]->cosi_s[offset], pos[frm]->cose_s[offset],
							phase,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].w.val,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].h.val,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].B0.val,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].g.val,
							dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].theta.val);

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_harmhapke_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		float *phase_f,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
	     && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			pos[frm]->b_s[offset] = intensity_factor[frm]
			            *dev_hapke_f(pos[frm]->cosi_s[offset], pos[frm]->cose_s[offset],
			            phase_f[frm],
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].w.val),
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].h.val),
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].B0.val),
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].g.val),
			            __double2float_rn(dmod->photo.optical[ap_ilaw].harmhapke.local[c][f].theta.val));

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_inhohapke_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	int c, f;
	float b;

	if (offset < nThreads) {
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body && ap_pos->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			ap_pos->b[i][j] = intensityfactor
					* dev_hapke(ap_pos->cosi[i][j], ap_pos->cose[i][j],
							phase,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].w.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].h.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].B0.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].g.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].theta.val);

			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_inhohapke_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		double *phase_d,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	float b;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			pos[frm]->b_s[offset] = intensity_factor[frm]
					* dev_hapke(pos[frm]->cosi_s[offset], pos[frm]->cose_s[offset],
							phase_d[frm],
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].w.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].h.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].B0.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].g.val,
							dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].theta.val);

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_inhohapke_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		float *phase_f,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			pos[frm]->b_s[offset] = intensity_factor[frm]
					* dev_hapke_f(pos[frm]->cosi_s[offset], pos[frm]->cose_s[offset],
							phase_f[frm],
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].w.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].h.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].B0.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].g.val),
							__double2float_rn(dmod->photo.optical[ap_ilaw].inhohapke.local[c][f].theta.val));

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_kaas_init_krnl(struct mod_t *dmod) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		phasefunc = dmod->photo.optical[ap_ilaw].kaas.A0.val
				* exp( -phase / dmod->photo.optical[ap_ilaw].kaas.D.val)
		+ dmod->photo.optical[ap_ilaw].kaas.k.val * phase + 1;

		scale_lommsee = (1 - dmod->photo.optical[ap_ilaw].kaas.wt.val)
		 	   * phasefunc * dmod->photo.optical[ap_ilaw].kaas.R.val/(4*PIE);
		scale_lambert = dmod->photo.optical[ap_ilaw].kaas.wt.val
			   * phasefunc * dmod->photo.optical[ap_ilaw].kaas.R.val/PIE;
	}
}
__global__ void ap_kaas_streams2_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		double *phase_d,
		double *phasefuncd,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int n = pos[frm]->n;
	int pos_spn = 2*n+1;
	int pxa = (j+n)*pos_spn + (i+n);

	if (threadIdx.x == 0) {
		phasefuncd[frm] = dmod->photo.optical[ap_ilaw].kaas.A0.val
				* exp( -phase_d[frm] / dmod->photo.optical[ap_ilaw].kaas.D.val)
		+ dmod->photo.optical[ap_ilaw].kaas.k.val * phase_d[frm] + 1;
	}
	__syncthreads();

	if (offset == 0 && frm == 0) {
		scale_lommsee = (1 - dmod->photo.optical[ap_ilaw].kaas.wt.val)
		 	   * phasefuncd[frm] * dmod->photo.optical[ap_ilaw].kaas.R.val/(4*PIE);
		scale_lambert = dmod->photo.optical[ap_ilaw].kaas.wt.val
			   * phasefuncd[frm] * dmod->photo.optical[ap_ilaw].kaas.R.val/PIE;
	}

	if (offset < nThreads) {
		if (pos[frm]->cose_s[pxa] > 0.0 && pos[frm]->cosi_s[pxa] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b_s[pxa] = intensity_factor[frm] * pos[frm]->cosi_s[pxa]
			    *(scale_lommsee / (pos[frm]->cosi_s[pxa] + pos[frm]->cose_s[pxa])
			    + scale_lambert);
			atomicAdd(&sum, pos[frm]->b_s[pxa]);
		}
	}
}
__global__ void ap_kaas_streams2_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		float *phase_f,
		float *phasefuncf,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int n = pos[frm]->n;
	int pos_spn = 2*n+1;
	int pxa = (j+n)*pos_spn + (i+n);

	if (threadIdx.x == 0) {
		phasefuncf[frm] = __double2float_rn(dmod->photo.optical[ap_ilaw].kaas.A0.val)
				* exp( -phase_f[frm] / __double2float_rn(dmod->photo.optical[ap_ilaw].kaas.D.val))
		+ __double2float_rn(dmod->photo.optical[ap_ilaw].kaas.k.val) * phase_f[frm] + 1;
	}
	__syncthreads();

	if (offset == 0 && frm == 0) {
		scale_lommseef = (1 - __double2float_rn(dmod->photo.optical[ap_ilaw].kaas.wt.val))
		 	   * phasefuncf[frm] * __double2float_rn(dmod->photo.optical[ap_ilaw].kaas.R.val)/(4*PIE);
		scale_lambertf = __double2float_rn(dmod->photo.optical[ap_ilaw].kaas.wt.val)
			   * phasefuncf[frm] * __double2float_rn(dmod->photo.optical[ap_ilaw].kaas.R.val)/PIE;
	}

	if (offset < nThreads) {
		if (pos[frm]->cose_s[pxa] > 0.0 && pos[frm]->cosi_s[pxa] > 0.0
		 && pos[frm]->body[i][j] == body) {
			pos[frm]->b_s[pxa] = intensity_factor[frm] * pos[frm]->cosi_s[pxa]
			    *(scale_lommseef / (pos[frm]->cosi_s[pxa] + pos[frm]->cose_s[pxa])
			    + scale_lambertf);
			atomicAdd(&sum, pos[frm]->b_s[pxa]);
		}
	}
}
__global__ void ap_kaas_streams_krnl(struct mod_t *dmod, struct pos_t **pos,
		int nThreads, int body, int2 span, int4 *xylim, float *dsum, int f) {
	/* Multi-threaded kernel
	 * This one is deprecated.  Don't use*/
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[f].w;
	int j = offset / span.x + xylim[f].y;
	int n = pos[f]->n;
	int pos_spn = 2*n+1;
	int pxa = (j+n)*pos_spn + (i+n);

	if (offset < nThreads) {
		if (pos[f]->cose_s[pxa] > 0.0 && pos[f]->cosi_s[pxa] > 0.0
		 && pos[f]->body[i][j] == body) {
			pos[f]->b_s[pxa] = intensityfactor * pos[f]->cosi_s[pxa]
			    *(scale_lommsee / (pos[f]->cosi_s[pxa] + pos[f]->cose_s[pxa])
			    + scale_lambert);
			atomicAdd(&dsum[f], pos[f]->b_s[pxa]);
		}
	}
}
__global__ void ap_harmkaas_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
	int j = offset / span.x + ap_xylim.y;
	int c, f;
	float b;

	if (offset < nThreads) {
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body && ap_pos->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			phasefunc = dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].A0.val
			* exp( -phase / dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].D.val)
			+ dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].k.val * phase + 1;

			scale_lommsee = (1 - dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val)
		    * phasefunc * dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val / (4*PIE);
			scale_lambert = dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val
			* phasefunc * dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val / PIE;
			ap_pos->b[i][j] = intensityfactor * ap_pos->cosi[i][j] * (scale_lommsee /
					(ap_pos->cosi[i][j] + ap_pos->cose[i][j]) + scale_lambert);

			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_harmkaas_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		double *phase_d,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	double phasefuncd;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			phasefuncd = dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].A0.val
			* exp( -phase_d[frm] / dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].D.val)
			+ dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].k.val * phase_d[frm] + 1;

			scale_lommsee = (1 - dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val)
		    * phasefuncd * dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val / (4*PIE);
			scale_lambert = dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val
			* phasefuncd * dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val / PIE;
			 pos[frm]->b_s[offset] = intensity_factor[frm] * pos[frm]->cosi_s[offset] * (scale_lommsee /
					( pos[frm]->cosi_s[offset] +  pos[frm]->cose_s[offset]) + scale_lambert);

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_harmkaas_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		float *phase_f,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	float phasefuncf;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			phasefuncf = __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].A0.val)
			* exp( -phase_f[frm] / __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].D.val))
			+ __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].k.val) * phase_f[frm] + 1;

			scale_lommseef = (1 - __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val))
		    * phasefuncf * __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val) / (4*PIE);
			scale_lambertf = __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].wt.val)
			* phasefuncf * __double2float_rn(dmod->photo.optical[ap_ilaw].harmkaas.local[c][f].R.val) / PIE;
			 pos[frm]->b_s[offset] = intensity_factor[frm] * pos[frm]->cosi_s[offset] * (scale_lommseef /
					( pos[frm]->cosi_s[offset] +  pos[frm]->cose_s[offset]) + scale_lambertf);

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_inhokaas_krnl(struct mod_t *dmod, int nThreads, int body, int2 span) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + ap_xylim.w;
		int j = offset / span.x + ap_xylim.y;
	int c, f;
	float b;

	if (offset < nThreads) {
		if (ap_pos->cose[i][j] > 0.0 && ap_pos->cosi[i][j] > 0.0
		 && ap_pos->body[i][j] == body && ap_pos->f[i][j] >= 0) {
			c = ap_pos->comp[i][j];
			f = ap_pos->f[i][j];
			phasefunc = dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].A0.val
			* exp( -phase / dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].D.val)
			+ dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].k.val * phase + 1;
			scale_lommsee = (1 - dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val)
		    * phasefunc * dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val / (4*PIE);
			scale_lambert = dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val
			* phasefunc * dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val / PIE;
			ap_pos->b[i][j] = intensityfactor * ap_pos->cosi[i][j] * (scale_lommsee /
					(ap_pos->cosi[i][j] + ap_pos->cose[i][j]) + scale_lambert);

			b = __double2float_rd(ap_pos->b[i][j]);
			atomicAdd(&sum, b);
		}
	}
}
__global__ void ap_inhokaas_streams_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		double *intensity_factor,
		double *phase_d,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	double phasefuncd;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			phasefuncd = dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].A0.val
			* exp( -phase_d[frm] / dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].D.val)
			+ dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].k.val * phase_d[frm] + 1;
			scale_lommsee = (1 - dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val)
		    * phasefuncd * dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val / (4*PIE);
			scale_lambert = dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val
			* phasefuncf * dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val / PIE;
			pos[frm]->b_s[offset] = intensity_factor[frm] * pos[frm]->cosi_s[offset] * (scale_lommsee /
					(pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]) + scale_lambert);

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_inhokaas_streams_f_krnl(
		struct mod_t *dmod,
		struct pos_t **pos,
		int nThreads,
		int body,
		int4 *xylim,
		int2 span,
		float *dsum,
		float *intensity_factor,
		float *phase_f,
		int frm) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % span.x + xylim[frm].w;
	int j = offset / span.x + xylim[frm].y;
	int c, f;
	float phasefuncf;

	if (offset < nThreads) {
		if (pos[frm]->cose_s[offset] > 0.0 && pos[frm]->cosi_s[offset] > 0.0
		 && pos[frm]->body[i][j] == body && pos[frm]->f[i][j] >= 0) {
			c = pos[frm]->comp[i][j];
			f = pos[frm]->f[i][j];
			phasefuncf = __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].A0.val)
			* exp( -phase_f[frm] / __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].D.val))
			+ __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].k.val) * phase_f[frm] + 1;
			scale_lommseef = (1 - __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val))
		    * phasefuncf * __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val) / (4*PIE);
			scale_lambertf = __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].wt.val)
			* phasefuncf * __double2float_rn(dmod->photo.optical[ap_ilaw].inhokaas.local[c][f].R.val) / PIE;
			pos[frm]->b_s[offset] = intensity_factor[frm] * pos[frm]->cosi_s[offset] * (scale_lommseef /
					(pos[frm]->cosi_s[offset] + pos[frm]->cose_s[offset]) + scale_lambertf);

			atomicAdd(&sum, pos[frm]->b_s[offset]);
		}
	}
}
__global__ void ap_get_sum_krnl() {
	/* Single-threaded kernel */
	/* Nothing really needs to be done, but we need a kernel that we can
	 * follow with copying the variable out  */
	if (threadIdx.x == 0)
		if (sum == 0)
			sum = TINY; //printf("\nsum =0!\n");
}
__global__ void ap_set_lghtcrv_y_streams2_krnl(struct dat_t *ddat, int s, float *dsum,
		int size) {
	/* Single-threaded kernel */
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		ddat->set[s].desc.lghtcrv.y[i] = dsum[i+1];
	}
}
__host__ double apply_photo_cuda(struct mod_t *dmod, struct dat_t *ddat, int body,
		int set, int frm)
{
	unsigned char *type;
	int n, nThreads;
	float hsum;
	dim3 BLK, THD;
	int4 xylim;
	int2 span;

	cudaCalloc1((void**)&type, sizeof(unsigned char), 2);
	/* Launch single-thread kernel to assign pos address and get type */
	ap_get_pos_krnl<<<1,1>>>(ddat, dmod, set, frm, type);
	checkErrorAfterKernelLaunch("ap_get_pos_krnl");
	deviceSyncAfterKernelLaunch("ap_get_pos_krnl");

//	gpuErrchk(hipMemcpyFromSymbol(&ilaw, HIP_SYMBOL(ap_ilaw), sizeof(ilaw), 0,
//			hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&n, HIP_SYMBOL(ap_posn),	sizeof(n), 0,
			hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&xylim, HIP_SYMBOL(ap_xylim), sizeof(int4), 0,
			hipMemcpyDeviceToHost));

	/* Calculate launch parameters for the pixel kernels */
	span.x = xylim.x - xylim.w + 1;
	span.y = xylim.z - xylim.y + 1;
	nThreads = span.x * span.y;
 	BLK.x = floor((maxThreadsPerBlock-1+nThreads)/maxThreadsPerBlock);
	THD.x = maxThreadsPerBlock; // Thread block dimensions

	switch (type[0]) {
	case LAMBERTLAW:
		/* Launch Lambert Law kernel */
		ap_lambertlaw_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
		checkErrorAfterKernelLaunch("ap_lambertlaw_krnl, line ");
		break;
	case HARMLAMBERT:
		/* Launch the HarmLambert kernel */
		ap_harmlambert_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
		checkErrorAfterKernelLaunch("ap_harmlambert_krnl, line ");
		break;
	case INHOLAMBERT:
		/* Launch the Inhomogeneous Lambert kernel */
		ap_inholambert_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
		checkErrorAfterKernelLaunch("ap_inholambert_krnl, line ");
		break;
	case LOMMEL:
		/* Launch the Lommel kernel */
		ap_lommel_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
		checkErrorAfterKernelLaunch("ap_lommel_krnl, line ");
		break;
	case HARMLOMMEL:
		/* Launch the HarmLommel kernel */
		ap_harmlommel_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
		checkErrorAfterKernelLaunch("ap_harmlommel_krnl, line ");
		break;
	case INHOLOMMEL:
		/* Launch the Inhomogeneous Lommel kernel */
		ap_harmlommel_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
		checkErrorAfterKernelLaunch("ap_inholommel_krnl, line ");
      break;
  case GEOMETRICAL:
	  /* Launch the Geometrical law kernel */
	  ap_geometrical_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
	  checkErrorAfterKernelLaunch("ap_geometrical_krnl, line ");
	  break;
  case HAPKE:
	  /* Launch the Hapke kernel */
	  ap_hapke_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
	  checkErrorAfterKernelLaunch("ap_hapke_krnl, line ");
      break;
  case HARMHAPKE:
	  /* Launch the HarmHapke kernel */
	  ap_harmhapke_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
	  checkErrorAfterKernelLaunch("ap_harmhapke_krnl, line ");
      break;
  case INHOHAPKE:
	  /* Launch the Inhomogeneous Hapke kernel */
	  ap_inhohapke_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
	  checkErrorAfterKernelLaunch("ap_inhohapke_krnl, line ");
	  break;
  case KAASALAINEN:
      /* Launch single-thread kernel to init Kaas */
	  ap_kaas_init_krnl<<<1,1>>>(dmod);
	  checkErrorAfterKernelLaunch("ap_kaas_init_krnl, line ");

	  /* Launch the main Kaasalainen kernel */
	  //ap_kaas_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
	  //checkErrorAfterKernelLaunch("ap_kaas_krnl, line ");
      break;
  case HARMKAAS:
	  /* Launch the HarmKaas kernel */
	  ap_harmkaas_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
	  checkErrorAfterKernelLaunch("ap_harmkaas_krnl, line ");
	  break;
  case INHOKAAS:
	  /* Launch the HarmKaas kernel */
	  ap_inhokaas_krnl<<<BLK,THD>>>(dmod, nThreads, body, span);
	  checkErrorAfterKernelLaunch("ap_inhokaas_krnl, line ");
	  break;
  case NOLAW:
	  bailout("apply_photo.c: can't set optical scattering law = \"none\" when optical data are used\n");
	  break;
  default:
	  bailout("apply_photo.c: can't handle that optical scattering law yet\n");
	}

	/* Launch single kernel to retrieve sum */
	ap_get_sum_krnl<<<1,1>>>();
	checkErrorAfterKernelLaunch("ap_get_sum_krnl, line ");
	gpuErrchk(hipMemcpyFromSymbol(&hsum, HIP_SYMBOL(sum), sizeof(sum), 0,
			hipMemcpyDeviceToHost));
	if (hsum == 0.0)
		hsum = TINY;
	//hipFree(type);
	return (double)hsum;
}

__host__ void apply_photo_cuda_streams(
		struct mod_t *dmod,
		struct dat_t *ddat,
		struct pos_t **pos,
		int4 *xylim,
		int2 *span,
		dim3 *BLKpx,
		int *nThreads,
		int body,
		int set,
		int nframes,
		hipStream_t *ap_stream)
{
	unsigned char *type, *htype;
	int f;
	float *hsum, *dsum;
	double *intensity_factor, *phase_d, *phasefuncd;
	dim3 BLK, THD;

	gpuErrchk(hipMalloc((void**)&type, sizeof(unsigned char) * 2));
	gpuErrchk(hipMalloc((void**)&dsum, sizeof(float)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&intensity_factor, sizeof(double)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&phase_d, sizeof(double)*(nframes+1)));
	htype = (unsigned char *) malloc(2*sizeof(unsigned char));
	hsum = (float *) malloc((nframes+1)*sizeof(float));

	/* Launch single-thread kernel to assign pos address and get type */
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + nframes) / THD.x);
	ap_init_streams_krnl<<<BLK,THD>>>(ddat, dmod, pos, set, nframes, type, dsum,
			intensity_factor, phase_d);
	checkErrorAfterKernelLaunch("ap_init_streams_krnl");

	gpuErrchk(hipMemcpy(htype, type, sizeof(unsigned char) *2,
			hipMemcpyDeviceToHost));

	for (f=0;f<=nframes;f++)
		hsum[f]=0.0;

	switch (htype[0]) {
	case LAMBERTLAW:
		/* Launch Lambert Law kernel */
		for (f=1; f<=nframes; f++)
			ap_lambertlaw_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,pos,
					xylim, nThreads[f], body, span[f], dsum, f);
		checkErrorAfterKernelLaunch("ap_lambertlaw_streams_krnl");
		break;
	case HARMLAMBERT:
		/* Launch the HarmLambert kernel */
		for (f=1; f<=nframes; f++)
			ap_harmlambert_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(
					dmod, pos, nThreads[f], body, xylim, span[f], dsum,
					intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_harmlambert_streams_krnl");
		break;
	case INHOLAMBERT:
		/* Launch the Inhomogeneous Lambert kernel */
		for (f=1; f<=nframes; f++)
			ap_inholambert_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor,f);
		checkErrorAfterKernelLaunch("ap_inholambert_streams_krnl");
		break;
	case LOMMEL:
		/* Launch the Lommel kernel */
		for (f=1; f<=nframes; f++)
			ap_lommel_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], dsum, intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_lommel_streams_krnl");
		break;
	case HARMLOMMEL:
		/* Launch the HarmLommel kernel */
		for (f=1; f<=nframes; f++)
			ap_harmlommel_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_harmlommel_streams_krnl");
		break;
	case INHOLOMMEL:
		/* Launch the Inhomogeneous Lommel kernel */
		for (f=1; f<=nframes; f++)
			ap_inholommel_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_inholommel_streams_krnl");
		break;
	case GEOMETRICAL:
		/* Launch the Geometrical law kernel */
		for (f=1; f<=nframes; f++)
			ap_geometrical_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum,
					intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_geometrical_streams_krnl");
		break;
	case HAPKE:
		/* Launch the Hapke kernel */
		for (f=1; f<=nframes; f++)
			ap_hapke_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_d, f);
		checkErrorAfterKernelLaunch("ap_hapke_streams_krnl");
		break;
	case HARMHAPKE:
		/* Launch the HarmHapke kernel */
		for (f=1; f<=nframes; f++)
			ap_harmhapke_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum,
					intensity_factor, phase_d, f);
		checkErrorAfterKernelLaunch("ap_harmhapke_streams_krnl");
		break;
	case INHOHAPKE:
		/* Launch the Inhomogeneous Hapke kernel */
		for (f=1; f<=nframes; f++)
			ap_inhohapke_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_d, f);
		checkErrorAfterKernelLaunch("ap_inhohapke_streams_krnl");
		break;
	case KAASALAINEN:
		/* Launch single-thread kernel to init Kaas */
//		ap_kaas_init_krnl<<<1,1>>>(dmod);
//		checkErrorAfterKernelLaunch("ap_kaas_init_krnl");
		gpuErrchk(hipMalloc((void**)&phasefuncd, sizeof(double)*(nframes+1)));
		/* Launch the main Kaasalainen kernel */
		for (f=1; f<=nframes; f++){
//			ap_kaas_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
//					nThreads[f], body, span[f], xylim, dsum, f);
			ap_kaas_streams2_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_d, phasefuncd, f);
			//hipDeviceSynchronize();
		}
		checkErrorAfterKernelLaunch("ap_kaas_streams_krnl");
		hipFree(phasefuncd);
		break;
	case HARMKAAS:
		/* Launch the HarmKaas kernel */
		for (f=1; f<=nframes; f++)
			ap_harmkaas_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_d, f);
		checkErrorAfterKernelLaunch("ap_harmkaas_streams_krnl");
		break;
	case INHOKAAS:
		/* Launch the HarmKaas kernel */
		for (f=1; f<=nframes; f++)
			ap_inhokaas_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_d, f);
		checkErrorAfterKernelLaunch("ap_inhokaas_streams_krnl");
		break;
	case NOLAW:
		bailout("apply_photo.c: can't set optical scattering law = \"none\" when optical data are used\n");
		break;
	default:
		bailout("apply_photo.c: can't handle that optical scattering law yet\n");
	}
	/* Now set the lghtcrv->y values with what was calculated here */
	BLK.x = floor((THD.x - 1 + (nframes+1)) / THD.x);
	ap_set_lghtcrv_y_streams2_krnl<<<BLK,THD>>>(ddat, set, dsum, (nframes+1));
	checkErrorAfterKernelLaunch("ap_set_lghtcrv_y_streams2_krnl");
	gpuErrchk(hipMemcpy(hsum, dsum, sizeof(float)*(nframes+1),
			hipMemcpyDeviceToHost));

	hipFree(dsum);
	hipFree(type);
	free(htype);
	free(hsum);
}
__host__ void apply_photo_cuda_streams_f(
		struct mod_t *dmod,
		struct dat_t *ddat,
		struct pos_t **pos,
		int4 *xylim,
		int2 *span,
		dim3 *BLKpx,
		int *nThreads,
		int body,
		int set,
		int nframes,
		hipStream_t *ap_stream)
{
	unsigned char *type, *htype;
	int f;
	float *hsum, *dsum;
	float *intensity_factor, *phase_f, *phasefuncf;
	dim3 BLK, THD;

	gpuErrchk(hipMalloc((void**)&type, sizeof(unsigned char) * 2));
	gpuErrchk(hipMalloc((void**)&dsum, sizeof(float)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&intensity_factor, sizeof(float)*(nframes+1)));
	gpuErrchk(hipMalloc((void**)&phase_f, sizeof(float)*(nframes+1)));
	htype = (unsigned char *) malloc(2*sizeof(unsigned char));
	hsum = (float *) malloc((nframes+1)*sizeof(float));

	/* Launch single-thread kernel to assign pos address and get type */
	THD.x = maxThreadsPerBlock;
	BLK.x = floor((THD.x - 1 + nframes) / THD.x);
	ap_init_streams_f_krnl<<<BLK,THD>>>(ddat, dmod, pos, set, nframes, type, dsum,
			intensity_factor, phase_f);
	checkErrorAfterKernelLaunch("ap_init_streams_f_krnl");
	gpuErrchk(hipMemcpy(htype, type, sizeof(unsigned char) *2,
			hipMemcpyDeviceToHost));

	for (f=0;f<=nframes;f++)
		hsum[f]=0.0;

	switch (htype[0]) {
	case LAMBERTLAW:
		/* Launch Lambert Law kernel */
		for (f=1; f<=nframes; f++)
			ap_lambertlaw_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,pos,
					xylim, nThreads[f], body, span[f], dsum, f);
		checkErrorAfterKernelLaunch("ap_lambertlaw_streams_krnl");
		break;
	case HARMLAMBERT:
		/* Launch the HarmLambert kernel */
		for (f=1; f<=nframes; f++)
			ap_harmlambert_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(
					dmod, pos, nThreads[f], body, xylim, span[f], dsum,
					intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_harmlambert_streams_f_krnl");
		break;
	case INHOLAMBERT:
		/* Launch the Inhomogeneous Lambert kernel */
		for (f=1; f<=nframes; f++)
			ap_inholambert_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor,f);
		checkErrorAfterKernelLaunch("ap_inholambert_streams_f_krnl");
		break;
	case LOMMEL:
		/* Launch the Lommel kernel */
		for (f=1; f<=nframes; f++)
			ap_lommel_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], dsum, intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_lommel_streams_f_krnl");
		break;
	case HARMLOMMEL:
		/* Launch the HarmLommel kernel */
		for (f=1; f<=nframes; f++)
			ap_harmlommel_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_harmlommel_streams_f_krnl");
		break;
	case INHOLOMMEL:
		/* Launch the Inhomogeneous Lommel kernel */
		for (f=1; f<=nframes; f++)
			ap_inholommel_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_inholommel_streams_f_krnl");
		break;
	case GEOMETRICAL:
		/* Launch the Geometrical law kernel */
		for (f=1; f<=nframes; f++)
			ap_geometrical_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum,
					intensity_factor, f);
		checkErrorAfterKernelLaunch("ap_geometrical_streams_f_krnl");
		break;
	case HAPKE:
		/* Launch the Hapke kernel */
		for (f=1; f<=nframes; f++)
			ap_hapke_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_f, f);
		checkErrorAfterKernelLaunch("ap_hapke_streams_f_krnl");
		break;
	case HARMHAPKE:
		/* Launch the HarmHapke kernel */
		for (f=1; f<=nframes; f++)
			ap_harmhapke_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum,
					intensity_factor, phase_f, f);
		checkErrorAfterKernelLaunch("ap_harmhapke_streams_f_krnl");
		break;
	case INHOHAPKE:
		/* Launch the Inhomogeneous Hapke kernel */
		for (f=1; f<=nframes; f++)
			ap_inhohapke_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_f, f);
		checkErrorAfterKernelLaunch("ap_inhohapke_streams_f_krnl");
		break;
	case KAASALAINEN:
		/* Launch single-thread kernel to init Kaas */
//		ap_kaas_init_krnl<<<1,1>>>(dmod);
//		checkErrorAfterKernelLaunch("ap_kaas_init_krnl");
		gpuErrchk(hipMalloc((void**)&phasefuncf, sizeof(double)*(nframes+1)));
		/* Launch the main Kaasalainen kernel */
		for (f=1; f<=nframes; f++){
//			ap_kaas_streams_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
//					nThreads[f], body, span[f], xylim, dsum, f);
			ap_kaas_streams2_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod, pos,
					nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_f, phasefuncf, f);
			//hipDeviceSynchronize();
		}
		checkErrorAfterKernelLaunch("ap_kaas_streams2f_krnl");
		hipFree(phasefuncf);
		break;
	case HARMKAAS:
		/* Launch the HarmKaas kernel */
		for (f=1; f<=nframes; f++)
			ap_harmkaas_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_f, f);
		checkErrorAfterKernelLaunch("ap_harmkaas_streams_f_krnl");
		break;
	case INHOKAAS:
		/* Launch the HarmKaas kernel */
		for (f=1; f<=nframes; f++)
			ap_inhokaas_streams_f_krnl<<<BLKpx[f],THD,0,ap_stream[f-1]>>>(dmod,
					pos, nThreads[f], body, xylim, span[f], dsum, intensity_factor,
					phase_f, f);
		checkErrorAfterKernelLaunch("ap_inhokaas_streams_f_krnl");
		break;
	case NOLAW:
		bailout("apply_photo.c: can't set optical scattering law = \"none\" when optical data are used\n");
		break;
	default:
		bailout("apply_photo.c: can't handle that optical scattering law yet\n");
	}
	/* Now set the lghtcrv->y values with what was calculated here */
	BLK.x = floor((THD.x - 1 + (nframes+1)) / THD.x);
	ap_set_lghtcrv_y_streams2_krnl<<<BLK,THD>>>(ddat, set, dsum, (nframes+1));
	checkErrorAfterKernelLaunch("ap_set_lghtcrv_y_streams2_krnl");
	gpuErrchk(hipMemcpy(hsum, dsum, sizeof(float)*(nframes+1),
			hipMemcpyDeviceToHost));

	hipFree(dsum);
	hipFree(type);
	free(htype);
	free(hsum);
}

#undef TINY

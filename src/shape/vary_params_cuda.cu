#include "hip/hip_runtime.h"
/*****************************************************************************************
                                                                            vary_params.c

This routine is called by every processing node for every trial value of every floating
parameter during a fit, in order to implement the "vary_radalb" "vary_optalb"
"vary_delcor0" and "vary_dopscale" parameters.  The code, which is essentially lifted from
calc_fits.c, computes up to four means:

a) mean distance towards Earth of the subradar point relative to the COM,
   for delay-Doppler frames whose 0th-order delay correction polynomial coefficient is not
   held constant; this is used to adjust the 0th-order delay correction polynomial
   coefficient if the "vary_delcor0" parameter is turned on.

b) mean "radar" projected area for (delay-)Doppler frames that are treated as absolute
   photometry; this is used to adjust the radar albedo (R) if the "vary_radalb" parameter
   is turned on.

c) mean "optical" unshadowed projected area for calculated lightcurve points that are
   treated as absolute photometry; this is used to adjust the optical albedo (R or w) if
   the "vary_optalb" parameter is turned on.  Note that plane-of-sky datasets are not used
   here, since these frames are always treated as relative photometry.

d) mean cos(subradar latitude) for (delay-)Doppler frames in datasets whose Doppler
   scaling parameter is allowed to float; this is used to adjust those parameters if the
   "vary_dopscale" parameter is turned on.

When a branch node calls this routine, it returns its datasets' summed contributions (NOT
mean contributions) to the four output parameters, deldop_zmax, rad_xsec, opt_brightness,
and cos_subradarlat.

When the root node calls this routine, it first computes its datasets' summed
contributions to these four parameters; then it receives and adds in the contributions
from the branch nodes; and finally it returns the mean (NOT summed) parameters.

Before calling vary_params, the model's size/shape and spin states must be realized
(realize_mod and realize_spin); if albedos are being varied jointly with other parameters,
the photometric state must also be realized (realize_photo); and in either case the
0th-order delay correction polynomial coefficients and the Doppler scaling factors must be
reset to their saved values via the appropriate calls to realize_delcor and
realize_dopscale, respectively.

Modified 2016 November 6 by ME:
	Split off from vary_params to create a version that performs almost exclusively
	on the GPU

Modified 2015 June 10 by CM:
    Implement smearing

Modified 2014 February 12 by CM:
    Add "ilaw" argument to the apply_photo routine

Modified 2012 March 23 by CM:
    Implement Doppler scaling

Modified 2011 September 10 by CM:
    Two small aesthetic changes in the lightcurve section of the code

Modified 2010 June 15 by CM:
    Revise arguments to pos2deldop and pos2doppler routines

Modified 2010 April 12 by CM:
    Include overflow region when computing cross sections
    Added comment about calling realize_delcor before calling vary_params

Modified 2009 March 29 by CM:
    For MPI_Recv calls, mpi_par[0] is no longer equal to the MPI action,
        since the message tag argument already serves that purpose (as of
        2008 April 10) -- so the other mpi_par elements are renumbered
    Add "warn_badradar" argument to pos2deldop and pos2doppler routines

Modified 2008 April 10 by CM:
    Use message tag argument to MPI_Recv to identify the MPI action

Modified 2007 August 18 by CM:
    Rename MPI_TAG to MPI_TAG_1 to avoid name conflict with mpich headers

Modified 2007 August 4 by CM:
    Add orbit_offset and body arguments to posvis routine and remove
        facet argument
    Add orbit_xoff, orbit_yoff, orbit_dopoff, and body arguments to
        pos2deldop and pos2doppler routines
    Add body argument to apply_photo routine

Written 2006 October 1 by CM
 *****************************************************************************************/

extern "C" {
#include "head.h"
}

__device__ unsigned char dtype;
__device__ int dcompute_xsec, dcompute_zmax, dcompute_brightness,
			   dcompute_cosdelta, vary_params_dnframes, dpos_n,
			   dndop, dndel, vp_xlim0, vp_xlim1, vp_ylim0, vp_ylim1, vp_n,
			   dncalc, dlghtcrv_bistatic, dlghtcrv_n;
__device__ double dweight, vp_oa[3][3], vp_to_earth[3], vp_deldop_zmax,
			vp_rad_xsec, vp_opt_brightness, vp_cos_subradarlat;
__device__ float vpzmax, sum_deldop_zmax, sum_rad_xsec,	sum_opt_brightness,
				sum_cos_subradarlat, deldop_cross_section, doppler_cross_section;
__device__ struct pos_t *vp_pos;

//__host__ int NearestPowerOf2(int n);

__global__ void vp_init_vars() {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		sum_deldop_zmax = 0.0;
		sum_rad_xsec = 0.0;
		sum_opt_brightness = 0.0;
		sum_cos_subradarlat = 0.0;
	}
}
__global__ void get_data_type_krnl(struct dat_t *ddat, int s) {
	/* nset-threaded kernel */
	if (threadIdx.x == 0) {
		dtype = ddat->set[s].type;
	}
}
__global__ void get_compute_flags_krnl(struct par_t *dpar, struct dat_t *ddat,
		int s, int f) {
	/* Single-threaded kernel */
	if (threadIdx.x ==0) {
		switch (ddat->set[s].type) {
		case DELAY:
			dcompute_zmax = (dpar->vary_delcor0 != VARY_NONE
					&& ddat->set[s].desc.deldop.delcor.a[0].state != 'c');
			dcompute_xsec = (dpar->vary_radalb != VARY_NONE
					&& ddat->set[s].desc.deldop.frame[f].cal.state == 'c');
			dcompute_cosdelta = (dpar->vary_dopscale != VARY_NONE
					&& ddat->set[s].desc.deldop.dopscale.state != 'c');
			vp_pos = &ddat->set[s].desc.deldop.frame[f].pos;
			dweight = ddat->set[s].desc.deldop.frame[f].weight;
			dndel = ddat->set[s].desc.deldop.frame[f].ndel;
			dndop = ddat->set[s].desc.deldop.frame[f].ndop;
			break;
		case DOPPLER:
			dcompute_xsec = (dpar->vary_radalb != VARY_NONE &&
					ddat->set[s].desc.doppler.frame[f].cal.state == 'c');
			dcompute_cosdelta = (dpar->vary_dopscale != VARY_NONE &&
					ddat->set[s].desc.doppler.dopscale.state != 'c');
			vp_pos = &ddat->set[s].desc.doppler.frame[f].pos;
			dweight = ddat->set[s].desc.doppler.frame[f].weight;
			dndop = ddat->set[s].desc.doppler.frame[f].ndop;
			break;
//		case LGHTCRV:
//			vp_pos = &ddat->set[s].desc.lghtcrv.rend[f].pos;
//			dlghtcrv_bistatic = vp_pos->bistatic;
//			dlghtcrv_n = ddat->set[s].desc.lghtcrv.n;
//			break;
		}
	}
}
__global__ void lghtcrv_set_pos_krnl(struct dat_t *ddat, int s, int f) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		vp_pos = &ddat->set[s].desc.lghtcrv.rend[f].pos; /* Backup - delete this later */
		dlghtcrv_bistatic = vp_pos->bistatic;
		dlghtcrv_n = ddat->set[s].desc.lghtcrv.n;

	}
}
__global__ void get_lghtcrv_cb_krnl(struct par_t *dpar, struct dat_t *ddat,
		int s) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		dcompute_brightness = (dpar->vary_optalb != VARY_NONE
				&& ddat->set[s].desc.lghtcrv.cal.state == 'c');
	}
}
__global__ void get_vary_params_nframes_krnl(struct dat_t *ddat, int s) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		switch (ddat->set[s].type) {
		case DELAY:
			vary_params_dnframes = ddat->set[s].desc.deldop.nframes;
			break;
		case DOPPLER:
			vary_params_dnframes = ddat->set[s].desc.doppler.nframes;
			break;
		case LGHTCRV:
			vp_n = ddat->set[s].desc.lghtcrv.n;
			dncalc = ddat->set[s].desc.lghtcrv.ncalc;
			dweight = ddat->set[s].desc.lghtcrv.weight;
		}
	}
}
__global__ void set_ae_oe_bistatic_krnl(struct dat_t *ddat, int s,
		int f) {
	/* 9-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = offset % 3;
	int j = offset / 3;

	if (offset < 9) {

		switch(ddat->set[s].type) {
		case DELAY:
			vp_pos->ae[i][j] =	ddat->set[s].desc.deldop.frame[f].view[ddat->set[s].desc.deldop.v0].ae[i][j];
			vp_pos->oe[i][j] =	ddat->set[s].desc.deldop.frame[f].view[ddat->set[s].desc.deldop.v0].oe[i][j];
			break;
		case DOPPLER:
			vp_pos->ae[i][j] =	ddat->set[s].desc.doppler.frame[f].view[ddat->set[s].desc.doppler.v0].ae[i][j];
			vp_pos->oe[i][j] =	ddat->set[s].desc.doppler.frame[f].view[ddat->set[s].desc.doppler.v0].oe[i][j];
			break;
		case LGHTCRV:
			vp_pos->ae[i][j] = ddat->set[s].desc.lghtcrv.rend[f].ae[i][j];
			vp_pos->oe[i][j] = ddat->set[s].desc.lghtcrv.rend[f].oe[i][j];
			vp_pos->se[i][j] = ddat->set[s].desc.lghtcrv.rend[f].se[i][j];
		}
		/* The following is a single-thread task */
		if (threadIdx.x == 0) {
			if (ddat->set[s].type == LGHTCRV)
				vp_pos->bistatic = 1;
			else
				vp_pos->bistatic = 0;
		}
	}
}
__global__ void get_pos_n_krnl()
{
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		dpos_n = vp_pos->n;
	}
}
__global__ void vp_get_xylims_krnl(struct mod_t *dmod, struct dat_t
		*ddat, int set, int frm) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		switch(ddat->set[set].type) {
		case DELAY:
			vp_xlim0 = ddat->set[set].desc.deldop.frame[frm].pos.xlim[0];
			vp_xlim1 = ddat->set[set].desc.deldop.frame[frm].pos.xlim[1];
			vp_ylim0 = ddat->set[set].desc.deldop.frame[frm].pos.ylim[0];
			vp_ylim1 = ddat->set[set].desc.deldop.frame[frm].pos.ylim[1];
			break;
		case DOPPLER:
			vp_xlim0 = ddat->set[set].desc.doppler.frame[frm].pos.xlim[0];
			vp_xlim1 = ddat->set[set].desc.doppler.frame[frm].pos.xlim[1];
			vp_ylim0 = ddat->set[set].desc.doppler.frame[frm].pos.ylim[0];
			vp_ylim1 = ddat->set[set].desc.doppler.frame[frm].pos.ylim[1];
			break;
		case LGHTCRV:
			vp_xlim0 = ddat->set[set].desc.lghtcrv.rend[frm].pos.xlim[0];
			vp_xlim1 = ddat->set[set].desc.lghtcrv.rend[frm].pos.xlim[1];
			vp_ylim0 = ddat->set[set].desc.lghtcrv.rend[frm].pos.ylim[0];
			vp_ylim1 = ddat->set[set].desc.lghtcrv.rend[frm].pos.ylim[1];
			break;
		}
		vpzmax = -HUGENUMBER;
	}
}
__global__ void deldop_clrvect_krnl(struct dat_t *ddat, int s, int f, int size) {
	/* Multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (offset < size) {
		ddat->set[s].desc.deldop.frame[f].fit_s[offset] = 0.0;
	}
}
__global__ void zmax_finalize_krnl(float value) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		sum_deldop_zmax += value*dweight;
	}
}
__global__ void compute_xsec_final_krnl(struct dat_t *ddat, float frm_xsec,
		int s, int f) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		switch (ddat->set[s].type) {
		case DELAY:
			deldop_cross_section = __double2float_rd(ddat->set[s].desc.deldop.frame[f].overflow_xsec);
			deldop_cross_section += frm_xsec; // fit is the end result of parallel reduction
			deldop_cross_section *= ddat->set[s].desc.deldop.frame[f].cal.val;
			sum_rad_xsec += deldop_cross_section*ddat->set[s].desc.deldop.frame[f].weight;
			break;
		case DOPPLER:
			doppler_cross_section = __double2float_rd(ddat->set[s].desc.doppler.frame[f].overflow_xsec);
			doppler_cross_section += frm_xsec;
			doppler_cross_section *= ddat->set[s].desc.doppler.frame[f].cal.val;
			sum_rad_xsec += doppler_cross_section*ddat->set[s].desc.doppler.frame[f].weight;

			break;
		}
	}
}
__global__ void compute_cosdelta_krnl(struct dat_t *ddat, int s, int f) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		int j, view;
		double cos_delta;

		switch(ddat->set[s].type){
		case DELAY:
			view = ddat->set[s].desc.deldop.v0;
			/* oa = matrix to transform body-fixed to observer coordinates  */
			/* to_earth = normalized target-to-Earth vector in body-fixed coords  */
			dev_mtrnsps( vp_oa, ddat->set[s].desc.deldop.frame[f].view[view].ae);
			dev_mmmul( vp_oa, ddat->set[s].desc.deldop.frame[f].view[view].oe, vp_oa);
			for (j=0; j<=2; j++)
				vp_to_earth[j] = vp_oa[2][j];
			cos_delta = sqrt(vp_to_earth[0]*vp_to_earth[0] + vp_to_earth[1]*vp_to_earth[1]);
			dweight = ddat->set[s].desc.deldop.frame[f].weight;
			sum_cos_subradarlat += cos_delta*dweight;
			break;
		case DOPPLER:
			view = ddat->set[s].desc.doppler.v0;
			/* oa = matrix to transform body-fixed to observer coordinates  */
			/* to_earth = normalized target-to-Earth vector in body-fixed coords  */
			dev_mtrnsps(vp_oa, ddat->set[s].desc.doppler.frame[f].view[view].ae);
			dev_mmmul(vp_oa, ddat->set[s].desc.doppler.frame[f].view[view].oe, vp_oa);
			for (j=0; j<=2; j++)
				vp_to_earth[j] = vp_oa[2][j];
			cos_delta = sqrt(vp_to_earth[0]*vp_to_earth[0] + vp_to_earth[1]*vp_to_earth[1]);
			dweight = ddat->set[s].desc.doppler.frame[f].weight;
			sum_cos_subradarlat += cos_delta*dweight;
		}
	}
}
__global__ void posclr_krnl(int n, int nx)
{
	/* Multi-threaded kernel (2*pos->n + 1)^2 threads) */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int i = (offset % nx) - n;
	int j = (offset / nx) - n;

	if (offset < (nx*nx)) {
		/* For each POS pixel, zero out the optical brightness (b) and
		 * cos(scattering angle), reset the z coordinate (distance from COM towards
		 * Earth) to a dummy value, and reset the body, component, and facet onto
		 * which the pixel center projects to  dummy values                  */
//		vp_pos->b[i][j] = vp_pos->cose[i][j] = 0.0;
//		vp_pos->z[i][j] = -HUGENUMBER;
		vp_pos->body[i][j] = vp_pos->comp[i][j] = vp_pos->f[i][j] = -1;

		vp_pos->b_s[offset] = vp_pos->cose_s[offset] = 0.0;
		vp_pos->z_s[offset] = -HUGENUMBER;

		/* In the x direction, reset the model's leftmost and rightmost
		 * pixel number to dummy values, and similarly for the y direction   */
		vp_pos->xlim[0] = vp_pos->ylim[0] =  n;
		vp_pos->xlim[1] = vp_pos->ylim[1] = -n;

		/* For a bistatic situation (lightcurve or plane-of-sky dataset), zero out
		 * cos(incidence angle) and reset the distance towards the sun, the body,
		 * component, and facet numbers as viewed from the sun, and the model's
		 * maximum projected extent as viewed from the sun to dummy values    */
		if (vp_pos->bistatic) {
			vp_pos->cosill[i][j] = 0.0;
			vp_pos->zill[i][j] = -HUGENUMBER;
			vp_pos->bodyill[i][j] = vp_pos->compill[i][j] = vp_pos->fill[i][j] = -1;

			vp_pos->cosill_s[offset] = 0.0;
			vp_pos->zill_s[offset] = -HUGENUMBER;

			vp_pos->xlim2[0] = vp_pos->ylim2[0] =  n;
			vp_pos->xlim2[1] = vp_pos->ylim2[1] = -n;
		}
	}
	__syncthreads();
}
__global__ void doppler_clrvect_krnl(struct dat_t *ddat, int s, int f) {
	/* ndop-threaded kernel */
	int i = 1 + blockIdx.x * blockDim.x + threadIdx.x;
	if (i <= dndop){
		ddat->set[s].desc.doppler.frame[f].fit_s[i] = 0.0;
	}
}
__global__ void doppler_compute_xsec_krnl(struct dat_t *ddat, int s,
		int f) {
	/* ndop-threaded kernel */
	/* Deprecated and for debug use only */
	int dop = blockIdx.x * blockDim.x + threadIdx.x;
	//float fit;	// the fit value for this thread/doppler bin only

	if (dop == 0)
		doppler_cross_section = __double2float_rd(ddat->set[s].desc.doppler.frame[f].overflow_xsec);
	__syncthreads();

	if (dop < dndop) {
		atomicAdd(&doppler_cross_section, ddat->set[s].desc.doppler.frame[f].fit_s[dop+1]);
	}
	__syncthreads();

	if (dop == 1) {
		doppler_cross_section *= ddat->set[s].desc.doppler.frame[f].cal.val;
		sum_rad_xsec += doppler_cross_section*dweight;
	}
 	__syncthreads();
}
__global__ void posmask_krnl(struct par_t *dpar, int nThreads, int xspan)
{
	/* multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int n = dpos_n;
	int i = offset % xspan - n;
	int j = offset / xspan - n;
	double tol = dpar->mask_tol;
	int im, jm, i1, j1, i2, j2, i_sign, j_sign;
	double xk[3], so[3][3], pixels_per_km, i0_dbl, j0_dbl, zill, t, u, bignum;

	if (offset == 0){
		bignum = 0.99*HUGENUMBER;  /* z = -HUGENUMBER for blank-sky pixels */

		dev_mtrnsps( so, vp_pos->oe);
		dev_mmmul( so, vp_pos->se, so);    /* so takes obs into src coords */
		pixels_per_km = 1/vp_pos->km_per_pixel;
	}
	__syncthreads();

	/*  Loop through all POS pixels  */
	if (offset < nThreads) {
		//	n = vp_pos->n;
		//	for (i=(-n); i<=n; i++) {               /* for each pixel in the */
		//		for (j=(-n); j<=n; j++) {             /* observer's view */
		if (vp_pos->cose_s[offset] != 0.0) {     /* if there's something there */
			xk[0] = i*vp_pos->km_per_pixel;     /* calculate 3D position */
			xk[1] = j*vp_pos->km_per_pixel;
			xk[2] = vp_pos->z_s[offset];

			/* Given the observer coordinates x of of POS pixel (i,j), find
			 * which pixel (im,jm) this corresponds to in the projected view as
			 * seen from the source (sun)             */

			dev_cotrans2( xk, so, xk, 1);           /* go into source coordinates */
			i0_dbl = xk[0]*pixels_per_km;     /* unrounded (double precision) */
			j0_dbl = xk[1]*pixels_per_km;
			im = dev_vp_iround( i0_dbl);            /* center of nearest pixel in mask */
			jm = dev_vp_iround( j0_dbl);

			/* If center of projected pixel "seen" from source (as determined
			 * by routine posvis) lies within the boundaries of the mask,
			 * projects onto model rather than onto blank space, and represents
			 * a body, component, and facet different from those seen in the
			 * POS, calculate distance from mask pixel to source and compare to
			 * distance from POS pixel to source.                             */

			if (fabs(i0_dbl) < n && fabs(j0_dbl) < n
					&& vp_pos->zill[im][jm] > -bignum
					&& (vp_pos->f[i][j]    != vp_pos->fill[im][jm]    ||
							vp_pos->comp[i][j] != vp_pos->compill[im][jm] ||
							vp_pos->body[i][j] != vp_pos->bodyill[im][jm]    )) {

				/* Rather than using distance towards source of CENTER of mask
				 * pixel, use bilinear interpolation to get distance towards
				 * source where the line between source and POS pixel's center
				 * intersects the mask pixel.                                */
				i1 = (int) floor( i0_dbl);
				j1 = (int) floor( j0_dbl);

				if (vp_pos->zill[i1][j1]     > -bignum &&
						vp_pos->zill[i1+1][j1]   > -bignum &&
						vp_pos->zill[i1][j1+1]   > -bignum &&
						vp_pos->zill[i1+1][j1+1] > -bignum    ) {

					/* Do standard bilinear interpolation: None of the four
					 * surrounding "grid square" pixels in the mask is
					 * blank sky                           */
					t = i0_dbl - i1;
					u = j0_dbl - j1;
					zill = (1 - t)*(1 - u)*vp_pos->zill[i1][j1]
                               + t*(1 - u)*vp_pos->zill[i1+1][j1]
                                     + t*u*vp_pos->zill[i1+1][j1+1]
	                           + (1 - t)*u*vp_pos->zill[i1][j1+1];
				} else {

					/* The following code block is a kludge: One or more of the
					 * four surrounding "grid square" pixels in mask is blank
					 * sky, so standard bilinear interpolation won't work.  */
					zill = vp_pos->zill[im][jm];

					i_sign = (i0_dbl >= im) ? 1 : -1;
					i2 = im + i_sign;
					if (abs(i2) <= n && vp_pos->zill[i2][jm] > -bignum) {
						zill += fabs(i0_dbl - im)
           				  * (vp_pos->zill[i2][jm] - vp_pos->zill[im][jm]);
					} else {
						i2 = im - i_sign;
						if (abs(i2) <= n && vp_pos->zill[i2][jm] > -bignum)
							zill -= fabs(i0_dbl - im)
							* (vp_pos->zill[i2][jm] - vp_pos->zill[im][jm]);
					}

					j_sign = (j0_dbl >= jm) ? 1 : -1;
					j2 = jm + j_sign;
					if (abs(j2) <= n && vp_pos->zill[im][j2] > -bignum) {
						zill += fabs(j0_dbl - jm)
                          * (vp_pos->zill[im][j2] - vp_pos->zill[im][jm]);
					} else {
						j2 = jm - j_sign;
						if (abs(j2) <= n && vp_pos->zill[im][j2] > -bignum)
							zill -= fabs(j0_dbl - jm)
							* (vp_pos->zill[im][j2] - vp_pos->zill[im][jm]);
					}
				}

				/* If interpolated point within mask pixel is at least tol km
				 * closer to source than is the center of POS pixel, the facet
				 * represented by the mask pixel is shadowing the POS pixel:
				 * represent this by setting
				 * 		cos(scattering angle) = 0.0 for the POS pixel.      */
				if (zill - xk[2] > tol)
					vp_pos->cose_s[offset] = 0.0;
			}
		}
	}
}
__global__ void posmask_universal_krnl(struct par_t *dpar, struct pos_t *pos,
		int nThreads, int xspan)
{
	/* multi-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int n = pos->n;
	int i = offset % xspan - n;
	int j = offset / xspan - n;
	double tol = dpar->mask_tol;
	int im, jm, i1, j1, i2, j2, i_sign, j_sign;
	double xk[3], so[3][3], pixels_per_km, i0_dbl, j0_dbl, zill, t, u, bignum;

	if (offset == 0){
		bignum = 0.99*HUGENUMBER;  /* z = -HUGENUMBER for blank-sky pixels */

		dev_mtrnsps( so, pos->oe);
		dev_mmmul( so, pos->se, so);    /* so takes obs into src coords */
		pixels_per_km = 1/pos->km_per_pixel;
	}
	__syncthreads();

	/*  Loop through all POS pixels  */
	if (offset < nThreads) {
		if (pos->cose_s[offset] != 0.0) {     /* if there's something there */
			xk[0] = i*pos->km_per_pixel;     /* calculate 3D position */
			xk[1] = j*pos->km_per_pixel;
			xk[2] = pos->z_s[offset];

			/* Given the observer coordinates x of of POS pixel (i,j), find
			 * which pixel (im,jm) this corresponds to in the projected view as
			 * seen from the source (sun)             */

			dev_cotrans2( xk, so, xk, 1);           /* go into source coordinates */
			i0_dbl = xk[0]*pixels_per_km;     /* unrounded (double precision) */
			j0_dbl = xk[1]*pixels_per_km;
			im = dev_vp_iround( i0_dbl);            /* center of nearest pixel in mask */
			jm = dev_vp_iround( j0_dbl);

			/* If center of projected pixel "seen" from source (as determined
			 * by routine posvis) lies within the boundaries of the mask,
			 * projects onto model rather than onto blank space, and represents
			 * a body, component, and facet different from those seen in the
			 * POS, calculate distance from mask pixel to source and compare to
			 * distance from POS pixel to source.                             */

			if (fabs(i0_dbl) < n && fabs(j0_dbl) < n
					&& pos->zill[im][jm] > -bignum
					&&(pos->f[i][j]    != pos->fill[im][jm]    ||
					   pos->comp[i][j] != pos->compill[im][jm] ||
					   pos->body[i][j] != pos->bodyill[im][jm]    )) {

				/* Rather than using distance towards source of CENTER of mask
				 * pixel, use bilinear interpolation to get distance towards
				 * source where the line between source and POS pixel's center
				 * intersects the mask pixel.                                */
				i1 = (int) floor( i0_dbl);
				j1 = (int) floor( j0_dbl);

				if (pos->zill[i1][j1] > -bignum && pos->zill[i1+1][j1] > -bignum &&
					pos->zill[i1][j1+1] > -bignum && pos->zill[i1+1][j1+1] > -bignum) {

					/* Do standard bilinear interpolation: None of the four
					 * surrounding "grid square" pixels in the mask is
					 * blank sky                           */
					t = i0_dbl - i1;
					u = j0_dbl - j1;
					zill = (1 - t)*(1 - u)*pos->zill[i1][j1]
                               + t*(1 - u)*pos->zill[i1+1][j1]
                                     + t*u*pos->zill[i1+1][j1+1]
	                           + (1 - t)*u*pos->zill[i1][j1+1];
				} else {

					/* The following code block is a kludge: One or more of the
					 * four surrounding "grid square" pixels in mask is blank
					 * sky, so standard bilinear interpolation won't work.  */
					zill = pos->zill[im][jm];

					i_sign = (i0_dbl >= im) ? 1 : -1;
					i2 = im + i_sign;
					if (abs(i2) <= n && pos->zill[i2][jm] > -bignum) {
						zill += fabs(i0_dbl - im)  *
								(pos->zill[i2][jm] - pos->zill[im][jm]);
					} else {
						i2 = im - i_sign;
						if (abs(i2) <= n && pos->zill[i2][jm] > -bignum)
							zill -= fabs(i0_dbl - im)
							* (pos->zill[i2][jm] - pos->zill[im][jm]);
					}

					j_sign = (j0_dbl >= jm) ? 1 : -1;
					j2 = jm + j_sign;
					if (abs(j2) <= n && pos->zill[im][j2] > -bignum) {
						zill += fabs(j0_dbl - jm)
                          * (pos->zill[im][j2] - pos->zill[im][jm]);
					} else {
						j2 = jm - j_sign;
						if (abs(j2) <= n && pos->zill[im][j2] > -bignum)
							zill -= fabs(j0_dbl - jm)
							* (pos->zill[im][j2] - pos->zill[im][jm]);
					}
				}

				/* If interpolated point within mask pixel is at least tol km
				 * closer to source than is the center of POS pixel, the facet
				 * represented by the mask pixel is shadowing the POS pixel:
				 * represent this by setting
				 * 		cos(scattering angle) = 0.0 for the POS pixel.      */
				if (zill - xk[2] > tol)
					pos->cose_s[offset] = 0.0;
			}
		}
	}
}
__global__ void lghtcrv_copy_y_krnl(struct dat_t *ddat, double host_value,
		int set, int i) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		ddat->set[set].desc.lghtcrv.y[i] = host_value;
	}
}
__global__ void lghtcrv_spline_krnl(struct dat_t *ddat, int set, double
		yp1, double ypn, double *u) {
	/*(double *x  - lghtcrv->x
	 * double *y  - lghtcrv->y
	 * int n      - calc
	 * double yp1 - 2.0e30
	 * double ypn - 2.0e30
	 * double *y2 - lghtcrv->y2)*/

	/* Multi-threaded kernel */
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int n = dncalc;
	int k = n - 1 - i;
	double *x = ddat->set[set].desc.lghtcrv.x;
	double *y = ddat->set[set].desc.lghtcrv.y;
	double *y2 = ddat->set[set].desc.lghtcrv.y2;
	double p,qn,sig,un;

	/* Perform single-thread task */
	if (i == 0) {
		if (yp1 > 0.99e30)
			y2[1]=u[1]=0.0;
		else {
			y2[1] = -0.5;
			u[1]=(3.0/(x[2]-x[1]))*((y[2]-y[1])/(x[2]-x[1])-yp1);
		}
	}
	__syncthreads();

	if (i > 1 && i < n-1) {

		sig=(x[i]-x[i-1])/(x[i+1]-x[i-1]);
		p=sig*y2[i-1]+2.0;
		y2[i]=(sig-1.0)/p;
		u[i]=(y[i+1]-y[i])/(x[i+1]-x[i]) - (y[i]-y[i-1])/(x[i]-x[i-1]);
		u[i]=(6.0*u[i]/(x[i+1]-x[i-1])-sig*u[i-1])/p;
	}
	__syncthreads();

	/* Perform another single-thread task */
	if (i == 1) {
		if (ypn > 0.99e30)
			qn=un=0.0;
		else {
			qn=0.5;
			un=(3.0/(x[n]-x[n-1]))*(ypn-(y[n]-y[n-1])/(x[n]-x[n-1]));
		}
		y2[n]=(un-qn*u[n-1])/(qn*y2[n-1]+1.0);
	}
	__syncthreads();

	if (k <= (n-1) && k >= 1)
		y2[k]=y2[k]*y2[k+1]+u[k];

	__syncthreads();
}
__global__ void lghtcrv_splint_krnl(struct dat_t *ddat, int set)
{
	/* This is an n-threaded kernel where n = lghtcrv-> */
	/* Parameters:
	 * double *xa  - lghtcrv->x
	 * double *ya  - lghtcrv->y
	 * double *y2a - lghtcrv->y2
	 * int n       - ncalc
	 * double x    - lghtcrv->t[i][lghtcrv->v0]
	 * double *y   - lghtcrv->fit[i]	 *
	 */

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double *xa	= ddat->set[set].desc.lghtcrv.x;
	double *ya	= ddat->set[set].desc.lghtcrv.y;
	double *y2a	= ddat->set[set].desc.lghtcrv.y2;
	double x 	= ddat->set[set].desc.lghtcrv.t[i][ddat->set[set].desc.lghtcrv.v0];
	double *y 	= &ddat->set[set].desc.lghtcrv.t[i][ddat->set[set].desc.lghtcrv.v0];
	int n = dncalc;

	int klo,khi,k;
	double h,b,a;

	if (i < dlghtcrv_n) {

		klo=1;
		khi=n;
		while (khi-klo > 1) {
			k=(khi + klo) >> 1;
			if (xa[k] > x) khi=k;
			else klo=k;
		}
		h = xa[khi] - xa[klo];
		if (h == 0.0) printf("Bad XA input to routine SPLINT");
		a=(xa[khi] - x) / h;
		b=(x - xa[klo]) / h;
		*y=a*ya[klo] + b*ya[khi] + ((a*a*a-a) * y2a[klo] + (b*b*b-b) * y2a[khi]) * (h*h) /6.0;
	}
}
__global__ void vp_set_four_parameters_krnl(struct dat_t *ddat) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		if (ddat->sum_deldop_zmax_weights > 0.0)
			vp_deldop_zmax = sum_deldop_zmax / ddat->sum_deldop_zmax_weights;
		else
			vp_deldop_zmax = 0.0;
		if (ddat->sum_rad_xsec_weights > 0.0) {
			vp_rad_xsec = sum_rad_xsec / ddat->sum_rad_xsec_weights;			}
		else
			vp_rad_xsec = 0.0;
		if (ddat->sum_opt_brightness_weights > 0.0)
			vp_opt_brightness = sum_opt_brightness / ddat->sum_opt_brightness_weights;
		else
			vp_opt_brightness = 0.0;
		if (ddat->sum_cos_subradarlat_weights > 0.0)
			vp_cos_subradarlat = sum_cos_subradarlat / ddat->sum_cos_subradarlat_weights;
		else
			vp_cos_subradarlat = 0.0;
	}
}

__host__ void vary_params_cuda( struct par_t *dpar, struct mod_t *dmod,
		struct dat_t *ddat, int action, double *deldop_zmax, double
		*rad_xsec, double *opt_brightness, double *cos_subradarlat, int nsets)
{
	/* Inputs:
	 * int action 				- dpar->action
	 * double *deldop_zmax 		- &deldop_zmax_save
	 * double *rad_xsec 		- &rad_xsec_save
	 * double *optbrightness	- &opt_brightness_save
	 * double *cos_subradarlat	- &cos_subradarlat_save
	 * int nsets 				- ddat->nsets
	 */

	double orbit_offset[3] = {0.0, 0.0, 0.0};
	int c=0, f, s, i, compute_xsec, compute_brightness, compute_zmax,
			compute_cosdelta, n, ncalc, pos_n, nx, lghtcrv_bistatic, nframes,
			xlim[2], ylim[2], xspan, lghtcrv_n;
	double weight;
	dim3 BLK,THD;
	unsigned char type;
	int ndel, ndop, nThreads;
	struct pos_t *pos;	/* Experimental for lghtcrv use and reuse */

	/*  Initialize variables  */
	vp_init_vars<<<1,1>>>();
	checkErrorAfterKernelLaunch("vp_init_krnl, line ");

	/* Process each dataset in turn */
	for (s=0; s<nsets; s++) {
		/* Get set's data type from GPU to host so we can SWITCH it */
		get_data_type_krnl<<<1,1>>>(ddat, s);
		checkErrorAfterKernelLaunch ("get_data_type_krnl, line ");
		gpuErrchk(hipMemcpyFromSymbol(&type, HIP_SYMBOL(dtype), sizeof(dtype), 0,
				hipMemcpyDeviceToHost));

		switch (type) {
		case DELAY:
			/* Get nframes */
			get_vary_params_nframes_krnl<<<1,1>>>(ddat, s);
			checkErrorAfterKernelLaunch ("get_data_type_krnl, line ");
			gpuErrchk(hipMemcpyFromSymbol(&nframes, HIP_SYMBOL(vary_params_dnframes),
				sizeof(vary_params_dnframes), 0, hipMemcpyDeviceToHost));

			for (f=0; f<nframes; f++) {
				/* Get the compute_zmax and compute x_sec flags in a single-
				 * threaded kernel */
				get_compute_flags_krnl<<<1,1>>>(dpar, ddat, s, f);
				checkErrorAfterKernelLaunch("get_compute_flags_krnl, line ");
				gpuErrchk(hipMemcpyFromSymbol(&compute_zmax, HIP_SYMBOL(dcompute_zmax),
						sizeof(dcompute_zmax), 0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&compute_xsec, HIP_SYMBOL(dcompute_xsec),
						sizeof(dcompute_xsec), 0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&compute_cosdelta, HIP_SYMBOL(dcompute_cosdelta),
						sizeof(dcompute_cosdelta), 0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&weight, HIP_SYMBOL(dweight),
						sizeof(dweight), 0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&ndel, HIP_SYMBOL(dndel),
						sizeof(dndel), 0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&ndop, HIP_SYMBOL(dndop),
						sizeof(dndop), 0, hipMemcpyDeviceToHost));

				/* Start the if block for computing zmax and/or cross-section */
				if (compute_zmax || compute_xsec) {
					/* Launch 9-threaded kernel to set up ae[3][3] and oe[3][3]
					 * and also set the bistatic flag 		 */
					THD.x = 9;
					set_ae_oe_bistatic_krnl<<<BLK,THD>>>(ddat, s, f);
					checkErrorAfterKernelLaunch("deldop_set_ae_oe_bistatic_krnl, line ");

					/* Need to get pos->n for kernel launch first */
					get_pos_n_krnl<<<1,1>>>();
					checkErrorAfterKernelLaunch("get_deldop_pos_n_krnl, line ");
					gpuErrchk(hipMemcpyFromSymbol(&pos_n, HIP_SYMBOL(dpos_n), sizeof(dpos_n),
							0, hipMemcpyDeviceToHost));

					/* Configure & launch posclr_krnl to initialize POS view */
					nThreads = (2*pos_n+1)*(2*pos_n+1);
					BLK.x = floor((maxThreadsPerBlock - 1 + nThreads) /
							maxThreadsPerBlock);
					THD.x = maxThreadsPerBlock; // Thread block dimensions
					nx = 2*pos_n + 1;
					posclr_krnl<<<BLK,THD>>>(pos_n, nx);
					checkErrorAfterKernelLaunch("posclr_krnl, line ");

					/*  Determine which POS pixels cover the target, and
	                    get the distance toward Earth of each POS pixel   */
					posvis_cuda_2(dpar, dmod, ddat, orbit_offset,s,f,
							0, 0, c);

					/* Zero out the fit delay-Doppler image and call pos2deldop
					 * to create the fit image by mapping power from the plane
					 * of sky to delay-Doppler space.    				      *
					 * (Call to clrmat has been removed. It simply zeroed out *
					 * frame->fit. I am using a separate array (fit) which is *
					 * automatically zeroed at creation time via cudaCalloc   */
					/* Configure and launch the clrmat kernel for deldop data */
					/* This will be a (ndel*ndop)-threaded kernel */

					/* Call the CUDA pos2deldop function */
					/* par, mod->photo,orbit_xoff,orbit_yoff,orbit_dopoff=0.0
					 * deldop, body = 0,s,f,v=0
					 * fit is the single pointer replacing the frame->fit array
					 * which did not copy well (?).  We use this separate fit
					 * array instead		 */

					/* Clear out the fit_s array first */
					BLK.x = floor((maxThreadsPerBlock - 1 + (ndel*ndop)) /
							maxThreadsPerBlock);
					THD.x = maxThreadsPerBlock; // Thread block dimensions
					deldop_clrvect_krnl<<<BLK,THD>>>(ddat, s, f, (ndel*ndop));
					checkErrorAfterKernelLaunch("deldop_clrvect_krnl in vary_params_cuda");

					pos2deldop_cuda_2(dpar,dmod,ddat,0.0,0.0,0.0,0,s,f,0);

					/* Compute distance toward Earth of the subradar point  */
					if (compute_zmax) {
						float zmax = 0.0;
						int size = 0;
						size = (2*pos_n+1)*(2*pos_n+1);

						/* Configure and launch the parallel reduction kernel to
						 * find distance toward Earth of the subradar point */
						zmax = compute_pos_zmax(ddat, size, s, f);
						zmax_finalize_krnl<<<1,1>>>(zmax);
						checkErrorAfterKernelLaunch("zmax_finalize_krnl in vary_params_cuda");
					}

					/*  Compute cross section  */
					if (compute_xsec) {
						float xsec=0.0;
						xsec = compute_deldop_xsec_snglkrnl(ddat, ndel, ndop, s, f);
						compute_xsec_final_krnl<<<1,1>>>(ddat, xsec, s, f);
						checkErrorAfterKernelLaunch("compute_xsec_final_krnl (deldop)");
					}
				}
				if (compute_cosdelta) {
					/* Launch single-thread kernel to compute sum_cos_subradarlat */
					compute_cosdelta_krnl<<<1,1>>>(ddat, s, f);
					checkErrorAfterKernelLaunch("deldop_compute_cosdelta_krnl, line ");
				}
			}
			break;
		case DOPPLER:
			/* Get nframes */
			get_vary_params_nframes_krnl<<<1,1>>>(ddat, s);
			checkErrorAfterKernelLaunch ("get_data_type_krnl, line ");
			gpuErrchk(hipMemcpyFromSymbol(&nframes, HIP_SYMBOL(vary_params_dnframes),
					sizeof(vary_params_dnframes), 0, hipMemcpyDeviceToHost));

			for (f=0; f<nframes; f++) {
				/* Launch single-thread kernel to get our compute flags first */
				get_compute_flags_krnl<<<1,1>>>(dpar, ddat, s, f);
				checkErrorAfterKernelLaunch("doppler_get_compute_flags, line ");
				gpuErrchk(hipMemcpyFromSymbol(&compute_xsec, HIP_SYMBOL(dcompute_xsec),
						sizeof(dcompute_xsec), 0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&compute_cosdelta, HIP_SYMBOL(dcompute_cosdelta),
						sizeof(dcompute_cosdelta), 0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&weight, HIP_SYMBOL(dweight),
						sizeof(dweight), 0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&ndop, HIP_SYMBOL(dndop),
						sizeof(dndop), 0, hipMemcpyDeviceToHost));

				if (compute_xsec) {
					/* Launch 9-threaded kernel to set up ae[3][3] and oe[3][3]
					 * and also set the bistatic flag 		 */
					THD.x = 9;
					set_ae_oe_bistatic_krnl<<<BLK,THD>>>(ddat, s, f);
					checkErrorAfterKernelLaunch("deldop_set_ae_oe_bistatic_krnl, line ");

					/* Need to get pos->n for kernel launch first */
					get_pos_n_krnl<<<1,1>>>();
					checkErrorAfterKernelLaunch("get_deldop_pos_n_krnl, line ");
					gpuErrchk(hipMemcpyFromSymbol(&pos_n, HIP_SYMBOL(dpos_n), sizeof(dpos_n),
							0, hipMemcpyDeviceToHost));

					/* Configure & launch posclr_krnl to initialize POS view */
					BLK.x = floor((maxThreadsPerBlock - 1 + (2*pos_n+1)*(2*pos_n+1)) /
							maxThreadsPerBlock);
					THD.x = maxThreadsPerBlock; // Thread block dimensions
					nx = 2*pos_n + 1;
					posclr_krnl<<<BLK,THD>>>(pos_n, nx);
					checkErrorAfterKernelLaunch("posclr_krnl, line ");

					/* Determine which POS pixels cover the target  */
					posvis_cuda_2(dpar,dmod,ddat,orbit_offset,s,f,0,0,c);

					/* Zero out the fit Doppler spectrum, then call pos2doppler to create the fit
					 * spectrum by mapping power from the plane of the sky to Doppler space.      */
					BLK.x = floor((maxThreadsPerBlock - 1 + ndop) / maxThreadsPerBlock);
					doppler_clrvect_krnl<<<BLK,THD>>>(ddat, s, f);
					checkErrorAfterKernelLaunch("doppler_clrvect_krnl, line ");

					pos2doppler_cuda_2(dpar,dmod,ddat,0.0,0.0,0.0,0,s,f,0);

					/* Compute cross section */
					float xsec=0.0;
					xsec = compute_doppler_xsec(ddat, ndop, s, f);
					compute_xsec_final_krnl<<<1,1>>>(ddat, xsec, s, f);
					checkErrorAfterKernelLaunch("compute_xsec_final_krnl (Doppler)");
				}
				if (compute_cosdelta) {
					/* Launch single-thread kernel to compute sum_cos_subradarlat */
					compute_cosdelta_krnl<<<1,1>>>(ddat, s, f);
					checkErrorAfterKernelLaunch("doppler_compute_cosdelta_krnl, line ");
				}
			}
			break;
		case POS:
			break;
		case LGHTCRV:
			/* Figure out the compute_brightness flag first */
			get_lghtcrv_cb_krnl<<<1,1>>>(dpar, ddat, s);
			checkErrorAfterKernelLaunch("get_lghtcrv_cb_krnl");
			gpuErrchk(hipMemcpyFromSymbol(&compute_brightness, HIP_SYMBOL(dcompute_brightness),
					sizeof(int), 0, hipMemcpyDeviceToHost));

			if (compute_brightness) {
				/* Launch single-thread kernel to get lghtcrv parameters */
				get_vary_params_nframes_krnl<<<1,1>>>(ddat, s);
				checkErrorAfterKernelLaunch("lghtcrv_get_params, line ");
				gpuErrchk(hipMemcpyFromSymbol(&n, HIP_SYMBOL(vp_n), sizeof(vp_n),
							0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&ncalc, HIP_SYMBOL(dncalc), sizeof(ncalc),
							0, hipMemcpyDeviceToHost));
				gpuErrchk(hipMemcpyFromSymbol(&weight, HIP_SYMBOL(dweight), sizeof(weight),
							0, hipMemcpyDeviceToHost));
				double lghtcrv_y;

				for (i=1; i<=ncalc; i++) {

					/* Launch kernel to get compute flags and set pos */
					lghtcrv_set_pos_krnl<<<1,1>>>(ddat, s, i);
					checkErrorAfterKernelLaunch("lghtcrv_set_pos_krnl");
					gpuErrchk(hipMemcpyFromSymbol(&lghtcrv_bistatic, HIP_SYMBOL(dlghtcrv_bistatic),
									sizeof(lghtcrv_bistatic), 0, hipMemcpyDeviceToHost));
					gpuErrchk(hipMemcpyFromSymbol(&lghtcrv_n, HIP_SYMBOL(dlghtcrv_n),
							sizeof(lghtcrv_n), 0, hipMemcpyDeviceToHost));

					/* Launch 9-threaded kernel to set up ae[3][3] and oe[3][3]
					 * and also set the bistatic flag 		 */
					THD.x = 9;
					set_ae_oe_bistatic_krnl<<<BLK,THD>>>(ddat, s, f);
					checkErrorAfterKernelLaunch("lghtcrv_set_ae_oe_bistatic_krnl, line ");

					/* Need to get pos->n for kernel launch first */
					get_pos_n_krnl<<<1,1>>>();
					checkErrorAfterKernelLaunch("get_lghtcrv_pos_n_krnl, line ");
					gpuErrchk(hipMemcpyFromSymbol(&pos_n, HIP_SYMBOL(dpos_n), sizeof(dpos_n),
							0, hipMemcpyDeviceToHost));

					/* Configure & launch posclr_krnl to initialize POS view */
					xspan = 2*pos_n+1;
					nThreads = xspan*xspan;
					BLK.x = floor((maxThreadsPerBlock-1 + nThreads)/maxThreadsPerBlock);
					THD.x = maxThreadsPerBlock;
					posclr_krnl<<<BLK,THD>>>(pos_n, xspan);
					checkErrorAfterKernelLaunch("posclr_krnl, line ");

					/* Determine which POS pixels cover the target */
//					for (c=0; c<mod->shape.ncomp; c++)
					if (STREAMS)
						printf("in vary params_cuda, fix this (lightcurve posvis call");
					//	posvis_cuda_streams(dpar, dmod, ddat, orbit_offset,s,i, 0, 0, c);
					else
						posvis_cuda_2(dpar, dmod, ddat, orbit_offset,s,i, 0, 0, c);

				 /* Now view the model from the source (sun) and get the facet
				  * number and distance toward the source of each pixel in this
				  * projected view; use this information to determine which POS
				  * pixels are shadowed */
					if (lghtcrv_bistatic) {
//						for (c=0; c<mod->shape.ncomp; c++)
						posvis_cuda_2(dpar,dmod,ddat,orbit_offset,s,i,1,0,c);

						/* Launch parameters still same as before for posclr */
						//posmask_universal_krnl<<<BLK,THD>>>(dpar, nThreads, xspan);
						posmask_krnl<<<BLK,THD>>>(dpar, nThreads, xspan);
						checkErrorAfterKernelLaunch("posmask_krnl (vary_params_cuda.cu)");
					}
					/* Compute model brightness for this lightcurve point */
					/* lghtcrv->y[ncalc]: calculated points for interpolation,
					 * ncalc-points total 					 */
					lghtcrv_y = apply_photo_cuda(dmod, ddat, 0, s, i);

					/* Now launch a kernel to copy it over to the actual lghtcrv */
					lghtcrv_copy_y_krnl<<<1,1>>>(ddat, lghtcrv_y, s, i);
					checkErrorAfterKernelLaunch("vp_copy_lghtcrv_y_krnl, line ");
				}

				/* Now that we have calculated the model lightcurve brightnesses
				 * y at each of the epochs x, we use cubic spline interpolation
				 * (Numerical Recipes routines spline and splint) to get model
				 * lightcurve brightness fit[i] at each OBSERVATION epoch t[i],
				 * with i=1,2,...,n.  This will allow us (in routine chi2) to
				 * compare model to data (fit[i] to obs[i]) to get chi squared.
				 * Note that vector y2 contains the second derivatives of the
				 * interpolating function at the calculation epochs x. */

				/* To-Do:   The splint kernel can be sped up by implementing a
				 * 			proper parallel reduction.				 */

				/* First make a pointer for u and hipMalloc device memory for it */
				double *u;
				gpuErrchk(hipMalloc((void**)&u, sizeof(double) * lghtcrv_n));

				BLK.x = floor((maxThreadsPerBlock - 1 + ncalc) /
						maxThreadsPerBlock);
				THD.x = maxThreadsPerBlock; // Thread block dimensions
				lghtcrv_spline_krnl<<<BLK,THD>>>(ddat, s, 2.0e30, 2.0e30, u);
				checkErrorAfterKernelLaunch("lghtcrv_spline_krnl, line ");

				BLK.x = floor((maxThreadsPerBlock - 1 + lghtcrv_n) /
						maxThreadsPerBlock);
				lghtcrv_splint_krnl<<<BLK,THD>>>(ddat, s);
				checkErrorAfterKernelLaunch("lghtcrv_splint_krnl, line ");
				/* Cleanup */
				hipFree(u);
			}
			break;
		default:
			bailout("vary_params.c: can't handle this dataset type yet\n");
		}
	}

	/* Launch single-threaded kernel to wrap things up in vary_params_cuda,
	 * includes manipulation of the hostside variables deldop_zmax, rad_xsec,
	 * opt_brightness, and cos_subradarlat		 */
	double dd_zmax, rd_xsec, opt_brtns, cs_sb_rdr_lat;
	vp_set_four_parameters_krnl<<<1,1>>>(ddat);
	checkErrorAfterKernelLaunch("vp_set_four_parameters, line ");
	gpuErrchk(hipMemcpyFromSymbol(&dd_zmax, HIP_SYMBOL(vp_deldop_zmax),
			sizeof(double), 0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&rd_xsec, HIP_SYMBOL(vp_rad_xsec),
			sizeof(double), 0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&opt_brtns, HIP_SYMBOL(vp_opt_brightness),
			sizeof(double), 0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&cs_sb_rdr_lat, HIP_SYMBOL(vp_cos_subradarlat),
			sizeof(double), 0, hipMemcpyDeviceToHost));

	*deldop_zmax = dd_zmax;
	*rad_xsec = rd_xsec;
	*opt_brightness = opt_brtns;
	*cos_subradarlat = cs_sb_rdr_lat;
	}


#include "hip/hip_runtime.h"

extern "C" {
#include "../shape/head.h"
#include "../shape/shape-cuda.h"
}
__device__ float bf_sum_oovs=0.0;
__device__ int dbg_ndop1, dbg_ndel1, dbg_xlim0, dbg_xlim1, dbg_ylim0, dbg_ylim1;
__device__ float zsum=0.0, cosa_sum=0.0;

__global__ void bf_deldop_dbg2_krnl(struct par_t *dpar, struct dat_t *ddat,
		int s, int f) {
	/* Single-threaded kernel */
	int idel, idop, i;

	if (threadIdx.x == 0) {
		int ndel = ddat->set[s].desc.deldop.frame[f].ndel;
		int ndop = ddat->set[s].desc.deldop.frame[f].ndop;
		int initial = ndel*ndop - 11;
		for (idel=1; idel<=ndel; idel++)
			for (idop=1; idop<=ndop; idop++)
				bf_sum_oovs += ddat->set[s].desc.deldop.frame[f].oneovervar[idel][idop];
	}
}
__global__ void dbg_print_fit_krnl1(struct dat_t *ddat, int s, int f){
	/* Single-threaded debug kernel */
	if (threadIdx.x == 0) {
		switch (ddat->set[s].type) {
		case DELAY:
			dbg_ndel1 = ddat->set[s].desc.deldop.frame[f].ndel;
			dbg_ndop1 = ddat->set[s].desc.deldop.frame[f].ndop;
			dbg_xlim0 = ddat->set[s].desc.deldop.frame[f].pos.xlim[0];
			dbg_xlim1 = ddat->set[s].desc.deldop.frame[f].pos.xlim[1];
			dbg_ylim0 = ddat->set[s].desc.deldop.frame[f].pos.ylim[0];
			dbg_ylim1 = ddat->set[s].desc.deldop.frame[f].pos.ylim[1];
			break;
		case DOPPLER:
			dbg_ndop1 = ddat->set[s].desc.doppler.frame[f].ndop;
			dbg_xlim0 = ddat->set[s].desc.doppler.frame[f].pos.xlim[0];
			dbg_xlim1 = ddat->set[s].desc.doppler.frame[f].pos.xlim[1];
			dbg_ylim0 = ddat->set[s].desc.doppler.frame[f].pos.ylim[0];
			dbg_ylim1 = ddat->set[s].desc.doppler.frame[f].pos.ylim[1];
			break;
		}
	}
}
__global__ void dbg_print_fit_krnl2(struct dat_t *ddat, double *fit, int s, int f) {
	/* ndop-threaded kernel */
	int idop = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (idop <= dbg_ndop1) {
		fit[idop] = ddat->set[s].desc.doppler.frame[f].fit_s[idop];
	}
}
__global__ void dbg_print_poz_krnl(struct dat_t *ddat, float *zz, int s, int f, int size) {
	/* ndop-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;

	if (offset < size) {
		if (ddat->set[s].type == DELAY)
			zz[offset] = ddat->set[s].desc.deldop.frame[f].pos.z_s[offset];
		if (ddat->set[s].type == DOPPLER)
			zz[offset] = ddat->set[s].desc.doppler.frame[f].pos.z_s[offset];
	}
}
__global__ void dbg_print_poz_af_krnl(struct dat_t *ddat, float *zz0, float *zz1,
		float *zz2, float *zz3, int s, int size) {
	/* ndop-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;

	if (offset < size) {
		if (ddat->set[s].type == DELAY) {
			zz0[offset] = ddat->set[s].desc.deldop.frame[0].pos.z_s[offset];
			zz1[offset] = ddat->set[s].desc.deldop.frame[1].pos.z_s[offset];
			zz2[offset] = ddat->set[s].desc.deldop.frame[2].pos.z_s[offset];
			zz3[offset] = ddat->set[s].desc.deldop.frame[3].pos.z_s[offset];
		}
		if (ddat->set[s].type == DOPPLER) {
			zz0[offset] = ddat->set[s].desc.doppler.frame[0].pos.z_s[offset];
			zz1[offset] = ddat->set[s].desc.doppler.frame[1].pos.z_s[offset];
			zz2[offset] = ddat->set[s].desc.doppler.frame[2].pos.z_s[offset];
			zz3[offset] = ddat->set[s].desc.doppler.frame[3].pos.z_s[offset];
		}
	}
}
__global__ void dbg_print_cose_krnl(struct dat_t *ddat, float *cose, int s, int f, int size) {
	/* ndop-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;

	if (offset < size) {
		if (ddat->set[s].type == DELAY)
			cose[offset] = ddat->set[s].desc.deldop.frame[f].pos.cose_s[offset];
		if (ddat->set[s].type == DOPPLER)
			cose[offset] = ddat->set[s].desc.doppler.frame[f].pos.cose_s[offset];
	}
}
__global__ void dbg_print_cos_af_krnl(struct dat_t *ddat, float *cos0, float *cos1,
		float *cos2, float *cos3, int s, int size) {
	/* ndop-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;

	if (offset < size) {
		if (ddat->set[s].type == DELAY) {
			cos0[offset] = ddat->set[s].desc.deldop.frame[0].pos.cose_s[offset];
			cos1[offset] = ddat->set[s].desc.deldop.frame[1].pos.cose_s[offset];
			cos2[offset] = ddat->set[s].desc.deldop.frame[2].pos.cose_s[offset];
			cos3[offset] = ddat->set[s].desc.deldop.frame[3].pos.cose_s[offset];
		}
		if (ddat->set[s].type == DOPPLER) {
			cos0[offset] = ddat->set[s].desc.doppler.frame[0].pos.cose_s[offset];
			cos1[offset] = ddat->set[s].desc.doppler.frame[1].pos.cose_s[offset];
			cos2[offset] = ddat->set[s].desc.doppler.frame[2].pos.cose_s[offset];
			cos3[offset] = ddat->set[s].desc.doppler.frame[3].pos.cose_s[offset];
		}
	}
}
__global__ void dbg_print_fit_deldop_krnl2(struct dat_t *ddat, double *fit, int s, int f){
	/* ndel*ndop-threaded kernel */
	int offset = blockIdx.x * blockDim.x + threadIdx.x;

	if (offset < (dbg_ndop1*dbg_ndel1))
		fit[offset] = ddat->set[s].desc.deldop.frame[f].fit_s[offset];
}
__host__ void dbg_print_fit(struct dat_t *ddat, int s, int f) {
	/* Debug function that prints all Doppler frame fit values to csv */

	int idop, nThreads, ndop, xlim[2], ylim[2];
	FILE *fp_fit;
	char *filename_fit;
	double *fit;
	dim3 BLK,THD;

	filename_fit = "dbg_fit_cuda.csv";
	printf("\n %sfile created",filename_fit);
	printf("\n\nFilename: %s",filename_fit);

	/* Launch 1st debug kernel to get ndop and xlim/ylim	 */
	dbg_print_fit_krnl1<<<1,1>>>(ddat, s, f);
	checkErrorAfterKernelLaunch("dbg_print_fit_krnl1");
	deviceSyncAfterKernelLaunch("dbg_print_fit_krnl2");
	gpuErrchk(hipMemcpyFromSymbol(&xlim[0], HIP_SYMBOL(dbg_xlim0), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&xlim[1], HIP_SYMBOL(dbg_xlim1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ylim[0], HIP_SYMBOL(dbg_ylim0), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ylim[1], HIP_SYMBOL(dbg_ylim1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ndop, HIP_SYMBOL(dbg_ndop1), sizeof(int),
			0, hipMemcpyDeviceToHost));

	nThreads = (xlim[1] - xlim[0] + 1) * (ylim[1] - ylim[0] + 1);
	cudaCalloc((void**)&fit, sizeof(double), ndop);
	fit -= 1;
	int maxThreads = 128;
	BLK.x = floor((maxThreads - 1 + ndop)/maxThreads);
	THD.x = maxThreads; // Thread block dimensions

	dbg_print_fit_krnl2<<<BLK,THD>>>(ddat, fit, s, f);
	checkErrorAfterKernelLaunch("dbg_print_fit_krnl_2");
	deviceSyncAfterKernelLaunch("dbg_print_fit_krnl_2");

	fp_fit = fopen(filename_fit, "w+");
	fprintf(fp_fit, "idop , ");
	for (idop=1; idop<=ndop; idop++)
		fprintf(fp_fit,	"\n%i , %g", idop, fit[idop]);
	fprintf(fp_fit, "\nxlim0 , %i", xlim[0]);
	fprintf(fp_fit, "\nxlim1 , %i", xlim[1]);
	fprintf(fp_fit, "\nylim0 , %i", ylim[0]);
	fprintf(fp_fit, "\nylim1 , %i", ylim[1]);
	fprintf(fp_fit, "\nthreads , %i", nThreads);
	fclose(fp_fit);
	//hipFree(fit);
}
void dbg_print_fit_host(struct dat_t *ddat, int s, int f) {
	/* Debug function that prints all Doppler frame fit values to csv */

	int idop, nThreads;
	FILE *fp_fit;
	char *filename_fit;
	filename_fit = "CPU_doppler_fit.csv";
	nThreads = (ddat->set[s].desc.doppler.frame[f].pos.xlim[1]-
			ddat->set[s].desc.doppler.frame[f].pos.xlim[0]+1)*
					(ddat->set[s].desc.doppler.frame[f].pos.ylim[1]-
							ddat->set[s].desc.doppler.frame[f].pos.ylim[0]+1);

	printf("\n %sfile created",filename_fit);
	fp_fit = fopen(filename_fit, "w+");

	fprintf(fp_fit, "idel , ");

	for (idop=1; idop<=ddat->set[s].desc.doppler.frame[f].ndop; idop++)
		fprintf(fp_fit,	"\n%i , %g", idop, ddat->set[s].desc.doppler.frame[f].fit[idop]);

	fprintf(fp_fit, "\nxlim0 , %i", ddat->set[s].desc.doppler.frame[f].pos.xlim[0]);
	fprintf(fp_fit, "\nxlim1 , %i", ddat->set[s].desc.doppler.frame[f].pos.xlim[1]);
	fprintf(fp_fit, "\nylim0 , %i", ddat->set[s].desc.doppler.frame[f].pos.ylim[0]);
	fprintf(fp_fit, "\nylim1 , %i", ddat->set[s].desc.doppler.frame[f].pos.ylim[1]);
	fprintf(fp_fit, "\nthreads , %i", nThreads);
	fclose(fp_fit);
}
__host__ void dbg_print_deldop_fit(struct dat_t *ddat, int s, int f) {
	/* Debug function that prints all Doppler frame fit values to csv */

	int idop, ndop, idel, ndel, nbins, nThreads, offset, xlim[2], ylim[2];
	FILE *fp_fit;
	char *filename_fit;
	double *fit_dd;
	dim3 BLK,THD;

	filename_fit = "dbg_fit_cuda.csv";
	printf("\n %sfile created",filename_fit);
	printf("\n\nFilename: %s",filename_fit);

	/* Launch 1st debug kernel to get ndop and xlim/ylim	 */
	dbg_print_fit_krnl1<<<1,1>>>(ddat, s, f);
	checkErrorAfterKernelLaunch("dbg_print_fit_krnl1");
	deviceSyncAfterKernelLaunch("dbg_print_fit_krnl2");
	gpuErrchk(hipMemcpyFromSymbol(&xlim[0], HIP_SYMBOL(dbg_xlim0), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&xlim[1], HIP_SYMBOL(dbg_xlim1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ylim[0], HIP_SYMBOL(dbg_ylim0), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ylim[1], HIP_SYMBOL(dbg_ylim1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ndop, HIP_SYMBOL(dbg_ndop1), sizeof(int),
			0, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpyFromSymbol(&ndel, HIP_SYMBOL(dbg_ndel1), sizeof(int),
				0, hipMemcpyDeviceToHost));

	nThreads = (xlim[1] - xlim[0] + 1) * (ylim[1] - ylim[0] + 1);
	nbins = ndop * ndel;
	cudaCalloc((void**)&fit_dd, sizeof(double), nbins);

	BLK.x = floor((maxThreadsPerBlock - 1 + nbins)/maxThreadsPerBlock);
	THD.x = maxThreadsPerBlock; // Thread block dimensions

	dbg_print_fit_deldop_krnl2<<<BLK,THD>>>(ddat, fit_dd, s, f);
	checkErrorAfterKernelLaunch("dbg_print_fit_deldop_krnl_2");
	deviceSyncAfterKernelLaunch("dbg_print_fit_deldop_krnl_2");

	fp_fit = fopen(filename_fit, "w+");

	/* Print top corner idop/idel label */
	fprintf(fp_fit, "idop/idel , ");

	/* Print top row idel values */
	for (idel=1; idel<=ndel; idel++)
		fprintf(fp_fit, "%i , ", idel);

	/* Print first entry in every row (except 1st): idop */
	for (idop=1; idop<=ndop; idop++) {
		fprintf(fp_fit,	"\n%i , ", idop);

		/* Write the rest of the row values: fit[idel][idop] */
		for (idel=1; idel<=ndel; idel++) {
			offset = (idop-1)*ndel + (idel-1);
			fprintf(fp_fit, " %g , ", fit_dd[offset]);
		}

	}
	fprintf(fp_fit, "\nxlim0 , %i", xlim[0]);
	fprintf(fp_fit, "\nxlim1 , %i", xlim[1]);
	fprintf(fp_fit, "\nylim0 , %i", ylim[0]);
	fprintf(fp_fit, "\nylim1 , %i", ylim[1]);
	fprintf(fp_fit, "\nthreads , %i", nThreads);
	fclose(fp_fit);
	//hipFree(fit_dd);
}
__host__ void dbg_print_deldop_fit_host(struct dat_t *ddat, int s, int f) {
	/* Debug function that prints all Delay-Doppler frame fit values to csv */

	int idop, ndop, idel, ndel, nThreads, xlim[2], ylim[2];
	FILE *fp_fit;
	char *filename_fit;

	filename_fit = "CPU_deldop_fit.csv";
	printf("\n %sfile created",filename_fit);

	for (idop=0;idop<2;idop++){
		xlim[idop] = ddat->set[s].desc.deldop.frame[f].pos.xlim[idop];
		ylim[idop] = ddat->set[s].desc.deldop.frame[f].pos.ylim[idop];}

	ndel = ddat->set[s].desc.deldop.frame[f].ndel;
	ndop = ddat->set[s].desc.deldop.frame[f].ndop;
	nThreads = (xlim[1] - xlim[0] + 1) * (ylim[1] - ylim[0] + 1);
	fp_fit = fopen(filename_fit, "w+");

	/* Print top corner idop/idel label */
	fprintf(fp_fit, "idop/idel , ");

	/* Print top row idel values */
	for (idel=1; idel<=ndel; idel++)
		fprintf(fp_fit, "%i , ", idel);

	/* Print first entry in every row (except 1st): idop */
	for (idop=1; idop<=ndop; idop++) {
		fprintf(fp_fit,	"\n%i , ", idop);

		/* Write the rest of the row values: fit[idel][idop] */
		for (idel=1; idel<=ndel; idel++)
			fprintf(fp_fit, " %g , ", ddat->set[s].desc.deldop.frame[f].fit[idel][idop]);
	}
	fprintf(fp_fit, "\nxlim0 , %i", xlim[0]);
	fprintf(fp_fit, "\nxlim1 , %i", xlim[1]);
	fprintf(fp_fit, "\nylim0 , %i", ylim[0]);
	fprintf(fp_fit, "\nylim1 , %i", ylim[1]);
	fprintf(fp_fit, "\nthreads , %i", nThreads);
	fclose(fp_fit);
}

__global__ void dbg_print_RandC_krnl(struct mod_t *dmod) {
	/* Single-threaded kernel */
	if (threadIdx.x == 0) {
		printf("C = %f\n", dmod->photo.radar[0].RC.C.val);
		printf("R = %f\n", dmod->photo.radar[0].RC.R.val);
	}
}
__host__ void dbg_print_RandC(struct mod_t *dmod) {
	/* Debug function that retrieves and prints the following:
	 * 	dmod->photo.radar[0].RC.C.val
	 * 	dmod->photo.radar[0].RC.R.val	 */

	/* Just call kernel */
	dbg_print_RandC_krnl<<<1,1>>>(dmod);
	checkErrorAfterKernelLaunch("dbg_print_RandC_krnl");
	deviceSyncAfterKernelLaunch("dbg_print_RandC_krnl");
}
void dbg_print_RandC_host(struct mod_t *dmod) {
	/* Same as the __host__function, but intended for the CPU code version */
	printf("C = %f\n", dmod->photo.radar[0].RC.C.val);
	printf("R = %f\n", dmod->photo.radar[0].RC.R.val);
}
__host__ void dbg_print_array(float *data, int x, int y) {
	/* Debug function that prints all elements in data to a csv in x col * y rows */

	int n, i, j;
	float *host;
	FILE *fp_fit;
	char *filename_fit;
	double *fit;
	dim3 BLK,THD;

	filename_fit = "dbg_array_cuda.csv";
	printf("\n %sfile created",filename_fit);
	printf("\n\nFilename: %s",filename_fit);

	n = x*y;
	host = (float *) malloc(n*sizeof(float));
	gpuErrchk(hipMemcpy(host, data, n*sizeof(float), hipMemcpyDeviceToHost));

	fp_fit = fopen(filename_fit, "w+");
	/* Print top corner idop/idel label */
	fprintf(fp_fit, "i/j , ");

	/* Print top row idel values */
	for (i=0; i<x; i++)
		fprintf(fp_fit, "%i , ", i);

	/* Print first entry in every row (except 1st): idop */
	for (j=1; j<y; j++) {
		fprintf(fp_fit,	"\n%i , ", j);

		/* Write the rest of the row values: fit[idel][idop] */
		for (i=0; i<x; i++)
			fprintf(fp_fit, " %g , ", host[j*x + i]);
	}
	fclose(fp_fit);
	free(host);
}
__host__ void dbg_print_array1D(float *data, int size) {
	/* Debug function that prints all elements in data to a csv */

	int i;
	float *host;
	FILE *fp_fit;
	char *filename_fit;
	double *fit;
	dim3 BLK,THD;

	filename_fit = "dbg_array1D_cuda.csv";
	printf("\n %sfile created",filename_fit);
	printf("\n\nFilename: %s",filename_fit);

	host = (float *) malloc(size*sizeof(float));
	gpuErrchk(hipMemcpy(host, data, size*sizeof(float), hipMemcpyDeviceToHost));

	fp_fit = fopen(filename_fit, "w+");
	/* Print top corner idop/idel label */
	fprintf(fp_fit, "i , ");

	/* Print top row idel values */
	for (i=0; i<size; i++)
		fprintf(fp_fit, "%i , ", i);

	/* Go to second row */
	fprintf(fp_fit, "\n , ");

	/* Write the rest of the row values: fit[idel][idop] */
	for (i=0; i<size; i++)
		fprintf(fp_fit, " %g , ", host[i]);

	fclose(fp_fit);
	free(host);
}
__global__ void dbg_sum_up_pos_krnl(struct dat_t *ddat, int s, int f) {
	/* Single-threaded kernel */
	int n, size, i = 0;
	if (threadIdx.x == 0) {
		cosa_sum = zsum = 0.0;
		switch (ddat->set[s].type) {
		case DELAY:
			n = ddat->set[s].desc.deldop.frame[f].pos.n;
			size = (2*n + 1) * (2*n + 1);
			for (i=0; i<size; i++) {
				if (ddat->set[s].desc.deldop.frame[f].pos.z_s[i] > 0.0)
					zsum += ddat->set[s].desc.deldop.frame[f].pos.z_s[i];
				cosa_sum += ddat->set[s].desc.deldop.frame[f].pos.cose_s[i];
			}
			printf("sum of z_s for Deldop frame #%i is %3.3f\n", f, zsum);
			printf("sum of cosa_s for Deldop frame #%i is %g\n", f, cosa_sum);
			break;
		case DOPPLER:
			n = ddat->set[s].desc.doppler.frame[f].pos.n;
			size = (2*n + 1) * (2*n + 1);
			for (i=0; i<size; i++) {
				if (ddat->set[s].desc.doppler.frame[f].pos.z_s[i] > 0.0)
					zsum += ddat->set[s].desc.doppler.frame[f].pos.z_s[i];
				cosa_sum += ddat->set[s].desc.doppler.frame[f].pos.cose_s[i];
			}
			printf("sum of z_s for Doppler frame #%i is %3.3f\n", f, zsum);
			printf("sum of cosa_s for Doppler frame #%i is %g\n", f, cosa_sum);
			break;
		}
	}
}
__host__ void dbg_sum_up_pos(struct dat_t *ddat, int s, int f) {
	/* Function sums up pos->z and pos->cosa */

	dbg_sum_up_pos_krnl<<<1,1>>>(ddat, s, f);
	checkErrorAfterKernelLaunch("dbg_sum_up_pos_krnl in debug_routines.cu");
}
void dbg_sum_up_pos_host(struct dat_t *ddat, int s, int f) {
	/* Same but for host operation this time */
	int x, y, n, size;
	double zsum_host=0.0, cosa_sum_host=0.0;

	switch (ddat->set[s].type) {
	case DELAY:
		n = ddat->set[s].desc.deldop.frame[f].pos.n;
		size = (2*n + 1) * (2*n + 1);
		for (x=-n; x<=n; x++)
			for (y=-n; y<=n; y++) {
				if (ddat->set[s].desc.deldop.frame[f].pos.z[x][y] > 0.0)
					zsum_host += ddat->set[s].desc.deldop.frame[f].pos.z[x][y];
				cosa_sum_host += ddat->set[s].desc.deldop.frame[f].pos.cose[x][y];
			}
		printf("sum of z for Deldop frame #%i is %3.3f\n", f, zsum_host);
		printf("sum of cose for Deldop frame #%i is %g\n", f, cosa_sum_host);
		break;
	case DOPPLER:
		n = ddat->set[s].desc.doppler.frame[f].pos.n;
		size = (2*n + 1) * (2*n + 1);
		for (x=-n; x<=n; x++)
			for (y=-n; y<=n; y++) {
				if (ddat->set[s].desc.doppler.frame[f].pos.z[x][y] > 0.0)
					zsum_host += ddat->set[s].desc.doppler.frame[f].pos.z[x][y];
				cosa_sum_host += ddat->set[s].desc.doppler.frame[f].pos.cose[x][y];
			}
		printf("sum of z for Doppler frame #%i is %3.3f\n", f, zsum_host);
		printf("sum of cose for Doppler frame #%i is %g\n", f, cosa_sum_host);
		break;
	}
}
__host__ void dbg_check_array_for_content(float *in, int size) {
	/* This debug function sums up the contents of the input array of size
	 * size and counts how many elements are not zero	 */
	int i, count = 0;
	float sum = 0.0, percent;

	for (i=0; i<size; i++) {
		sum += in[i];
		if (in[i] != 0)	count++;
	}
	percent = ((float)count/(float)size) * 100;
	printf("\nInput array sums up to %g and contains %i elements != 0.", sum, count);
	printf("\n(about %2.2f percent of the total elements)", percent);

	fflush( stdout);
}
__host__ void dbg_print_array1(float *in, int size) {
	/* This debug function prints each array value */
	int i;

	for (i=0; i<size; i++) {
		printf("\narray[%i]=%g", i, in[i]);
	}
}
__host__ void dbg_print_pos_z(struct dat_t *ddat, int set, int frm, int n) {
	/* This debug function prints out each pos->z value from z_s */
	/* Debug function that prints all Doppler frame fit values to csv */

	int nThreads, i, j, offset, nx;
	FILE *fp_z;
	char *filename_z;
	float *zz;
	dim3 BLK,THD;

	nx = 2*n + 1;
	filename_z = "dbg_zz_cuda.csv";
	printf("\n %sfile created",filename_z);
	printf("\n\nFilename: %s",filename_z);

	nThreads = (2*n+1)*(2*n+1);
	hipMallocManaged((void**)&zz, sizeof(float)*nThreads, hipMemAttachHost);

	BLK.x = floor((maxThreadsPerBlock - 1 + nThreads)/maxThreadsPerBlock);
	THD.x = maxThreadsPerBlock; // Thread block dimensions

	dbg_print_poz_krnl<<<BLK,THD>>>(ddat, zz, set, frm, nThreads);
	checkErrorAfterKernelLaunch("dbg_print_fit_deldop_krnl_2");
	deviceSyncAfterKernelLaunch("dbg_print_fit_deldop_krnl_2");

	fp_z = fopen(filename_z, "w+");

	/* Print top corner label */
	fprintf(fp_z, "zz , ");

	/* Print top row pos->z index values */
	for (int i=0; i<nx; i++)
		fprintf(fp_z, "%i , ", i);

	/* Print first entry in every row (except 1st): j */
	for (j=0; j<nx; j++) {
		fprintf(fp_z,	"\n%i , ", j);

		/* Write the rest of the row values: fit[idel][idop] */
		for (i=0; i<nx; i++) {
			offset = j*nx + i;
			fprintf(fp_z, " %g , ", zz[offset]);
		}

	}

	fclose(fp_z);
	hipFree(zz);

}
__host__ void dbg_print_pos_z_af(struct dat_t *ddat, int set, int n) {
	/* This debug function prints out each pos->z value from z_s */
	/* Debug function that prints all Doppler frame fit values to csv */

	int nThreads, i, j, offset, nx;
	FILE *fp_z0, *fp_z1, *fp_z2, *fp_z3;
	char *filename_z0, *filename_z1, *filename_z2, *filename_z3;
	float *zz0, *zz1, *zz2, *zz3;
	dim3 BLK,THD;

	nx = 2*n + 1;
	filename_z0 = "dbg_zz0_cuda.csv";
	filename_z1 = "dbg_zz1_cuda.csv";
	filename_z2 = "dbg_zz2_cuda.csv";
	filename_z3 = "dbg_zz3_cuda.csv";
	printf("\n %sfile created",filename_z0);
	printf("\n\nFilename: %s",filename_z0);

	nThreads = (2*n+1)*(2*n+1);
	cudaCalloc((void**)&zz0, sizeof(float), nThreads);
	cudaCalloc((void**)&zz1, sizeof(float), nThreads);
	cudaCalloc((void**)&zz2, sizeof(float), nThreads);
	cudaCalloc((void**)&zz3, sizeof(float), nThreads);

	BLK.x = floor((maxThreadsPerBlock - 1 + nThreads)/maxThreadsPerBlock);
	THD.x = maxThreadsPerBlock; // Thread block dimensions

	dbg_print_poz_af_krnl<<<BLK,THD>>>(ddat, zz0, zz1, zz2, zz3, set, nThreads);
	checkErrorAfterKernelLaunch("dbg_print_fit_deldop_krnl_2");
	deviceSyncAfterKernelLaunch("dbg_print_fit_deldop_krnl_2");

	fp_z0 = fopen(filename_z0, "w+");
	fp_z1 = fopen(filename_z1, "w+");
	fp_z2 = fopen(filename_z2, "w+");
	fp_z3 = fopen(filename_z3, "w+");

	/* Print top corner label */
	fprintf(fp_z0, "zz0 , ");
	fprintf(fp_z1, "zz1 , ");
	fprintf(fp_z2, "zz2 , ");
	fprintf(fp_z3, "zz3 , ");

	/* Print top row pos->z index values */
	for (int i=0; i<nx; i++) {
		fprintf(fp_z0, "%i , ", i);
		fprintf(fp_z1, "%i , ", i);
		fprintf(fp_z2, "%i , ", i);
		fprintf(fp_z3, "%i , ", i);
	}


	/* Print first entry in every row (except 1st): j */
	for (j=0; j<nx; j++) {
		fprintf(fp_z0,	"\n%i , ", j);
		fprintf(fp_z1,	"\n%i , ", j);
		fprintf(fp_z2,	"\n%i , ", j);
		fprintf(fp_z3,	"\n%i , ", j);

		/* Write the rest of the row values: fit[idel][idop] */
		for (i=0; i<nx; i++) {
			offset = j*nx + i;
			fprintf(fp_z0, " %g , ", zz0[offset]);
			fprintf(fp_z1, " %g , ", zz1[offset]);
			fprintf(fp_z2, " %g , ", zz2[offset]);
			fprintf(fp_z3, " %g , ", zz3[offset]);
		}

	}

	fclose(fp_z0);
	fclose(fp_z1);
	fclose(fp_z2);
	fclose(fp_z3);
	hipFree(zz0);
	hipFree(zz1);
	hipFree(zz2);
	hipFree(zz3);

}
__host__ void dbg_print_pos_cose_s(struct dat_t *ddat, int set, int frm, int n) {
	/* This debug function prints out each pos->z value from z_s */
	/* Debug function that prints all Doppler frame fit values to csv */

	int nThreads, i, j, offset, nx;
	FILE *fp_z;
	char *filename_z;
	float *cose;
	dim3 BLK,THD;

	nx = 2*n + 1;
	filename_z = "dbg_cose_s_cuda.csv";
	printf("\n %sfile created",filename_z);
	printf("\n\nFilename: %s",filename_z);

	nThreads = (2*n+1)*(2*n+1);
	hipMallocManaged((void**)&cose, sizeof(float)*nThreads, hipMemAttachHost);

	BLK.x = floor((maxThreadsPerBlock - 1 + nThreads)/maxThreadsPerBlock);
	THD.x = maxThreadsPerBlock; // Thread block dimensions

	dbg_print_cose_krnl<<<BLK,THD>>>(ddat, cose, set, frm, nThreads);
	checkErrorAfterKernelLaunch("dbg_print_pos_cose_s_krnl");
	deviceSyncAfterKernelLaunch("dbg_print_pos_cose_s_krnl");

	fp_z = fopen(filename_z, "w+");

	/* Print top corner label */
	fprintf(fp_z, "cose_s , ");

	/* Print top row pos->z index values */
	for (int i=0; i<nx; i++)
		fprintf(fp_z, "%i , ", i);

	/* Print first entry in every row (except 1st): j */
	for (j=0; j<nx; j++) {
		fprintf(fp_z,	"\n%i , ", j);

		/* Write the rest of the row values: fit[idel][idop] */
		for (i=0; i<nx; i++) {
			offset = j*nx + i;
			fprintf(fp_z, " %g , ", cose[offset]);
		}

	}

	fclose(fp_z);
	hipFree(cose);

}
__host__ void dbg_print_cose_af(struct dat_t *ddat, int set, int n) {
	/* This debug function prints out each pos->z value from z_s */
	/* Debug function that prints all Doppler frame fit values to csv */

	int nThreads, i, j, offset, nx;
	FILE *fp_cos0, *fp_cos1, *fp_cos2, *fp_cos3;
	char *filename_cos0, *filename_cos1, *filename_cos2, *filename_cos3;
	float *cos0, *cos1, *cos2, *cos3;
	dim3 BLK,THD;

	nx = 2*n + 1;
	filename_cos0 = "dbg_cos0_cuda.csv";
	filename_cos1 = "dbg_cos1_cuda.csv";
	filename_cos2 = "dbg_cos2_cuda.csv";
	filename_cos3 = "dbg_cos3_cuda.csv";
	printf("\n %sfile created",filename_cos0);

	nThreads = (2*n+1)*(2*n+1);
	cudaCalloc((void**)&cos0, sizeof(float), nThreads);
	cudaCalloc((void**)&cos1, sizeof(float), nThreads);
	cudaCalloc((void**)&cos2, sizeof(float), nThreads);
	cudaCalloc((void**)&cos3, sizeof(float), nThreads);

	BLK.x = floor((maxThreadsPerBlock - 1 + nThreads)/maxThreadsPerBlock);
	THD.x = maxThreadsPerBlock; // Thread block dimensions

	dbg_print_cos_af_krnl<<<BLK,THD>>>(ddat, cos0, cos1, cos2, cos3, set, nThreads);
	checkErrorAfterKernelLaunch("dbg_print_fit_deldop_krnl_2");
	deviceSyncAfterKernelLaunch("dbg_print_fit_deldop_krnl_2");

	fp_cos0 = fopen(filename_cos0, "w+");
	fp_cos1 = fopen(filename_cos1, "w+");
	fp_cos2 = fopen(filename_cos2, "w+");
	fp_cos3 = fopen(filename_cos3, "w+");

	/* Print top corner label */
	fprintf(fp_cos0, "cos0 , ");
	fprintf(fp_cos1, "cos1 , ");
	fprintf(fp_cos2, "cos2 , ");
	fprintf(fp_cos3, "cos3 , ");

	/* Print top row pos->z index values */
	for (int i=0; i<nx; i++) {
		fprintf(fp_cos0, "%i , ", i);
		fprintf(fp_cos1, "%i , ", i);
		fprintf(fp_cos2, "%i , ", i);
		fprintf(fp_cos3, "%i , ", i);
	}


	/* Print first entry in every row (except 1st): j */
	for (j=0; j<nx; j++) {
		fprintf(fp_cos0,	"\n%i , ", j);
		fprintf(fp_cos1,	"\n%i , ", j);
		fprintf(fp_cos2,	"\n%i , ", j);
		fprintf(fp_cos3,	"\n%i , ", j);

		/* Write the rest of the row values: fit[idel][idop] */
		for (i=0; i<nx; i++) {
			offset = j*nx + i;
			fprintf(fp_cos0, " %g , ", cos0[offset]);
			fprintf(fp_cos1, " %g , ", cos1[offset]);
			fprintf(fp_cos2, " %g , ", cos2[offset]);
			fprintf(fp_cos3, " %g , ", cos3[offset]);
		}

	}

	fclose(fp_cos0);
	fclose(fp_cos1);
	fclose(fp_cos2);
	fclose(fp_cos3);
	hipFree(cos0);
	hipFree(cos1);
	hipFree(cos2);
	hipFree(cos3);

}
